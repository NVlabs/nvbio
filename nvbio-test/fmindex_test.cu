#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// fmindex_test.cpp
//

#define MOD_NAMESPACE
#define MOD_NAMESPACE_NAME fmitest
#define MOD_NAMESPACE_BEGIN namespace fmitest {
#define MOD_NAMESPACE_END   }

//#define NVBIO_CUDA_DEBUG
//#define NVBIO_CUDA_ASSERTS

#include <nvbio/basic/omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/dna.h>
#include <nvbio/basic/cached_iterator.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/deinterleaved_iterator.h>
#include <nvbio/fmindex/bwt.h>
#include <nvbio/fmindex/ssa.h>
#include <nvbio/fmindex/fmindex.h>
#include <nvbio/fmindex/backtrack.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/io/fmindex/fmindex.h>

using namespace nvbio;

struct ssa_nop {};

namespace { // anonymous namespace

template <uint32 OCC_INTERVAL,typename FMIndexType, typename word_type>
__global__ void locate_kernel(
    const uint32        n_queries,
    const uint32        QUERY_LEN,
    const uint32        genome_length,
    const word_type*    genome_stream,
    const FMIndexType   fmi,
    const uint32*       input,
    uint32*             output)
{
    typedef typename FMIndexType::index_type index_type;
    typedef typename FMIndexType::range_type range_type;

    const uint32 thread_id = threadIdx.x + blockIdx.x*blockDim.x;
    if (thread_id >= n_queries)
        return;

    typedef const_cached_iterator<const word_type*>                     cached_stream_type;
    typedef PackedStream<cached_stream_type,uint8,2,true,index_type>    genome_stream_type;

    const cached_stream_type cached_genome_stream( genome_stream );
    const genome_stream_type genome( cached_genome_stream );

    const range_type range = match(
        fmi,
        genome + input[ thread_id ],
        QUERY_LEN );

    output[ thread_id ] = uint32( locate( fmi, range.x ) );
}

// test the gpu SSA against the cpu one
template <typename SSA_device, typename SSA_host>
void test_ssa(
    const SSA_device&   ssa_device,
    const SSA_host&     ssa)
{
    thrust::host_vector<typename SSA_device::value_type> d_ssa = ssa_device.m_ssa;
    for (uint32 i = 0; i < d_ssa.size(); ++i)
    {
        if (d_ssa[i] != ssa.m_ssa[i])
        {
            fprintf(stderr, "  \nerror : expected SSA[%u] = %u, got: %u\n", i, (uint32)ssa.m_ssa[i], (uint32)d_ssa[i]);
            exit(1);
        }
    }
}

template <typename index_type>
struct HostData
{
    uint32                              primary;
    thrust::host_vector<index_type>     text;
    thrust::host_vector<index_type>     bwt;
    thrust::host_vector<index_type>     occ;
    thrust::host_vector<index_type>     bwt_occ;
    thrust::host_vector<index_type>     L2;
    thrust::host_vector<uint32>         count_table;
    thrust::host_vector<uint32>         input;
    thrust::host_vector<uint32>         output;
};
template <typename index_type>
struct DeviceData
{
    uint32                              primary;
    thrust::device_vector<index_type>   text;
    thrust::device_vector<index_type>   bwt;
    thrust::device_vector<index_type>   occ;
    thrust::device_vector<index_type>   bwt_occ;
    thrust::device_vector<index_type>   L2;
    thrust::device_vector<uint32>       count_table;
    thrust::device_vector<uint32>       input;
    thrust::device_vector<uint32>       output;

    DeviceData(const HostData<index_type>& data) :
        primary( data.primary ),
        text( data.text ),
        bwt( data.bwt ),
        occ( data.occ ),
        bwt_occ( data.bwt_occ ),
        L2( data.L2 ),
        count_table( data.count_table ),
        input( data.input ),
        output( data.output ) {}
};

template <uint32 OCC_INT, uint32 SA_INT, typename BwtIterator, typename OccIterator, typename SSA, typename index_type>
void do_synthetic_test_device(
    const uint32                    REQS,
    const uint32                    LEN,
    const uint32                    PLEN,
    const HostData<index_type>&     host_data,
    const SSA&                      ssa,
          DeviceData<index_type>&   device_data,
    const OccIterator               occ_it,
    const BwtIterator               bwt_it)
{
    typedef cuda::ldg_pointer<uint32> count_table_type;
    const count_table_type count_table( thrust::raw_pointer_cast( &device_data.count_table.front() ) );

    typedef PackedStream<BwtIterator,uint8,2u,true,index_type> bwt_type;
    typedef rank_dictionary< 2u, OCC_INT, bwt_type, OccIterator, count_table_type > rank_dict_type;
    rank_dict_type rank_dict(
        bwt_type( bwt_it ),
        occ_it,
        count_table );

    typedef SSA_index_multiple_context<SA_INT,const index_type*> ssa_type;
    typedef fm_index< rank_dict_type, ssa_type > fm_index_type;
    fm_index_type temp_fmi(
        LEN,
        device_data.primary,
        thrust::raw_pointer_cast( &device_data.L2.front() ),
        rank_dict,
        ssa_type() );

    //SSA_value_multiple_device ssa_device( ssa );
    //SSA_index_multiple_device<SA_INT> ssa_device( ssa );

    fprintf(stderr,  "    SSA gpu... started\n" );
    Timer timer;
    timer.start();

    SSA_index_multiple_device<SA_INT,index_type> ssa_device( temp_fmi );

    timer.stop();
    fprintf(stderr,  "    SSA gpu... done: %.3fs\n", timer.seconds() );

    // test the gpu SSA against the cpu one
    test_ssa( ssa_device, ssa );

    fprintf(stderr, "    gpu alignment... started\n");

    fm_index_type fmi(
        LEN,
        device_data.primary,
        thrust::raw_pointer_cast( &device_data.L2.front() ),
        rank_dict,
        ssa_device.get_context() );

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );

    const uint32 BLOCK_SIZE = 256;
    const uint32 n_blocks = (REQS + BLOCK_SIZE-1) / BLOCK_SIZE;

    locate_kernel<OCC_INT> <<<n_blocks,BLOCK_SIZE>>>(
        REQS,
        PLEN,
        LEN,
        thrust::raw_pointer_cast( &device_data.text.front() ),
        fmi,
        thrust::raw_pointer_cast( &device_data.input.front() ),
        thrust::raw_pointer_cast( &device_data.output.front() ) );

    hipDeviceSynchronize();

    float time;
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );

    fprintf(stderr, "    gpu alignment... done: %.1fms, A/s: %.2f M\n", time, REQS/(time*1000.0f) );

    thrust::host_vector<uint32> output_h( device_data.output );

    for (uint32 i = 0; i < REQS; ++i)
    {
        if (host_data.output[i] != output_h[i])
        {
            fprintf(stderr, "\nerror : mismatch at %u: expected %u, got %u\n", i, host_data.output[i], output_h[i] );
            exit(1);
        }
    }
}

template <uint32 OCC_INT, uint32 SA_INT, typename SSA>
void synthetic_test_device(
    const uint32            REQS,
    const uint32            LEN,
    const uint32            PLEN,
    const uint32            WORDS,
    const uint32            OCC_WORDS,
    const HostData<uint32>& host_data,
    const SSA&              ssa)
{
    try
    {
        DeviceData<uint32> device_data( host_data );

        // test an FM-index with separate bwt/occ tables
        {
            typedef cuda::ldg_pointer<uint4> iterator_type;

            iterator_type occ_it( (const uint4*)thrust::raw_pointer_cast( &device_data.occ.front() ) );
            iterator_type bwt_it( (const uint4*)thrust::raw_pointer_cast( &device_data.bwt.front() ) );

            do_synthetic_test_device<OCC_INT, SA_INT>(
                REQS,
                LEN,
                PLEN,
                host_data,
                ssa,
                device_data,
                occ_it,
                bwt_it );
        }

        // test an FM-index with interleaved bwt/occ tables
        if (WORDS == OCC_WORDS)
        {
            typedef cuda::ldg_pointer<uint4> bwt_occ_texture;
            bwt_occ_texture bwt_occ_tex( (const uint4*)thrust::raw_pointer_cast( &device_data.bwt_occ.front() ) );

            typedef deinterleaved_iterator<2,0,bwt_occ_texture> bwt_iterator;
            typedef deinterleaved_iterator<2,1,bwt_occ_texture> occ_iterator;

            occ_iterator occ_it( bwt_occ_tex );
            bwt_iterator bwt_it( bwt_occ_tex );

            do_synthetic_test_device<OCC_INT, SA_INT>(
                REQS,
                LEN,
                PLEN,
                host_data,
                ssa,
                device_data,
                occ_it,
                bwt_it );
        }
    }
    catch (std::exception exception)
    {
        fprintf(stderr, "  \nerror : exception caught : %s\n", exception.what());
        exit(1);
    }
    catch (...)
    {
        fprintf(stderr, "  \nerror : unknown exception\n");
        exit(1);
    }
}

template <uint32 OCC_INT, uint32 SA_INT, typename SSA>
void synthetic_test_device(
    const uint32            REQS,
    const uint32            LEN,
    const uint32            PLEN,
    const uint32            WORDS,
    const uint32            OCC_WORDS,
    const HostData<uint64>& host_data,
    const SSA&              ssa)
{
    try
    {
        DeviceData<uint64> device_data( host_data );

        // test an FM-index with separate bwt/occ tables
        {
            typedef cuda::ldg_pointer<uint64> iterator_type;

            iterator_type occ_it( (const uint64*)thrust::raw_pointer_cast( &device_data.occ.front() ) );
            iterator_type bwt_it( (const uint64*)thrust::raw_pointer_cast( &device_data.bwt.front() ) );

            do_synthetic_test_device<OCC_INT, SA_INT>(
                REQS,
                LEN,
                PLEN,
                host_data,
                ssa,
                device_data,
                occ_it,
                bwt_it );
        }

        // test an FM-index with interleaved bwt/occ tables
        if (WORDS == OCC_WORDS)
        {
            typedef cuda::ldg_pointer<uint64> bwt_occ_texture;
            bwt_occ_texture bwt_occ_tex( (const uint64*)thrust::raw_pointer_cast( &device_data.bwt_occ.front() ) );

            typedef deinterleaved_iterator<2,0,bwt_occ_texture> bwt_iterator;
            typedef deinterleaved_iterator<2,1,bwt_occ_texture> occ_iterator;

            occ_iterator occ_it( bwt_occ_tex );
            bwt_iterator bwt_it( bwt_occ_tex );

            do_synthetic_test_device<OCC_INT, SA_INT>(
                REQS,
                LEN,
                PLEN,
                host_data,
                ssa,
                device_data,
                occ_it,
                bwt_it );
        }
    }
    catch (std::exception exception)
    {
        fprintf(stderr, "  \nerror : exception caught : %s\n", exception.what());
        exit(1);
    }
    catch (...)
    {
        fprintf(stderr, "  \nerror : unknown exception\n");
        exit(1);
    }
}

// perform an alignment test on the cpu
//
template <
    typename TextType,
    typename FMIndexType,
    typename index_type>
void synthetic_test_host(
    const uint32                REQS,
    const uint32                PLEN,
    const TextType              text,
    const FMIndexType           fmi,
          HostData<index_type>& data)
{
    fprintf(stderr, "    cpu alignment... started" );

    typedef typename FMIndexType::range_type range_type;

    Timer timer;
    timer.start();
    for (uint32 i = 0; i < REQS; ++i)
    {
        if ((i & 1023) == 0)
            fprintf(stderr, "\r    cpu alignment... started:  %.1f%%   ", 100.0f*float(i)/float(REQS) );
        const range_type range = match(
            fmi,
            text + data.input[i],
            PLEN );

        if (range.y < range.x)
        {
            fprintf(stderr, "  \nerror: unable to match pattern %u\n", data.input[i]);
            exit(1);
        }
        data.output[i] = uint32( locate( fmi, range.x ) );
    }
    timer.stop();

    fprintf(stderr, "\n    cpu alignment... done: %.1fms, A/s: %.2f M\n", timer.seconds()*1000.0f, REQS/(timer.seconds()*1.0e6f) );
}

} // anonymous namespace

template <typename index_type>
void synthetic_test(const uint32 LEN, const uint32 QUERIES)
{
    fprintf(stderr, "  %u-bits synthetic test\n", uint32(sizeof(index_type)*8));

    const uint32 OCC_INT = sizeof(index_type) == sizeof(uint32) ? 64 : 128;
    const uint32 SA_INT  = 32;

    const uint32 SYM_PER_WORD = 4*sizeof(index_type);

    const uint32 PLEN      = 8;
    const uint32 REQS      = nvbio::min( uint32(LEN-PLEN-1u), QUERIES );
    const uint32 WORDS     = (LEN+SYM_PER_WORD-1)/SYM_PER_WORD;
    const uint32 OCC_WORDS = ((LEN+OCC_INT-1) / OCC_INT) * 4;

    Timer timer;

    const uint64 memory_footprint =
        sizeof(index_type)*WORDS +
        sizeof(index_type)*WORDS +
        sizeof(index_type)*OCC_WORDS +
        sizeof(index_type)*uint64(LEN+SA_INT)/SA_INT;

    fprintf(stderr, "  memory  : %.1f MB\n", float(memory_footprint)/float(1024*1024));

    HostData<index_type> data;
    data.text.resize( align<4>(WORDS),      0u );
    data.bwt.resize(  align<4>(WORDS),      0u );
    data.occ.resize(  align<4>(OCC_WORDS),  0u );
    data.L2.resize( 5 );
    data.count_table.resize( 256 );
    data.input.resize( REQS );
    data.output.resize( REQS );

    typedef PackedStream<index_type*,uint8,2,true,index_type> stream_type;
    stream_type text( &data.text[0] );

    for (uint32 i = 0; i < LEN; ++i)
        text[i] = (rand() % 4);

    // print the string
    if (LEN < 64)
    {
        char string[64];
        dna_to_string(
            text,
            text + LEN,
            string );

        fprintf(stderr, "  string : %s\n", string);
    }

    // generate the suffix array
    std::vector<int32> sa( LEN+1, 0u );

    gen_sa( LEN, text, &sa[0] );

    stream_type bwt( &data.bwt[0] );

    data.primary = gen_bwt_from_sa( LEN, text, &sa[0], bwt );

    // set sa[0] to -1 so as to get a modulo for free
    sa[0] = -1;

    // print the string
    if (LEN < 64)
    {
        char string[64];
        dna_to_string(
            bwt,
            bwt + LEN,
            string );

        fprintf(stderr, "  bwt    : %s\n", string);
    }
    fprintf(stderr,"  primary : %d\n", data.primary );

    // buld the occurrence table
    build_occurrence_table<2u,OCC_INT>(
        bwt,
        bwt + LEN,
        &data.occ[0],
        &data.L2[1] );

    // transform the L2 table into a cumulative sum
    data.L2[0] = 0;
    for (uint32 c = 0; c < 4; ++c)
        data.L2[c+1] += data.L2[c];

    // print the L2
    if (LEN < 64)
    {
        for (uint32 i = 0; i < 5; ++i)
            fprintf(stderr, "  L2[%u] : %u\n", i, uint32( data.L2[i] ));
    }

    // generate the count table
    gen_bwt_count_table( &data.count_table[0] );

    // build the interleaved bwt/occ array
    if (WORDS == OCC_WORDS)
    {
        fprintf(stderr,  "  building interleaved bwt/occ... started\n" );

        data.bwt_occ.resize( WORDS*2 );
        if (sizeof(index_type) == 4)
        {
            for (uint32 w = 0; w < WORDS; w += 4)
            {
                data.bwt_occ[ w*2+0 ] = data.bwt[ w+0 ];
                data.bwt_occ[ w*2+1 ] = data.bwt[ w+1 ];
                data.bwt_occ[ w*2+2 ] = data.bwt[ w+2 ];
                data.bwt_occ[ w*2+3 ] = data.bwt[ w+3 ];
                data.bwt_occ[ w*2+4 ] = data.occ[ w+0 ];
                data.bwt_occ[ w*2+5 ] = data.occ[ w+1 ];
                data.bwt_occ[ w*2+6 ] = data.occ[ w+2 ];
                data.bwt_occ[ w*2+7 ] = data.occ[ w+3 ];
            }
        }
        else
        {
            for (uint32 w = 0; w < WORDS; ++w)
            {
                data.bwt_occ[ w*2+0 ] = data.bwt[ w ];
                data.bwt_occ[ w*2+1 ] = data.occ[ w ];
            }
        }
        fprintf(stderr,  "  building interleaved bwt/occ... done\n" );
    }

    typedef PackedStream<const index_type*,uint8,2u,true,index_type> bwt_type;
    typedef rank_dictionary<2u, OCC_INT, bwt_type, const index_type*, const uint32*> rank_dict_type;

    typedef fm_index<rank_dict_type, ssa_nop> temp_fm_index_type;
    temp_fm_index_type temp_fmi(
        LEN,
        data.primary,
        &data.L2[0],
        rank_dict_type(
            bwt_type( &data.bwt[0] ),
            &data.occ[0],
            &data.count_table[0] ),
        ssa_nop() );

  #if 0
    // test the Sampled Suffix Array class
    typedef SSA_value_multiple SSA_type;

    SSA_value_multiple ssa( temp_fmi, SA_INT );
    SSA_value_multiple::context_type ssa_context = ssa.get_context();
  #else
    // test the Sampled Suffix Array class
    typedef SSA_index_multiple<SA_INT,index_type> SSA_type;

    timer.start();

    SSA_type ssa( temp_fmi );

    timer.stop();
    fprintf(stderr, "  SSA cpu time: %.3fs\n", timer.seconds() );

    typename SSA_type::context_type ssa_context = ssa.get_context();
  #endif

    fprintf(stderr, "  SSA test... started\n" );
    for (uint32 i = 1; i < LEN; ++i)
    {
        index_type val;
        if (ssa_context.fetch( index_type(i), val ) && (val != (uint32)sa[i]))
        {
            fprintf(stderr, "  SSA mismatch at %u: expected %d, got: %u\n", i, uint32( sa[i] ), uint32( val ));
            exit(1);
        }
    }
    fprintf(stderr, "  SSA test... done\n" );

    typedef fm_index<rank_dict_type, typename SSA_type::context_type> fm_index_type;
    fm_index_type fmi(
        LEN,
        data.primary,
        &data.L2[0],
        rank_dict_type(
            bwt_type( &data.bwt[0] ),
            &data.occ[0],
            &data.count_table[0] ),
        ssa_context );

    typedef typename fm_index_type::range_type range_type;

    uint8 pattern[PLEN];
    char  pattern_str[PLEN+1];

    fprintf(stderr, "  alignment test... started:" );
    for (uint32 i = 0; i < 1000; ++i)
    {
        fprintf(stderr, "\r  alignment test... started:  %.1f%%   ", 100.0f*float(i)/1000.0f );
        for (uint32 j = 0; j < PLEN; ++j)
            pattern[j] = text[i+j];

        dna_to_string(
            pattern,
            pattern + PLEN,
            pattern_str );

        range_type range = match(
            fmi,
            pattern,
            PLEN );

        if (range.x > range.y)
        {
            fprintf(stderr, "  \nerror : searching for %s @ %u, resulted in (%u,%u)\n", pattern_str, i, uint32( range.x ), uint32( range.y ));
            exit(1);
        }

        // locate the first 100 alignments
        range.y = nvbio::min( range.x + 10u, range.y );

        for (index_type x = range.x; x <= range.y; ++x)
        {
            const uint32 prefix = locate( fmi, x );
            if (prefix >= LEN)
            {
                const range_type inv = inv_psi( fmi, x );
                fprintf(stderr, "  \nerror : searching for %s @ %u, resulted in prefix out of bounds: %u (= sa[%u] + %u)\n", pattern_str, i, prefix, uint32(inv.x), uint32(inv.y));
                exit(1);
            }

            char found_str[PLEN+1];
            dna_to_string(
                text + prefix,
                text + prefix + PLEN,
                found_str );

            if (strcmp( found_str, pattern_str ) != 0)
            {
                const range_type inv = inv_psi( fmi, x );
                fprintf(stderr, "  \nerror : locating %s @ %u at SA=%u in SA(%u,%u), resulted in %s @ %u (= sa[%u] + %u)\n", pattern_str, i, uint32( x ), uint32( range.x ), uint32( range.y ), found_str, prefix, uint32(inv.x), uint32(inv.y));
                exit(1);
            }
            /*{
                const uint2 inv = inv_psi( fmi, x );
                fprintf(stderr, "  locating %s @ %u at %u, matched at %u (= sa[%u] + %u)\n", pattern_str, i, x, prefix, inv.x, inv.y);
            }*/
        }
    }
    fprintf(stderr, "\n  alignment test... done\n" );

    const uint32 SPARSITY = 100;

    data.input[0] = 0;
    for (uint32 i = 1; i < REQS; ++i)
        data.input[i] = (data.input[i-1] + (rand() % SPARSITY)) % (LEN - PLEN);

    fprintf(stderr, "  sorted alignment tests... started\n" );

    synthetic_test_host(
        REQS,
        PLEN,
        text,
        fmi,
        data );

    synthetic_test_device<OCC_INT,SA_INT>(
        REQS,
        LEN,
        PLEN,
        WORDS,
        OCC_WORDS,
        data,
        ssa );

    fprintf(stderr, "  sorted alignment tests... done\n" );

    fprintf(stderr, "  shuffled alignment tests... started\n" );

    for (uint32 i = 0; i < REQS; ++i)
    {
        const uint32 j = i + rand() % (REQS - i);
        std::swap( data.input[i], data.input[j] );
    }

    synthetic_test_host(
        REQS,
        PLEN,
        text,
        fmi,
        data );

    synthetic_test_device<OCC_INT,SA_INT>(
        REQS,
        LEN,
        PLEN,
        WORDS,
        OCC_WORDS,
        data,
        ssa );

    fprintf(stderr, "  shuffled alignment tests... done\n" );
}

//
// A backtracking delegate used to count the total number of occurrences
//
struct CountDelegate
{
    // constructor
    //
    // \param count     pointer to the global counter
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    CountDelegate(uint32* count) : m_count( count ) {}

    // main functor operator
    //
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void operator() (const uint2 range) const
    {
      #if defined(NVBIO_DEVICE_COMPILATION)
        atomicAdd( m_count, range.y + 1u - range.x );
      #else
        *m_count += range.y + 1u - range.x;
      #endif
    }

private:
    uint32* m_count;    // global counter
};

//
// k-mer counting kernel
//
template <typename ReadsView, typename FMIndexType>
NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
void count_core(
    const uint32      read_id,          // read id
    const ReadsView   reads,            // reads view
    const FMIndexType fmi,              // FM-index
    const uint32      len,              // pattern length
    const uint32      seed,             // exact-matching seed length
    const uint32      mismatches,       // number of allowed mismatches after the seed
          uint32*     count)            // global output counter
{
    CountDelegate counter( count );

    typedef typename ReadsView::sequence_stream_type read_stream_type;

    uint4 stack[32*4];

    hamming_backtrack(
        fmi,
        reads.get_read( read_id ).begin(),
        len,
        seed,
        mismatches,
        stack,
        counter );
}

//
// k-mer counting kernel
//
template <typename ReadsView, typename FMIndexType>
__global__
void count_kernel(
    const ReadsView   reads,            // reads view
    const FMIndexType fmi,              // FM-index
    const uint32      len,              // pattern length
    const uint32      seed,             // exact-matching seed length
    const uint32      mismatches,       // number of allowed mismatches after the seed
          uint32*     count)            // global output counter
{
    const uint32 thread_id = threadIdx.x + blockIdx.x*blockDim.x;
    if (thread_id >= reads.size())
        return;

    count_core( thread_id, reads, fmi, len, seed, mismatches, count );
}

//
// run a set of backtracking tests with real data
//
void backtrack_test(const char* index_file, const char* reads_name, const uint32 n_reads)
{
    io::FMIndexDataHost h_fmi;
    if (h_fmi.load( index_file, io::FMIndexData::FORWARD ))
    {
        typedef io::FMIndexData::partial_fm_index_type     host_fmindex_type;
        typedef io::FMIndexDataDevice::fm_index_type       cuda_fmindex_type;

        io::FMIndexDataDevice d_fmi( h_fmi, io::FMIndexDataDevice::FORWARD );

        host_fmindex_type h_fmindex = h_fmi.partial_index();
        cuda_fmindex_type d_fmindex = d_fmi.index();

        io::SequenceDataStream* reads_file = io::open_sequence_file(
            reads_name,
            io::Phred,
            n_reads,
            50 );

        if (reads_file == NULL)
        {
            log_error(stderr, "unable to load \"%s\"\n", reads_name);
            exit(1);
        }

        // create a host-side read batch
        io::SequenceDataHost h_reads_data;

        // load a batch
        if (io::next( DNA_N, &h_reads_data, reads_file, n_reads ) == 0)
        {
            log_error(stderr, "unable to fetch reads from file \"%s\"\n", reads_name);
            exit(1);
        }

        // create a device-side read_batch
        const io::SequenceDataDevice d_reads_data( h_reads_data );

        // create a host-side read batch
        typedef io::SequenceDataAccess<DNA_N> read_access_type;

        // create a read access
        const read_access_type h_reads_view( h_reads_data );
        const read_access_type d_reads_view( d_reads_data );

        thrust::device_vector<uint32> counter(1);
        counter[0] = 0;

        const uint32 blockdim = 128;
        const uint32 n_blocks = (d_reads_data.size() + blockdim - 1) / blockdim;

        // 20-mers, distance=0
        {
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );

            hipEventRecord( start, 0 );

            count_kernel<<<n_blocks,blockdim>>>(
                d_reads_view,
                d_fmindex,
                20u,
                0u,
                0u,
                thrust::raw_pointer_cast( &counter.front() ) );

            hipDeviceSynchronize();
            nvbio::cuda::check_error("count_kernel");

            float time;
            hipEventRecord( stop, 0 );
            hipEventSynchronize( stop );
            hipEventElapsedTime( &time, start, stop );

            fprintf(stderr, "  gpu backtracking (20,0,0)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }
        {
            Timer timer;
            timer.start();

            uint32 counter = 0;
            #pragma omp parallel for
            for (int i = 0; i < (int)h_reads_data.size(); ++i)
            {
                count_core(
                    i,
                    h_reads_view,
                    h_fmindex,
                    20u,
                    0u,
                    0u,
                    &counter );
            }

            timer.stop();
            float time = timer.seconds() * 1000.0f;

            fprintf(stderr, "  cpu backtracking (20,0,0)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }
        // 32-mers, distance=1
        {
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );

            hipEventRecord( start, 0 );

            count_kernel<<<n_blocks,blockdim>>>(
                d_reads_view,
                d_fmindex,
                32u,
                0u,
                1u,
                thrust::raw_pointer_cast( &counter.front() ) );

            hipDeviceSynchronize();
            nvbio::cuda::check_error("count_kernel");

            float time;
            hipEventRecord( stop, 0 );
            hipEventSynchronize( stop );
            hipEventElapsedTime( &time, start, stop );

            fprintf(stderr, "  gpu backtracking (32,1,0)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }
        {
            Timer timer;
            timer.start();

            uint32 counter = 0;

            #pragma omp parallel for
            for (int i = 0; i < (int)h_reads_data.size(); ++i)
            {
                count_core(
                    i,
                    h_reads_view,
                    h_fmindex,
                    32u,
                    0u,
                    1u,
                    &counter );
            }

            timer.stop();
            float time = timer.seconds() * 1000.0f;

            fprintf(stderr, "  cpu backtracking (32,1,0)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }
        // 50-mers, distance=2, seed=25
        {
            hipEvent_t start, stop;
            hipEventCreate( &start );
            hipEventCreate( &stop );

            hipEventRecord( start, 0 );

            count_kernel<<<n_blocks,blockdim>>>(
                d_reads_view,
                d_fmindex,
                50u,
                25u,
                2u,
                thrust::raw_pointer_cast( &counter.front() ) );

            hipDeviceSynchronize();
            nvbio::cuda::check_error("count_kernel");

            float time;
            hipEventRecord( stop, 0 );
            hipEventSynchronize( stop );
            hipEventElapsedTime( &time, start, stop );

            fprintf(stderr, "  gpu backtracking (50,2,25)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }
        {
            Timer timer;
            timer.start();

            uint32 counter = 0;

            #pragma omp parallel for
            for (int i = 0; i < (int)h_reads_data.size(); ++i)
            {
                count_core(
                    i,
                    h_reads_view,
                    h_fmindex,
                    50u,
                    25u,
                    2u,
                    &counter );
            }

            timer.stop();
            float time = timer.seconds() * 1000.0f;

            fprintf(stderr, "  cpu backtracking (52,2,25)... done: %.1fms, A/s: %.3f M\n", time, d_reads_data.size()/(time*1000.0f) );
        }

        delete reads_file;
    }
    else
        log_warning(stderr, "unable to load \"%s\"\n", index_file);
}

int fmindex_test(int argc, char* argv[])
{
    uint32 synth_len     = 10000000;
    uint32 synth_queries = 64*1024;

    const char* index_name = "./data/human.NCBI36/Human.NCBI36";
    const char* reads_name = "./data/SRR493095_1.fastq.gz";
    uint32 backtrack_queries = 64*1024;
    uint32 threads           = omp_get_num_procs();

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-synth-length" ) == 0)
            synth_len = atoi( argv[++i] )*1000;
        else if (strcmp( argv[i], "-synth-queries" ) == 0)
            synth_queries = atoi( argv[++i] )*1000;
        else if (strcmp( argv[i], "-backtrack-queries" ) == 0)
            backtrack_queries = atoi( argv[++i] ) * 1024;
        else if (strcmp( argv[i], "-index" ) == 0)
            index_name = argv[++i];
        else if (strcmp( argv[i], "-reads" ) == 0)
            reads_name = argv[++i];
        else if (strcmp( argv[i], "-threads" ) == 0)
            threads = atoi( argv[++i] );
    }

    omp_set_num_threads( threads );

    fprintf(stderr, "FM-index test... started\n");

    if (synth_len && synth_queries)
    {
        synthetic_test<uint32>( synth_len, synth_queries );
        synthetic_test<uint64>( synth_len, synth_queries );
    }

    if (backtrack_queries)
        backtrack_test( index_name, reads_name, backtrack_queries );

    fprintf(stderr, "FM-index test... done\n");
    return 0;
}
