/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// nvSetBWT.cu
//

//#define NVBIO_CUDA_DEBUG

#include <hipcub/hipcub.hpp>
#include <nvbio/basic/omp.h>

#include "input_thread.h"
#include <nvbio/basic/pipeline.h>
#include <nvbio/sufsort/sufsort.h>
#include <nvbio/sufsort/sufsort_utils.h>
#include <nvbio/sufsort/file_bwt.h>
#include <nvbio/sufsort/bwte.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/exceptions.h>
#include <nvbio/basic/dna.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/system.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/strings/string_set.h>
#include <nvbio/io/sequence/sequence.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

using namespace nvbio;

static const uint32 SYMBOL_SIZE = io::SequenceDataAccess<DNA>::SEQUENCE_BITS;
static const bool   BIG_ENDIAN  = io::SequenceDataAccess<DNA>::SEQUENCE_BIG_ENDIAN;

typedef io::SequenceDataAccess<DNA>::sequence_storage_iterator  storage_iterator;
typedef io::SequenceDataAccess<DNA>::index_iterator             offsets_iterator;

typedef BWTEContext<SYMBOL_SIZE,BIG_ENDIAN,storage_iterator,offsets_iterator> BWTE_context_type;

///
/// A small class implementing a Pipeline stage reading sequence batches from a file
///
struct SortStage
{
    typedef io::SequenceDataHost   argument_type;
    typedef BWTEBlock              return_type;

    /// constructor
    ///
    ///\param file          input sequence file
    ///\param max_strings   maximum number of strings per batch
    ///\param max_bps       maximum number of base pairs per batch
    ///
    SortStage(BWTE_context_type& context) : m_context( context ) {}

    /// fill the next batch
    ///
    bool process(PipelineContext& context)
    {
        // fetch the input
        io::SequenceDataHost* h_read_data = context.input<io::SequenceDataHost>(0);

        // fetch the output
        BWTEBlock* block = context.output<BWTEBlock>();

        // build a view
        const io::SequenceDataAccess<DNA> h_read_view( *h_read_data );

        m_context.sort_block(
            0u,
            h_read_data->size(),
            h_read_view.sequence_string_set(),
            *block );

        return true;
    }

    BWTE_context_type&  m_context;
};

///
/// A small class implementing a Pipeline stage reading sequence batches from a file
///
struct SinkStage
{
    typedef io::SequenceDataHost   argument_type;

    /// constructor
    ///
    ///\param file          input sequence file
    ///\param max_strings   maximum number of strings per batch
    ///\param max_bps       maximum number of base pairs per batch
    ///
    SinkStage(
        BWTE_context_type&                  context,
        PagedText<SYMBOL_SIZE,BIG_ENDIAN>&  bwt,
        SparseSymbolSet&                    dollars) :
        m_context( context ),
        m_bwt( bwt ),
        m_dollars( dollars ),
        n_reads( 0 ),
        m_time( 0.0f )
    {}

    /// fill the next batch
    ///
    bool process(PipelineContext& context)
    {
        const ScopedTimer<float> timer( &m_time );

        // fetch the input
        io::SequenceDataHost* h_read_data = context.input<io::SequenceDataHost>( 0 );

        // build a view
        const io::SequenceDataAccess<DNA> h_read_view( *h_read_data );

        log_info(stderr, "  block [%u, %u] (%u / %.2fG bps, %.1f M suffixes/s)\n",
            n_reads, n_reads + h_read_data->size(), h_read_data->bps(),
            1.0e-9f * m_bwt.size(),
            m_time ? (1.0e-6f * m_bwt.size()) / m_time : 0.0f );
        log_debug(stderr,"  peak memory : %.1f GB\n", float( peak_resident_memory() ) / float(1024*1024*1024));

        /*
        m_context.append_block(
            0u,
            h_read_data->size(),
            h_read_view.sequence_string_set(),
            m_bwt,
            m_dollars,
            true );
            */

        // fetch the second input
        BWTEBlock* block = context.input<BWTEBlock>( 1 );

        m_context.merge_block(
            0u,
            h_read_data->size(),
            h_read_view.sequence_string_set(),
            *block,
            m_bwt,
            m_dollars,
            true );

        n_reads += h_read_data->size();
        return true;
    }

    BWTE_context_type&                  m_context;
    PagedText<SYMBOL_SIZE,BIG_ENDIAN>&  m_bwt;
    SparseSymbolSet&                    m_dollars;
    uint32                              n_reads;
    float                               m_time;
};

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        log_visible(stderr, "nvSetBWT - Copyright 2013-2014, NVIDIA Corporation\n");
        log_info(stderr, "usage:\n");
        log_info(stderr, "  nvSetBWT [options] input_file output_file\n");
        log_info(stderr, "  options:\n");
        log_info(stderr, "   -v       | --verbosity     int (0-6) [5]\n");
        log_info(stderr, "   -c       | --compression   string    [1R]   (e.g. \"1\", ..., \"9\", \"1R\")\n");
        log_info(stderr, "   -t       | --threads       int       [auto]\n");
        log_info(stderr, "   -b       | --bucketing     int       [16]   (# of bits used for bucketing)\n");
        log_info(stderr, "   -F       | --skip-forward\n");
        log_info(stderr, "   -R       | --skip-reverse\n");
        log_info(stderr, "  output formats:\n");
        log_info(stderr, "    .txt      ASCII\n");
        log_info(stderr, "    .txt.gz   ASCII, gzip compressed\n");
        log_info(stderr, "    .txt.bgz  ASCII, block-gzip compressed\n");
        log_info(stderr, "    .bwt      2-bit packed binary\n");
        log_info(stderr, "    .bwt.gz   2-bit packed binary, gzip compressed\n");
        log_info(stderr, "    .bwt.bgz  2-bit packed binary, block-gzip compressed\n");
        log_info(stderr, "    .bwt4     4-bit packed binary\n");
        log_info(stderr, "    .bwt4.gz  4-bit packed binary, gzip compressed\n");
        log_info(stderr, "    .bwt4.bgz 4-bit packed binary, block-gzip compressed\n");
        return 0;
    }

    const char* reads_name        = argv[argc-2];
    const char* output_name       = argv[argc-1];
    bool  forward                 = true;
    bool  reverse                 = true;
    const char* comp_level        = "1R";
    io::QualityEncoding qencoding = io::Phred33;
    int   threads                 = 0;

    for (int i = 0; i < argc - 2; ++i)
    {
        if ((strcmp( argv[i], "-v" )             == 0) ||
            (strcmp( argv[i], "-verbosity" )     == 0) ||
            (strcmp( argv[i], "--verbosity" )    == 0))
        {
            set_verbosity( Verbosity( atoi( argv[++i] ) ) );
        }
        else if ((strcmp( argv[i], "-F" )             == 0) ||
                 (strcmp( argv[i], "--skip-forward" ) == 0))  // skip forward strand
        {
            forward = false;
        }
        else if ((strcmp( argv[i], "-R" )             == 0) ||
                 (strcmp( argv[i], "--skip-reverse" ) == 0))  // skip reverse strand
        {
            reverse = false;
        }
        else if ((strcmp( argv[i], "-c" )             == 0) ||
                 (strcmp( argv[i], "--compression" )  == 0))  // setup compression level
        {
            comp_level = argv[++i];
        }
        else if ((strcmp( argv[i], "-t" )             == 0) ||
                 (strcmp( argv[i], "--threads" )      == 0))  // setup number of threads
        {
            threads = atoi( argv[++i] );
        }
    }

    try
    {
        log_visible(stderr,"nvSetBWT... started\n");

        // build an output file
        SharedPointer<SetBWTHandler> output_handler = SharedPointer<SetBWTHandler>( open_bwt_file( output_name, comp_level ) );
        if (output_handler == NULL)
        {
            log_error(stderr, "  failed to create an output handler\n");
            return 1;
        }

        // gather device memory stats
        size_t free_device, total_device;
        hipMemGetInfo(&free_device, &total_device);
        cuda::check_error("cuda-check");

        log_stats(stderr, "  device has %ld of %ld MB free\n", free_device/1024/1024, total_device/1024/1024);

    #ifdef _OPENMP
        // now set the number of CPU threads
        omp_set_num_threads( threads > 0 ? threads : omp_get_num_procs() );
        omp_set_nested(1);
        #pragma omp parallel
        {
            log_verbose(stderr, "  running on multiple threads (%d)\n", omp_get_thread_num());
        }
    #endif

        uint32       encoding_flags  = 0u;
        if (forward) encoding_flags |= io::FORWARD;
        if (reverse) encoding_flags |= io::REVERSE_COMPLEMENT;


        log_visible(stderr, "opening read file \"%s\"\n", reads_name);
        SharedPointer<nvbio::io::SequenceDataStream> read_data_file(
            nvbio::io::open_sequence_file(
                reads_name,
                qencoding,
                uint32(-1),
                uint32(-1),
                io::SequenceEncoding( encoding_flags ) )
        );

        if (read_data_file == NULL || read_data_file->is_ok() == false)
        {
            log_error(stderr, "    failed opening file \"%s\"\n", reads_name);
            return false;
        }

        // output vectors
        PagedText<SYMBOL_SIZE,BIG_ENDIAN> bwt;
        SparseSymbolSet                   dollars;

        // get the current device
        int current_device;
        hipGetDevice( &current_device );

        // build a BWTEContext
        BWTE_context_type bwte_context( current_device );

        // find out how big a block can we alloc
        uint32 max_block_suffixes = 256*1024*1024;
        uint32 max_block_strings  =  16*1024*1024;

        while (bwte_context.needed_device_memory( max_block_strings, max_block_suffixes ) + 256u*1024u*1024u >= free_device)
            max_block_suffixes /= 2;

        log_verbose(stderr, "  block size: %u\n", max_block_suffixes);

        // reserve enough space for the block processing
        bwte_context.reserve( max_block_strings, max_block_suffixes );

        hipMemGetInfo(&free_device, &total_device);
        log_stats(stderr, "  device has %ld of %ld MB free\n", free_device/1024/1024, total_device/1024/1024);

        // build the input stage
        InputStage input_stage( read_data_file.get(), max_block_strings, max_block_suffixes - max_block_strings );

        // build the sort stage
        SortStage sort_stage( bwte_context );

        // build the sink
        SinkStage sink_stage( bwte_context, bwt, dollars );

        // build the pipeline
        Pipeline pipeline;
        const uint32 in0 = pipeline.append_stage( &input_stage, 4u );
        const uint32 in1 = pipeline.append_stage( &sort_stage, 4u );
        const uint32 out = pipeline.append_sink( &sink_stage );
        pipeline.add_dependency( in0, out );
        pipeline.add_dependency( in0, in1 );
        pipeline.add_dependency( in1, out );

        Timer timer;
        timer.start();

        // and run it!
        pipeline.run();

        log_info(stderr,"  writing output... started\n");

        // write out the results
        for (uint32 i = 0; i < bwt.page_count(); ++i)
        {
            // find the dollars corresponding to this page
            const uint64 page_begin = bwt.get_page_offset(i);
            const uint64 page_end   = bwt.get_page_offset(i+1);

            const uint64 dollars_begin = nvbio::lower_bound_index(
                page_begin,
                dollars.pos(),
                dollars.size() );

            const uint64 dollars_end = nvbio::lower_bound_index(
                page_end,
                dollars.pos(),
                dollars.size() );

            //log_debug(stderr,"    page[%u] : %llu symbols (%llu,%llu), %llu dollars (%llu,%llu)\n", i, page_end - page_begin, page_begin, page_end, dollars_end - dollars_begin, dollars_begin, dollars_end);

            // and output the page
            output_handler->process(
                bwt.get_page_size(i),
                SYMBOL_SIZE,
                (const uint32*)bwt.get_page(i),
                dollars_end - dollars_begin,
                dollars.pos() + dollars_begin,
                dollars.ids() + dollars_begin );
        }

        log_info(stderr,"  writing output... done\n");

        timer.stop();
        const float time = timer.seconds();

        log_verbose(stderr,"  total time  : %.1fs\n", time);
        log_verbose(stderr,"  peak memory : %.1f GB\n", float( peak_resident_memory() ) / float(1024*1024*1024));
        log_visible(stderr,"nvSetBWT... done\n");
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
        return 1;
    }
    return 0;
}
