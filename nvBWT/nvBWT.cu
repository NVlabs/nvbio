#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// nvBWT.cu
//

#define NVBIO_CUDA_DEBUG

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <string>
#include <vector>
#include <algorithm>
#include <crc/crc.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/exceptions.h>
#include <nvbio/basic/bnt.h>
#include <nvbio/basic/numbers.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/thrust_view.h>
#include <nvbio/basic/dna.h>
#include <nvbio/basic/exceptions.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/fmindex/bwt.h>
#include <nvbio/fasta/fasta.h>
#include <nvbio/io/fmindex/fmindex.h>
#include <nvbio/sufsort/sufsort.h>
#include "filelist.h"

// PAC File Type
enum PacType { BPAC = 0, WPAC = 1 };

using namespace nvbio;

unsigned char nst_nt4_table[256] = {
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 5 /*'-'*/, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4
};


#define RAND    0
#define RAND48  1

#if (GENERATOR == RAND) || ((GENERATOR == RAND48) && defined(WIN32))

// generate random base pairs using rand()
inline void  srand_bp(const unsigned int s) { srand(s); }
inline float frand() { return float(rand()) / float(RAND_MAX); }
inline uint8 rand_bp() { return uint8( frand() * 4 ) & 3; }

#elif (GENERATOR == RAND48)

// generate random base pairs using rand48()
inline void  srand_bp(const unsigned int s) { srand48(s); }
inline uint8 rand_bp() { return uint8( drand48() * 4 ) & 3; }

#endif

struct Counter
{
    Counter() : m_size(0), m_reads(0) {}

    void begin_read() { m_reads++; }
    void end_read() {}

    void id(const uint8 c) {}
    void read(const uint8 c) { m_size++; }

    uint64 m_size;
    uint32 m_reads;
};

template <typename stream_type>
struct Writer
{
    Writer(stream_type stream, const uint32 reads, const uint64 max_size) :
        m_max_size(max_size), m_size(0), m_stream( stream )
    {
        m_bntseq.seed = 11;
        m_bntseq.anns_data.resize( reads );
        m_bntseq.anns_info.resize( reads );

        srand_bp( m_bntseq.seed );

        for (uint32 i = 0; i < 4; ++i)
            m_freq[i] = 0;
    }

    void begin_read()
    {
        BNTAnnData& ann_data = m_bntseq.anns_data[ m_bntseq.n_seqs ];
        ann_data.len    = 0;
        ann_data.gi     = 0;
        ann_data.offset = m_size;
        ann_data.n_ambs = 0;

        BNTAnnInfo& ann_info = m_bntseq.anns_info[ m_bntseq.n_seqs ];
        ann_info.anno   = "null";

        m_lasts = 0;
    }
    void end_read()
    {
        m_bntseq.n_seqs++;
    }

    void id(const uint8 c)
    {
        m_bntseq.anns_info[ m_bntseq.n_seqs ].name.push_back(char(c));
    }
    void read(const uint8 s)
    {
        if (m_size < m_max_size)
        {
            const uint8 c = nst_nt4_table[s];

            const uint8 sc = c < 4 ? c : rand_bp();

            m_stream[ m_size ] = sc;

            // keep track of the symbol frequencies
            ++m_freq[sc];

            if (c >= 4) // we have an N
            {
                if (m_lasts == s) // contiguous N
                {
                    // increment length of the last hole
                    ++m_bntseq.ambs.back().len;
                }
                else
                {
                    // beginning of a new hole
                    BNTAmb amb;
                    amb.len    = 1;
                    amb.offset = m_size;
                    amb.amb    = s;

                    m_bntseq.ambs.push_back( amb );

                    ++m_bntseq.anns_data[ m_bntseq.n_seqs ].n_ambs;
                    ++m_bntseq.n_holes;
                }
            }
            // save last symbol
            m_lasts = s;

            // update sequence length
            BNTAnnData& ann_data = m_bntseq.anns_data[ m_bntseq.n_seqs ];
            ann_data.len++;
        }

        m_bntseq.l_pac++;

        m_size++;
    }

    uint64      m_max_size;
    uint64      m_size;
    stream_type m_stream;

    BNTSeq      m_bntseq;
    uint8       m_lasts;

    uint32      m_freq[4];
};

template <typename StreamType>
bool save_stream(FILE* output_file, const uint64 seq_words, const StreamType* stream)
{
    for (uint64 words = 0; words < seq_words; words += 1024)
    {
        const uint32 n_words = (uint32)nvbio::min( uint64(1024u), uint64(seq_words - words) );
        if (fwrite( stream + words, sizeof(StreamType), n_words, output_file ) != n_words)
            return false;
    }
    return true;
}

//
// .wpac file
//
void save_wpac(const uint32 seq_length, const uint32* string_storage, const char* pac_name)
{
    log_info(stderr, "\nwriting \"%s\"... started\n", pac_name);

    const uint32 seq_words = util::divide_ri( seq_length, 16 );

    FILE* output_file = fopen( pac_name, "wb" );
    if (output_file == NULL)
    {
        log_error(stderr, "  could not open output file \"%s\"!\n", pac_name );
        exit(1);
    }

    // write the sequence length as a uint64
    const uint64 len = seq_length;
    fwrite( &len, sizeof(len), 1u, output_file );

    // save the uint32 stream
    if (save_stream( output_file, seq_words, string_storage ) == false)
    {
        log_error(stderr, "  writing failed!\n");
        exit(1);
    }

    fclose( output_file );
    log_info(stderr, "writing \"%s\"... done\n", pac_name);
}

//
// .pac file
//
void save_bpac(const uint32 seq_length, const uint32* string_storage, const char* pac_name)
{
    typedef PackedStream<const uint32*,uint8,2,true,int64>       stream_type;
    typedef PackedStream<      uint8*, uint8,2,true,int64>   pac_stream_type;

    log_info(stderr, "\nwriting \"%s\"... started\n", pac_name);

    const uint32 bps_per_byte = 4u;
    const uint64 seq_bytes    = (seq_length + bps_per_byte - 1u) / bps_per_byte;

    FILE* output_file = fopen( pac_name, "wb" );
    if (output_file == NULL)
    {
        log_error(stderr, "  could not open output file \"%s\"!\n", pac_name );
        exit(1);
    }

    // copy the uint32 packed stream into a uint8 pac stream
    thrust::host_vector<uint8> pac_storage( seq_bytes );
    pac_stream_type pac_string( nvbio::plain_view( pac_storage ) );
        stream_type     string( string_storage );

    for (uint32 i = 0; i < seq_length; ++i)
        pac_string[i] = string[i];

    // save the uint8 stream
    if (save_stream( output_file, seq_bytes, nvbio::raw_pointer( pac_storage ) ) == false)
    {
        log_error(stderr, "  writing failed!\n");
        exit(1);
    }
	// the following code makes the pac file size always (l_pac/4+1+1)
    if (seq_length % 4 == 0)
    {
	    const uint8 ct = 0;
	    fwrite( &ct, 1, 1, output_file );
    }
    {
        const uint8 ct = seq_length % 4;
        fwrite( &ct, 1, 1, output_file );
    }

    fclose( output_file );
    log_info(stderr, "writing \"%s\"... done\n", pac_name);
}

//
// .pac | .wpac file
//
void save_pac(const uint32 seq_length, const uint32* string_storage, const char* pac_name, const PacType pac_type)
{
    if (pac_type == BPAC)
        save_bpac( seq_length, string_storage, pac_name );
    else
        save_wpac( seq_length, string_storage, pac_name );
}

//
// .bwt file
//
void save_bwt(const uint32 seq_length, const uint32 seq_words, const uint32 primary, const uint32* cumFreq, const uint32* h_bwt_storage, const char* bwt_name)
{
    log_info(stderr, "\nwriting \"%s\"... started\n", bwt_name);
    FILE* output_file = fopen( bwt_name, "wb" );
    if (output_file == NULL)
    {
        log_error(stderr, "  could not open output file \"%s\"!\n", bwt_name );
        exit(1);
    }
    fwrite( &primary, sizeof(uint32), 1, output_file );
    fwrite( cumFreq,  sizeof(uint32), 4, output_file );
    if (save_stream( output_file, seq_words, h_bwt_storage ) == false)
    {
        log_error(stderr, "  writing failed!\n");
        exit(1);
    }
    fclose( output_file );
    log_info(stderr, "writing \"%s\"... done\n", bwt_name);
}

//
// .sa file
//
void save_ssa(const uint32 seq_length, const uint32 sa_intv, const uint32 ssa_len, const uint32 primary, const uint32* cumFreq, const uint32* h_ssa, const char* sa_name)
{
    log_info(stderr, "\nwriting \"%s\"... started\n", sa_name);
    FILE* output_file = fopen( sa_name, "wb" );
    if (output_file == NULL)
    {
        log_error(stderr, "  could not open output file \"%s\"!\n", sa_name );
        exit(1);
    }

    fwrite( &primary,       sizeof(uint32),     1u,         output_file );
    fwrite( &cumFreq,       sizeof(uint32),     4u,         output_file );
    fwrite( &sa_intv,       sizeof(uint32),     1u,         output_file );
    fwrite( &seq_length,    sizeof(uint32),     1u,         output_file );
    fwrite( &h_ssa[1],      sizeof(uint32),     ssa_len-1,  output_file );
    fclose( output_file );
    log_info(stderr, "writing \"%s\"... done\n", sa_name);
}

int build(
    const char*  input_name,
    const char*  output_name,
    const char*  pac_name,
    const char*  rpac_name,
    const char*  bwt_name,
    const char*  rbwt_name,
    const char*  sa_name,
    const char*  rsa_name,
    const uint64 max_length,
    const PacType pac_type,
    const bool    compute_crc)
{
    std::vector<std::string> sortednames;
    list_files(input_name, sortednames);

    uint32 n_inputs = (uint32)sortednames.size();
    log_info(stderr, "\ncounting bps... started\n");
    // count entire sequence length
    Counter counter;

    for (uint32 i = 0; i < n_inputs; ++i)
    {
        log_info(stderr, "  counting \"%s\"\n", sortednames[i].c_str());

        FASTA_inc_reader fasta( sortednames[i].c_str() );
        if (fasta.valid() == false)
        {
            log_error(stderr, "  unable to open file\n");
            exit(1);
        }

        while (fasta.read( 1024, counter ) == 1024);
    }
    log_info(stderr, "counting bps... done\n");

    const uint64 seq_length   = nvbio::min( (uint64)counter.m_size, (uint64)max_length );
    const uint32 bps_per_word = sizeof(uint32)*4u;
    const uint64 seq_words    = (seq_length + bps_per_word - 1u) / bps_per_word;

    log_info(stderr, "\nstats:\n");
    log_info(stderr, "  reads           : %u\n", counter.m_reads );
    log_info(stderr, "  sequence length : %llu bps (%.1f MB)\n",
        seq_length,
        float(seq_words*sizeof(uint32))/float(1024*1024));
    log_info(stderr, "  buffer size     : %.1f MB\n",
        2*seq_words*sizeof(uint32)/1.0e6f );

    const uint32 sa_intv = nvbio::io::FMIndexData::SA_INT;
    const uint32 ssa_len = (seq_length + sa_intv) / sa_intv;

    // allocate the actual storage
    thrust::host_vector<uint32> h_string_storage( seq_words+1 );
    thrust::host_vector<uint32> h_bwt_storage( seq_words+1 );
    thrust::host_vector<uint32> h_ssa( ssa_len );

    typedef PackedStream<const uint32*,uint8,io::FMIndexData::BWT_BITS,io::FMIndexData::BWT_BIG_ENDIAN> const_stream_type;
    typedef PackedStream<      uint32*,uint8,io::FMIndexData::BWT_BITS,io::FMIndexData::BWT_BIG_ENDIAN>       stream_type;

    stream_type h_string( nvbio::plain_view( h_string_storage ) );

    uint32 cumFreq[4] = { 0, 0, 0, 0 };

    log_info(stderr, "\nbuffering bps... started\n");
    // read all files
    {
        Writer<stream_type> writer( h_string, counter.m_reads, seq_length );

        for (uint32 i = 0; i < n_inputs; ++i)
        {
            log_info(stderr, "  buffering \"%s\"\n", sortednames[i].c_str());

            FASTA_inc_reader fasta( sortednames[i].c_str() );
            if (fasta.valid() == false)
            {
                log_error(stderr, "  unable to open file!\n");
                exit(1);
            }

            while (fasta.read( 1024, writer ) == 1024);
        }

        save_bns( writer.m_bntseq, output_name );

        // compute the cumulative symbol frequencies
        cumFreq[0] = writer.m_freq[0];
        cumFreq[1] = writer.m_freq[1] + cumFreq[0];
        cumFreq[2] = writer.m_freq[2] + cumFreq[1];
        cumFreq[3] = writer.m_freq[3] + cumFreq[2];

        if (cumFreq[3] != seq_length)
        {
            log_error(stderr, "  mismatching symbol frequencies!\n");
            log_error(stderr, "    (%u, %u, %u, %u)\n", cumFreq[0], cumFreq[1], cumFreq[2], cumFreq[3]);
            exit(1);
        }
    }
    log_info(stderr, "buffering bps... done\n");

    if (compute_crc)
    {
        const uint32 crc = crcCalc( h_string, uint32(seq_length) );
        log_info(stderr, "  crc: %u\n", crc);
    }

    try
    {
        BWTParams params;
        uint32    primary;

        thrust::device_vector<uint32> d_string_storage( h_string_storage );
        thrust::device_vector<uint32> d_bwt_storage( seq_words+1 );

        const_stream_type d_string( nvbio::plain_view( d_string_storage ) );
              stream_type d_bwt(    nvbio::plain_view( d_bwt_storage ) );

        Timer timer;

        log_info(stderr, "\nbuilding forward BWT... started\n");
        timer.start();
        {
            StringBWTSSAHandler<const_stream_type,stream_type,uint32*> output(
                seq_length,                         // string length
                d_string,                           // string
                sa_intv,                            // SSA sampling interval
                d_bwt,                              // output bwt iterator
                nvbio::plain_view( h_ssa ) );       // output ssa iterator

            cuda::blockwise_suffix_sort(
                seq_length,
                d_string,
                output,
                &params );

            // remove the dollar symbol
            output.remove_dollar();

            primary = output.primary();
        }
        timer.stop();
        log_info(stderr, "building forward BWT... done: %um:%us\n", uint32(timer.seconds()/60), uint32(timer.seconds())%60);
        log_info(stderr, "  primary: %u\n", primary);

        // save everything to disk
        {
            // copy to the host
            thrust::copy( d_bwt_storage.begin(),
                          d_bwt_storage.begin() + seq_words,
                          h_bwt_storage.begin() );

            if (compute_crc)
            {
                const_stream_type h_bwt( nvbio::plain_view( h_bwt_storage ) );
                const uint32 crc = crcCalc( h_bwt, uint32(seq_length) );
                log_info(stderr, "  crc: %u\n", crc);
            }

            save_pac( seq_length, nvbio::plain_view( h_string_storage ),                           pac_name, pac_type );
            save_bwt( seq_length, seq_words, primary, cumFreq, nvbio::plain_view( h_bwt_storage ), bwt_name );
            save_ssa( seq_length, sa_intv, ssa_len, primary, cumFreq, nvbio::plain_view( h_ssa ),  sa_name );
        }

        // reverse the string in h_string_storage
        {
            // reuse the bwt storage to build the reverse
            uint32* h_rbase_stream = nvbio::plain_view( h_bwt_storage );
            stream_type h_rstring( h_rbase_stream );

            // reverse the string
            for (uint32 i = 0; i < seq_length; ++i)
                h_rstring[i] = h_string[ seq_length - i - 1u ];

            // and now swap the vectors
            h_bwt_storage.swap( h_string_storage );
            h_string = stream_type( nvbio::plain_view( h_string_storage ) );

            // and copy back the new string to the device
            d_string_storage = h_string_storage;
        }

        log_info(stderr, "\nbuilding reverse BWT... started\n");
        timer.start();
        {
            StringBWTSSAHandler<const_stream_type,stream_type,uint32*> output(
                seq_length,                         // string length
                d_string,                           // string
                sa_intv,                            // SSA sampling interval
                d_bwt,                              // output bwt iterator
                nvbio::plain_view( h_ssa ) );       // output ssa iterator

            cuda::blockwise_suffix_sort(
                seq_length,
                d_string,
                output,
                &params );

            // remove the dollar symbol
            output.remove_dollar();

            primary = output.primary();
        }
        timer.stop();
        log_info(stderr, "building reverse BWT... done: %um:%us\n", uint32(timer.seconds()/60), uint32(timer.seconds())%60);
        log_info(stderr, "  primary: %u\n", primary);

        // save everything to disk
        {
            // copy to the host
            thrust::copy( d_bwt_storage.begin(),
                          d_bwt_storage.begin() + seq_words,
                          h_bwt_storage.begin() );

            if (compute_crc)
            {
                const_stream_type h_bwt( nvbio::plain_view( h_bwt_storage ) );
                const uint32 crc = crcCalc( h_bwt, uint32(seq_length) );
                log_info(stderr, "  crc: %u\n", crc);
            }

            save_pac( seq_length, nvbio::plain_view( h_string_storage ),                           rpac_name, pac_type );
            save_bwt( seq_length, seq_words, primary, cumFreq, nvbio::plain_view( h_bwt_storage ), rbwt_name );
            save_ssa( seq_length, sa_intv, ssa_len, primary, cumFreq, nvbio::plain_view( h_ssa ),  rsa_name );
        }
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (...)
    {
        log_error(stderr,"unknown exception caught!\n");
        exit(1);
    }
    return 0;
}

int main(int argc, char* argv[])
{
    crcInit();

    if (argc < 2)
    {
        log_info(stderr, "please specify input and output file names, e.g:\n");
        log_info(stderr, "  nvBWT [options] myinput.*.fa output-prefix\n");
        log_info(stderr, "  options:\n");
        log_info(stderr, "    -v | --verbosity      select verbosity\n");
        log_info(stderr, "    -m | --max-length     clamp input to max_length\n");
        log_info(stderr, "    -b | --byte-packing   output byte packed .pac\n");
        log_info(stderr, "    -w | --word-packing   output word packed .wpac\n");
        log_info(stderr, "    -c | --crc            compute crcs\n");
        log_info(stderr, "    -d | --device         cuda device\n");
        exit(0);
    }

    const char* file_names[2] = { NULL, NULL };
    uint64  max_length  = uint64(-1);
    PacType pac_type    = BPAC;
    bool    crc         = false;
    int     cuda_device = -1;

    uint32 n_files = 0;
    for (int32 i = 1; i < argc; ++i)
    {
        const char* arg = argv[i];

        if ((strcmp( arg, "-m" )                    == 0) ||
            (strcmp( arg, "--max-length" )          == 0))
        {
            max_length = atoi( argv[++i] );
        }
        else if ((strcmp( argv[i], "-v" )           == 0) ||
                 (strcmp( argv[i], "-verbosity" )   == 0) ||
                 (strcmp( argv[i], "--verbosity" )  == 0))
        {
            set_verbosity( Verbosity( atoi( argv[++i] ) ) );
        }
        else if ((strcmp( arg, "-b" )               == 0) ||
                 (strcmp( arg, "--byte-packing" )   == 0))
        {
            pac_type = BPAC;
        }
        else if ((strcmp( arg, "-w" )               == 0) ||
                 (strcmp( arg, "--word-packing" )   == 0))
        {
            pac_type = WPAC;
        }
        else if ((strcmp( arg, "-c" )               == 0) ||
                 (strcmp( arg, "--crc" )            == 0))
        {
            crc = true;
        }
        else if ((strcmp( arg, "-d" )               == 0) ||
                 (strcmp( arg, "--device" )         == 0))
        {
            cuda_device = atoi( argv[++i] );
        }
        else
            file_names[ n_files++ ] = argv[i];
    }

    const char* input_name  = file_names[0];
    const char* output_name = file_names[1];
    std::string pac_string  = std::string( output_name ) + (pac_type == BPAC ? ".pac" : ".wpac");
    const char* pac_name    = pac_string.c_str();
    std::string rpac_string = std::string( output_name ) + (pac_type == BPAC ? ".rpac" : ".rwpac");
    const char* rpac_name   = rpac_string.c_str();
    std::string bwt_string  = std::string( output_name ) + ".bwt";
    const char* bwt_name    = bwt_string.c_str();
    std::string rbwt_string = std::string( output_name ) + ".rbwt";
    const char* rbwt_name   = rbwt_string.c_str();
    std::string sa_string   = std::string( output_name ) + ".sa";
    const char* sa_name     = sa_string.c_str();
    std::string rsa_string  = std::string( output_name ) + ".rsa";
    const char* rsa_name    = rsa_string.c_str();

    log_info(stderr, "max length : %lld\n", max_length);
    log_info(stderr, "input      : \"%s\"\n", input_name);
    log_info(stderr, "output     : \"%s\"\n", output_name);

    try
    {
        int device_count;
        hipGetDeviceCount(&device_count);
        cuda::check_error("cuda-check");

        log_verbose(stderr, "  cuda devices : %d\n", device_count);

        // inspect and select cuda devices
        if (device_count)
        {
            if (cuda_device == -1)
            {
                int            best_device = 0;
                hipDeviceProp_t best_device_prop;
                hipGetDeviceProperties( &best_device_prop, best_device );

                for (int device = 0; device < device_count; ++device)
                {
                    hipDeviceProp_t device_prop;
                    hipGetDeviceProperties( &device_prop, device );
                    log_verbose(stderr, "  device %d has compute capability %d.%d\n", device, device_prop.major, device_prop.minor);
                    log_verbose(stderr, "    SM count          : %u\n", device_prop.multiProcessorCount);
                    log_verbose(stderr, "    SM clock rate     : %u Mhz\n", device_prop.clockRate / 1000);
                    log_verbose(stderr, "    memory clock rate : %.1f Ghz\n", float(device_prop.memoryClockRate) * 1.0e-6f);

                    if (device_prop.major >= best_device_prop.major &&
                        device_prop.minor >= best_device_prop.minor)
                    {
                        best_device_prop = device_prop;
                        best_device      = device;
                    }
                }
                cuda_device = best_device;
            }
            log_verbose(stderr, "  chosen device %d\n", cuda_device);
            {
                hipDeviceProp_t device_prop;
                hipGetDeviceProperties( &device_prop, cuda_device );
                log_verbose(stderr, "    device name        : %s\n", device_prop.name);
                log_verbose(stderr, "    compute capability : %d.%d\n", device_prop.major, device_prop.minor);
            }
            hipSetDevice( cuda_device );
        }

        size_t free, total;
        hipMemGetInfo(&free, &total);
        NVBIO_CUDA_DEBUG_STATEMENT( log_info(stderr,"device mem : total: %.1f GB, free: %.1f GB\n", float(total)/float(1024*1024*1024), float(free)/float(1024*1024*1024)) );

        cuda::check_error("cuda-memory-check");

        return build( input_name, output_name, pac_name, rpac_name, bwt_name, rbwt_name, sa_name, rsa_name, max_length, pac_type, crc );
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (thrust::system::system_error e)
    {
        log_error(stderr, "caught a thrust::system_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
        return 1;
    }
}

