/*
 * Copyright (c) 2012-14, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 *
 *
 *
 *
 *
 *
 *
 */

#include <stdio.h>

#include "util.h"

// pick the best GPU to run on
void gpu_init(void)
{
    int device_count = 0;
    hipError_t err;

    hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax);

    err = hipGetDeviceCount(&device_count);
    if (err != hipSuccess)
    {
        fprintf(stderr, "gpu_init: error enumerating GPUs (%d)\n", err);
        exit(1);
    }

    if (device_count == 0)
    {
        fprintf(stderr, "gpu_init: no devices found\n");
        exit(1);
    }

    // pick the best device to run on
    hipDeviceProp_t best_dev_prop;
    int best_dev = 0;
    int dev;

    hipGetDeviceProperties(&best_dev_prop, 0);

    for(dev = 0; dev < device_count; dev++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev);

        if (prop.major >= best_dev_prop.major &&
            prop.minor >= best_dev_prop.minor)
        {
            best_dev_prop = prop;
            best_dev = dev;
        }
    }

    hipSetDevice(best_dev);
    fprintf(stderr, "Running on %s (%d MB)\n", best_dev_prop.name, best_dev_prop.totalGlobalMem / 1024 / 1024);
}
