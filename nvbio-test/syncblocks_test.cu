#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// syncblocks_test.cu
//
#define NVBIO_CUDA_DEBUG

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/basic/cuda/syncblocks.h>

namespace nvbio {

__global__
void print_kernel(const uint32 n_barriers, cuda::syncblocks barrier, uint32* queue_ptr, uint2* queue)
{
    for (uint32 i = 0; i < n_barriers; ++i)
    {
        if (threadIdx.x == 0)
        {
            const uint32 slot = atomicAdd( queue_ptr, 1u );
            queue[slot] = make_uint2( i, blockIdx.x );
            //NVBIO_CUDA_DEBUG_ASSERT( slot >= i*gridDim.x, "block[%u] got slot %u at iteration %u\n", blockIdx.x, slot, i );
        }

        barrier.enact();
    }
}
__global__
void speed_kernel(const uint32 n_barriers, cuda::syncblocks barrier, uint2* output)
{
    for (uint32 i = 0; i < n_barriers; ++i)
        barrier.enact();

    output[blockIdx.x] = make_uint2( blockIdx.x, 0 );
}

int syncblocks_test()
{
    const uint32 n_barriers = 100;
    cuda::syncblocks_storage barrier_st;

    cuda::syncblocks barrier = barrier_st.get();

    log_info( stderr, "syncblocks test... started\n" );

    const uint32 blockdim = 128;
    const uint32 n_blocks = max_active_blocks( print_kernel, blockdim, 0u );
    log_info( stderr, "  %u blocks\n", n_blocks );

    thrust::device_vector<uint32> dqueue_head( 1u );
    thrust::device_vector<uint2>  dqueue( n_barriers*n_blocks );

    uint32* dqueue_head_ptr = thrust::raw_pointer_cast( &dqueue_head.front() );
    uint2*  dqueue_ptr      = thrust::raw_pointer_cast( &dqueue.front() );

    thrust::host_vector<uint2> hqueue;
    log_info( stderr, "  correctness test... started\n" );

    for (uint32 i = 0; i < 20; ++i)
    {
        // initialize the queue pointer
        dqueue_head[0] = 0;

        // call the testing kernel
        print_kernel<<<n_blocks,blockdim>>>( n_barriers, barrier, dqueue_head_ptr, dqueue_ptr );
        hipDeviceSynchronize();

        nvbio::cuda::thrust_copy_vector(hqueue, dqueue);

        for (uint32 n = 0; n < n_barriers; ++n)
        {
            for (uint32 j = 0; j < n_blocks; ++j)
            {
                const uint2 val = hqueue[n*n_blocks + j];
                if (val.x != n)
                {
                    log_error( stderr, "  found (%u,%u) at position %u:%u, launch %u\n", val.x, val.y, n, j, i );
                    return 1;
                }
            }
        }
    }
    log_info( stderr, "  correctness test... done\n" );

    const uint32 n_tests = 100;

    log_info( stderr, "  speed test... started\n" );

    Timer timer;
    timer.start();

    for (uint32 i = 0; i < n_tests; ++i)
        speed_kernel<<<n_blocks,blockdim>>>( n_barriers, barrier, dqueue_ptr+1 );

    hipDeviceSynchronize();
    timer.stop();

    const float time = timer.seconds() / (n_tests*n_barriers);

    log_info( stderr, "  speed test... done: %.1f ns\n", time * 1.0e6f );

    log_info( stderr, "syncblocks test... done\n" );
    return 0;
}

} // namespace nvbio
