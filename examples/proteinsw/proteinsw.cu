#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// proteinsw.cu
//

#include <nvbio/basic/console.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/strings/string.h>
#include <nvbio/strings/alphabet.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

int8 s_blosum62[] =
{
 4,  0, -2, -1, -2,  0, -2, -1, -1, -1, -1, -2, -4, -1, -1, -1,  1,  0,  0, -3, -2, -2, -1,  0,
 0,  9, -3, -4, -2, -3, -3, -1, -3, -1, -1, -3, -4, -3, -3, -3, -1, -1, -1, -2, -2, -3, -3, -2,
-2, -3,  6,  2, -3, -1, -1, -3, -1, -4, -3,  1, -4, -1,  0, -2,  0, -1, -3, -4, -3,  4,  1, -1,
-1, -4,  2,  5, -3, -2,  0, -3,  1, -3, -2,  0, -4, -1,  2,  0,  0, -1, -2, -3, -2,  1,  4, -1,
-2, -2, -3, -3,  6, -3, -1,  0, -3,  0,  0, -3, -4, -4, -3, -3, -2, -2, -1,  1,  3, -3, -3, -1,
 0, -3, -1, -2, -3,  6, -2, -4, -2, -4, -3,  0, -4, -2, -2, -2,  0, -2, -3, -2, -3, -1, -2, -1,
-2, -3, -1,  0, -1, -2,  8, -3, -1, -3, -2,  1, -4, -2,  0,  0, -1, -2, -3, -2,  2,  0,  0, -1,
-1, -1, -3, -3,  0, -4, -3,  4, -3,  2,  1, -3, -4, -3, -3, -3, -2, -1,  3, -3, -1, -3, -3, -1,
-1, -3, -1,  1, -3, -2, -1, -3,  5, -2, -1,  0, -4, -1,  1,  2,  0, -1, -2, -3, -2,  0,  1, -1,
-1, -1, -4, -3,  0, -4, -3,  2, -2,  4,  2, -3, -4, -3, -2, -2, -2, -1,  1, -2, -1, -4, -3, -1,
-1, -1, -3, -2,  0, -3, -2,  1, -1,  2,  5, -2, -4, -2,  0, -1, -1, -1,  1, -1, -1, -3, -1, -1,
-2, -3,  1,  0, -3,  0,  1, -3,  0, -3, -2,  6, -4, -2,  0,  0,  1,  0, -3, -4, -2,  3,  0, -1,
-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
-1, -3, -1, -1, -4, -2, -2, -3, -1, -3, -2, -2, -4,  7, -1, -2, -1, -1, -2, -4, -3, -2, -1, -2,
-1, -3,  0,  2, -3, -2,  0, -3,  1, -2,  0,  0, -4, -1,  5,  1,  0, -1, -2, -2, -1,  0,  3, -1,
-1, -3, -2,  0, -3, -2,  0, -3,  2, -2, -1,  0, -4, -2,  1,  5, -1, -1, -3, -3, -2, -1,  0, -1,
 1, -1,  0,  0, -2,  0, -1, -2,  0, -2, -1,  1, -4, -1,  0, -1,  4,  1, -2, -3, -2,  0,  0,  0,
 0, -1, -1, -1, -2, -2, -2, -1, -1, -1, -1,  0, -4, -1, -1, -1,  1,  5,  0, -2, -2, -1, -1,  0,
 0, -1, -3, -2, -1, -3, -3,  3, -2,  1,  1, -3, -4, -2, -2, -3, -2,  0,  4, -3, -1, -3, -2, -1,
-3, -2, -4, -3,  1, -2, -2, -3, -3, -2, -1, -4, -4, -4, -2, -3, -3, -2, -3, 11,  2, -4, -3, -2,
-2, -2, -3, -2,  3, -3,  2, -1, -2, -1, -1, -2, -4, -3, -1, -2, -2, -2, -1,  2,  7, -3, -2, -1,
-2, -3,  4,  1, -3, -1,  0, -3,  0, -4, -3,  3, -4, -2,  0, -1,  0, -1, -3, -4, -3,  4,  1, -1,
-1, -3,  1,  4, -3, -2,  0, -3,  1, -3, -1,  0, -4, -1,  3,  0,  0, -1, -2, -3, -2,  1,  4, -1,
 0, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -4, -2, -1, -1,  0,  0, -1, -2, -1, -1, -1, -1,
};

// A scoring scheme class that will be used in conjunction with the \ref GotohAligner;
//
template <typename matrix_iterator>
struct BlosumScheme
{
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE BlosumScheme(const matrix_iterator m, const int32 gap_open, const int32 gap_ext) :
        m_matrix(m), m_gap_open(gap_open), m_gap_ext(gap_ext) {}

    // return the maximum match bonus at the given quality score
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 match(const uint8 q = 0) const { return 11; };

    // return the substitution score at the given (reference,query) position (r_i,q_j),
    // with values (r,q) and quality score qq
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 substitution(
        const uint32 r_i, const uint32 q_j,
        const uint8  r,  const uint8   q,
        const uint8  qq = 0) const { return int8( m_matrix[ r + q*24 ] ); };

    // return gap open and extension penalties for the pattern (query) and the text (reference)
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_open()      const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_extension() const { return m_gap_ext; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_open()         const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_extension()    const { return m_gap_ext; };

    const matrix_iterator   m_matrix;
    const int32             m_gap_open;
    const int32             m_gap_ext;
};

// main test entry point
//
int main(int argc, char* argv[])
{
    log_info(stderr, "protein SW... started\n");
    const uint32 n_tests   = 10;
    const uint32 n_strings = 50000;
    const uint32 P         = 128;
    const uint32 T         = 1024;

    // alloc a device vector for holding the Blosum-62 scoring matrix
    nvbio::vector<device_tag,uint8> d_blosum62( 24*24 );

    // copy the matrix to the device
    thrust::copy( s_blosum62, s_blosum62 + 24*24, d_blosum62.begin() );

    // alloc the storage for the host strings
    nvbio::vector<host_tag,uint8>  h_pattern_strings( P * n_strings );
    nvbio::vector<host_tag,uint8>  h_text_strings( T * n_strings );

    // fill the strings with random characters
    LCG_random rand;
    for (uint32 i = 0; i < P * n_strings; ++i)
        h_pattern_strings[i] = nvbio::min( uint8( rand.next() * 24.0f ), (uint8)23u );
    for (uint32 i = 0; i < T * n_strings; ++i)
        h_text_strings[i] = nvbio::min( uint8( rand.next() * 24.0f ), (uint8)23u );

    // copy to the device
    nvbio::vector<device_tag,uint8>  d_pattern_strings( h_pattern_strings );
    nvbio::vector<device_tag,uint8>  d_text_strings( h_text_strings );
    nvbio::vector<device_tag,uint32> d_pattern_offsets( n_strings + 1 );
    nvbio::vector<device_tag,uint32> d_text_offsets( n_strings + 1 );

    // build the string offsets
    thrust::sequence( d_pattern_offsets.begin(), d_pattern_offsets.end(), 0u, P );
    thrust::sequence( d_text_offsets.begin(),    d_text_offsets.end(),    0u, T );

    // prepare a vector of alignment sinks
    nvbio::vector< device_tag, aln::BestSink< uint32 > > sinks( n_strings );

    {
        const aln::SimpleGotohScheme scoring( 1, -1, -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::PatternBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadScheduler(),
                P, T );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Constant/P): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }
    {
        const aln::SimpleGotohScheme scoring( 1, -1, -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadScheduler(),
                P, T );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Constant/T): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }
    {
        const BlosumScheme< cuda::ldg_pointer<uint8> > scoring( cuda::make_ldg_pointer( raw_pointer( d_blosum62 ) ), -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::PatternBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadBlockScheduler<128,1>(),
                P, T );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Blosum62/P): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }
    {
        const BlosumScheme< cuda::ldg_pointer<uint8> > scoring( cuda::make_ldg_pointer( raw_pointer( d_blosum62 ) ), -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadBlockScheduler<128,1>(),
                P, T );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Blosum62/T): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }

    log_info(stderr, "protein SW... done\n");
    return 0;
}
