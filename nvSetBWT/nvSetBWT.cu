#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// nvSetBWT.cu
//

#define NVBIO_CUDA_DEBUG

#include <hipcub/hipcub.hpp>
#include <omp.h>

#include <nvbio/sufsort/sufsort.h>
#include <nvbio/sufsort/sufsort_utils.h>
#include <nvbio/sufsort/file_bwt.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/string_set.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/io/reads/reads.h>
#include <nvbio/fmindex/dna.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

using namespace nvbio;

/// Our in-memory reads container
///
struct Reads
{
    typedef uint32 word_type;

    static const uint32 WORD_SIZE        = 32;
    static const uint32 SYMBOL_SIZE      = 2;
    static const uint32 SYMBOLS_PER_WORD = 16;

    typedef PackedStream<word_type*,uint8,SYMBOL_SIZE,true,uint64>  packed_stream_type;

    Reads() : n_reads(0), n_symbols(0), min_len(uint32(-1)), max_len(0) {}

    uint32                      n_reads;            // number of reads
    uint64                      n_symbols;          // number of symbols
    uint32                      min_len;            // minimum read length
    uint32                      max_len;            // maximum read length
    thrust::host_vector<uint32> h_read_storage;     // read storage
    thrust::host_vector<uint64> h_read_index;       // read index
};

bool read(const char* reads_name, const io::QualityEncoding qencoding, const io::ReadEncoding flags, Reads* reads)
{
    typedef Reads::word_type word_type;
    const uint32 WORD_SIZE        = Reads::WORD_SIZE;
    const uint32 SYMBOL_SIZE      = Reads::SYMBOL_SIZE;
    const uint32 SYMBOLS_PER_WORD = Reads::SYMBOLS_PER_WORD;

    log_visible(stderr, "opening read file \"%s\"\n", reads_name);
    SharedPointer<nvbio::io::ReadDataStream> read_data_file(
        nvbio::io::open_read_file(reads_name,
        qencoding,
        uint32(-1),
        uint32(-1),
        flags )
    );

    if (read_data_file == NULL || read_data_file->is_ok() == false)
    {
        log_error(stderr, "    failed opening file \"%s\"\n", reads_name);
        return false;
    }

    const uint32 batch_size = 512*1024;

    float io_time = 0.0f;

    while (1)
    {
        nvbio::Timer timer;
        timer.start();

        SharedPointer<io::ReadData> h_read_data( read_data_file->next( batch_size ) );
        if (h_read_data == NULL)
            break;

        reads->h_read_storage.resize( 
            reads->h_read_storage.size() + 
            h_read_data->m_read_stream_words/2 ); // convert from 4-bits to 2-bits per symbol

        // pack the first few symbols to fill the last word
        const uint32 word_offset = reads->n_symbols & (SYMBOLS_PER_WORD-1);
              uint32 word_rem    = 0;

        typedef io::ReadData::const_read_stream_type src_read_stream_type;
        const src_read_stream_type src( h_read_data->read_stream() );

        if (word_offset)
        {
            const uint64 word_idx = reads->n_symbols / SYMBOLS_PER_WORD;

            // compute how many symbols we still need to encode to fill the current word
            word_rem = SYMBOLS_PER_WORD - word_offset;

            // fetch the word in question
            word_type word = reads->h_read_storage[ word_idx ];

            for (uint32 i = 0; i < word_rem; ++i)
            {
                const uint32       bit_idx = (word_offset + i) * SYMBOL_SIZE;
                const uint32 symbol_offset = (WORD_SIZE - SYMBOL_SIZE - bit_idx);
                const word_type     symbol = word_type(src[i]) << symbol_offset;

                // set bits
                word |= symbol;
            }

            // write out the word
            reads->h_read_storage[ word_idx ] = word;
        }

        #pragma omp parallel for
        for (int i = word_rem; i < int( h_read_data->m_read_stream_len ); i += SYMBOLS_PER_WORD)
        {
            // encode a word's worth of characters
            word_type word = 0u;

            const uint32 n_symbols = nvbio::min( SYMBOLS_PER_WORD, h_read_data->m_read_stream_len - i );

            for (uint32 j = 0; j < n_symbols; ++j)
            {
                const uint32       bit_idx = j * SYMBOL_SIZE;
                const uint32 symbol_offset = (WORD_SIZE - SYMBOL_SIZE - bit_idx);
                const word_type     symbol = word_type(src[i + j]) << symbol_offset;

                // set bits
                word |= symbol;
            }

            // write out the given word
            const uint64 word_idx = (reads->n_symbols + i) / SYMBOLS_PER_WORD;

            reads->h_read_storage[ word_idx ] = word;
        }

        // update the read index
        const uint32* src_index = h_read_data->read_index();
        reads->h_read_index.resize( reads->n_reads + h_read_data->size() + 1u );

        for (uint32 i = 0; i < h_read_data->size(); ++i)
            reads->h_read_index[ reads->n_reads + i ] = reads->n_symbols + src_index[i];

        // advance the destination pointer
        reads->n_symbols += h_read_data->m_read_stream_len;
        reads->n_reads   += h_read_data->size();
        reads->min_len = nvbio::min( reads->min_len, h_read_data->min_read_len() );
        reads->max_len = nvbio::max( reads->max_len, h_read_data->max_read_len() );

        timer.stop();
        io_time += timer.seconds();

        log_verbose(stderr,"\r    %u reads, %llu symbols read (%.1fs)    ", reads->n_reads, reads->n_symbols, io_time);
    }
    log_verbose_cont(stderr,"\n");
    return true;
}

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        log_visible(stderr, "nvSetBWT - Copyright 2013-2014, NVIDIA Corporation\n");
        log_info(stderr, "usage:\n");
        log_info(stderr, "  nvSetBWT [options] input_file output_file\n");
        log_info(stderr, "  options:\n");
        log_info(stderr, "   -v       | --verbosity     int (0-6) [5]\n");
        log_info(stderr, "   -cpu-mem | --cpu-memory    int (MB)  [8192]\n");
        log_info(stderr, "   -gpu-mem | --gpu-memory    int (MB)  [4096]\n");
        log_info(stderr, "   -c       | --compression   string    [1R]   (e.g. \"1\", ..., \"9\", \"1R\")\n");
        log_info(stderr, "   -F       | --skip-forward\n");
        log_info(stderr, "   -R       | --skip-reverse\n");
        log_info(stderr, "  output formats:\n");
        log_info(stderr, "    .txt      ASCII\n");
        log_info(stderr, "    .txt.gz   ASCII, gzip compressed\n");
        log_info(stderr, "    .txt.bgz  ASCII, block-gzip compressed\n");
        log_info(stderr, "    .bwt      2-bit packed binary\n");
        log_info(stderr, "    .bwt.gz   2-bit packed binary, gzip compressed\n");
        log_info(stderr, "    .bwt.bgz  2-bit packed binary, block-gzip compressed\n");
        log_info(stderr, "    .bwt4     4-bit packed binary\n");
        log_info(stderr, "    .bwt4.gz  4-bit packed binary, gzip compressed\n");
        log_info(stderr, "    .bwt4.bgz 4-bit packed binary, block-gzip compressed\n");
        return 0;
    }

    typedef Reads::word_type word_type;
    NVBIO_VAR_UNUSED static const uint32 SYMBOL_SIZE = Reads::SYMBOL_SIZE;

    const char* reads_name        = argv[argc-2];
    const char* output_name       = argv[argc-1];
    bool  forward                 = true;
    bool  reverse                 = true;
    const char* comp_level        = "1R";
    io::QualityEncoding qencoding = io::Phred33;

    BWTParams params;

    for (int i = 0; i < argc - 2; ++i)
    {
        if ((strcmp( argv[i], "-cpu-mem" )            == 0) ||
            (strcmp( argv[i], "--cpu-memory" )        == 0))
        {
            params.host_memory = atoi( argv[++i] ) * uint64(1024u*1024u);
        }
        else if ((strcmp( argv[i], "-gpu-mem" )       == 0) ||
                 (strcmp( argv[i], "-gpu-memory" )    == 0))
        {
            params.device_memory = atoi( argv[++i] ) * uint64(1024u*1024u);
        }
        else if ((strcmp( argv[i], "-v" )             == 0) ||
                 (strcmp( argv[i], "-verbosity" )     == 0) ||
                 (strcmp( argv[i], "--verbosity" )    == 0))
        {
            set_verbosity( Verbosity( atoi( argv[++i] ) ) );
        }
        else if ((strcmp( argv[i], "-F" )             == 0) ||
                 (strcmp( argv[i], "--skip-forward" ) == 0))  // skip forward strand
        {
            forward = false;
        }
        else if ((strcmp( argv[i], "-R" )             == 0) ||
                 (strcmp( argv[i], "--skip-reverse" ) == 0))  // skip reverse strand
        {
            reverse = false;
        }
        else if ((strcmp( argv[i], "-c" )             == 0) ||
                 (strcmp( argv[i], "--compression" )  == 0))  // setup compression level
        {
            comp_level = argv[++i];
        }
    }

    try
    {
        log_visible(stderr,"nvSetBWT... started\n");

        // build an output file
        SharedPointer<BaseBWTHandler> output_handler = SharedPointer<BaseBWTHandler>( open_bwt_file( output_name, comp_level ) );
        if (output_handler == NULL)
        {
            log_error(stderr, "  failed to create an output handler\n");
            return 1;
        }

        // gather device memory stats
        size_t free_device, total_device;
        hipMemGetInfo(&free_device, &total_device);
        log_stats(stderr, "  device has %ld of %ld MB free\n", free_device/1024/1024, total_device/1024/1024);

        // now set the number of CPU threads
        omp_set_num_threads( omp_get_num_procs() );
        #pragma omp parallel
        {
            log_verbose(stderr, "  running on multiple threads (%d)\n", omp_get_thread_num());
        }

        Reads reads;

        log_info(stderr,"  reading input... started\n");

        // NOTE: at the moment the forward and reverse strands are not interleaved: we place
        // first all the forward and then all the reverse strands - might want to fix this.
        if (forward)
        {
            if (read( reads_name, qencoding, io::ReadEncoding(0), &reads ) == false)
                return 1;
        }
        if (reverse)
        {
            if (read( reads_name, qencoding, io::ReadEncoding(io::REVERSE | io::COMPLEMENT), &reads ) == false)
                return 1;
        }

        // push sentinel value
        reads.h_read_index[ reads.n_reads ] = reads.n_symbols;

        log_info(stderr,"  reading input... done\n");

        const uint64 input_size = reads.h_read_storage.size() * sizeof(word_type);
        log_stats(stderr,"    reads   : %u (min len: %u, max len: %u)\n", reads.n_reads, reads.min_len, reads.max_len);
        log_stats(stderr,"    symbols : %llu\n", reads.n_symbols);
        log_stats(stderr,"    size    : %llu MB\n", input_size / uint64(1024*1024));

        typedef Reads::packed_stream_type                               packed_stream_type;
        typedef packed_stream_type::iterator                            packed_stream_iterator;
        typedef ConcatenatedStringSet<packed_stream_iterator,uint64*>   string_set;

        // start the real work
        log_info(stderr, "  bwt... started\n");

        nvbio::Timer timer;
        timer.start();

        const uint64 guard_band = 2u * uint64(1024*1024*1024);

        if (input_size + guard_band < free_device)
        {
            log_verbose(stderr, "  using fast path\n");

            thrust::device_vector<word_type> d_read_storage( reads.h_read_storage );
            thrust::device_vector<uint64>    d_read_index( reads.h_read_index );

            const packed_stream_type d_packed_string( (word_type*)nvbio::plain_view( d_read_storage ) );

            const string_set d_string_set(
                reads.n_reads,
                d_packed_string.begin(),
                nvbio::plain_view( d_read_index ) );

            cuda::bwt<SYMBOL_SIZE,true>(
                d_string_set,
                *output_handler,
                &params );
        }
        else
        {
            log_verbose(stderr, "  using hybrid path\n");

            const packed_stream_type h_packed_string( (word_type*)nvbio::plain_view( reads.h_read_storage ) );

            const string_set h_string_set(
                reads.n_reads,
                h_packed_string.begin(),
                nvbio::plain_view( reads.h_read_index ) );

            large_bwt<SYMBOL_SIZE,true>(
                h_string_set,
                *output_handler,
                &params );
        }

        timer.stop();

        //if (output_handler->n_dollars != reads.n_reads)
        //    log_warning(stderr, "    expected %u dollars, found %u\n", reads.n_reads, output_handler->n_dollars );

        log_info(stderr, "  bwt... done: %.2fs\n", timer.seconds());

        log_visible(stderr,"nvSetBWT... done\n");
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        return 1;
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
        return 1;
    }
    return 0;
}
