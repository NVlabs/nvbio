#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//#define NVBIO_ENABLE_PROFILING

#define MOD_NAMESPACE
#define MOD_NAMESPACE_BEGIN namespace bowtie2 { namespace driver {
#define MOD_NAMESPACE_END   }}
#define MOD_NAMESPACE_NAME bowtie2::driver

#include <nvBowtie/bowtie2/cuda/compute_thread.h>
#include <nvBowtie/bowtie2/cuda/defs.h>
#include <nvBowtie/bowtie2/cuda/fmindex_def.h>
#include <nvBowtie/bowtie2/cuda/params.h>
#include <nvBowtie/bowtie2/cuda/stats.h>
#include <nvBowtie/bowtie2/cuda/persist.h>
#include <nvBowtie/bowtie2/cuda/scoring.h>
#include <nvBowtie/bowtie2/cuda/mapq.h>
#include <nvBowtie/bowtie2/cuda/aligner.h>
#include <nvBowtie/bowtie2/cuda/aligner_inst.h>
#include <nvBowtie/bowtie2/cuda/input_thread.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/options.h>
#include <nvbio/basic/threads.h>
#include <nvbio/basic/atomics.h>
#include <nvbio/basic/html.h>
#include <nvbio/basic/version.h>
#include <nvbio/fmindex/bwt.h>
#include <nvbio/fmindex/ssa.h>
#include <nvbio/fmindex/fmindex.h>
#include <nvbio/fmindex/fmindex_device.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <algorithm>
#include <numeric>
#include <functional>

namespace nvbio {
namespace bowtie2 {
namespace cuda {

ComputeThreadSE::ComputeThreadSE(
    const uint32                             _thread_id,
    const uint32                             _device_id,
    const io::SequenceData&                  _reference_data,
    const io::FMIndexData&                   _driver_data,
    const std::map<std::string,std::string>& _options,
    const Params&                            _params,
          Stats&                             _stats) :
    thread_id( _thread_id ),
    device_id( _device_id ),
    reference_data_host( _reference_data ),
    driver_data_host( _driver_data ),
    options( _options ),
    input_thread( NULL ),
    output_file( NULL ),
    params( _params ),
    stats( _stats )
{
    log_visible(stderr, "[%u] nvBowtie cuda driver created on device %u\n", thread_id, device_id);

    // initialize the selected device
    hipSetDevice( device_id );
    hipSetDeviceFlags( hipDeviceMapHost | hipDeviceLmemResizeToMax );

    aligner = SharedPointer<Aligner>( new Aligner() );

    Timer timer;
    timer.start();

    const bool need_reverse =
        (params.allow_sub == 0 && USE_REVERSE_INDEX) ||
        (params.allow_sub == 1 && params.subseed_len == 0 && params.mode == BestMappingApprox);

    reference_data_device.reset( new io::SequenceDataDevice( reference_data_host ) );

    driver_data_device.reset( new io::FMIndexDataDevice( driver_data_host,
                        io::FMIndexDataDevice::FORWARD |
        (need_reverse ? io::FMIndexDataDevice::REVERSE : 0u) |
                        io::FMIndexDataDevice::SA ) );

    timer.stop();

    log_stats(stderr, "[%u]   allocated device driver data (%.2f GB - %.1fs)\n", thread_id, float(driver_data_device->allocated()) / 1.0e9f, timer.seconds() );
}

// gauge the favourite batch size
//
uint32 ComputeThreadSE::gauge_batch_size()
{
    // switch to the selected device
    hipSetDevice( device_id );

    uint32 BATCH_SIZE;

    for (BATCH_SIZE = params.max_batch_size*1024; BATCH_SIZE >= 16*1024; BATCH_SIZE /= 2)
    {
        std::pair<uint64,uint64> mem_stats;

        // gauge how much memory we'd need
        if (aligner->init_alloc( BATCH_SIZE, params, kSingleEnd, false, &mem_stats ) == true)
        {
            log_stats(stderr, "[%u]   estimated allocation sizes: HOST %lu MB, DEVICE %lu MB)\n",
                thread_id,
                mem_stats.first / (1024*1024),
                mem_stats.second / (1024*1024) );
            break;
        }
    }

    return BATCH_SIZE;
}

void ComputeThreadSE::do_run()
{
    log_visible(stderr, "[%u] nvBowtie cuda driver... started\n", thread_id);

    // switch to the selected device
    hipSetDevice( device_id );

    // build an empty report
    FILE* html_output = (params.report != std::string("")) ? fopen( params.report.c_str(), "w" ) : NULL;
    if (html_output)
    {
        // encapsulate the document
        {
            html::html_object html( html_output );
            {
                const char* meta_list = "<meta http-equiv=\"refresh\" content=\"1\" />";

                { html::header_object hd( html_output, "Bowtie2 Report", html::style(), meta_list ); }
                { html::body_object body( html_output ); }
            }
        }
        fclose( html_output );
    }

    Timer timer;

    io::SequenceDataDevice& reference_data = *reference_data_device.get();
    io::FMIndexDataDevice&  driver_data    = *driver_data_device.get();

    log_stats(stderr, "[%u]   allocated device driver data (%.2f GB - %.1fs)\n", thread_id, float(driver_data.allocated()) / 1.0e9f, timer.seconds() );

    typedef FMIndexDef::type fm_index_type;

    fm_index_type fmi  = driver_data.index();
    fm_index_type rfmi = driver_data.rindex();

    size_t free, total;
    hipMemGetInfo(&free, &total);
    log_stats(stderr, "[%u]   device has %ld of %ld MB free\n", thread_id, free/1024/1024, total/1024/1024);

    const uint32 BATCH_SIZE = input_thread->batch_size();

    log_stats(stderr, "[%u]   processing reads in batches of %uK\n", thread_id, BATCH_SIZE/1024);

    // setup the output file
    aligner->output_file = output_file;

    // initialize the aligner
    if (aligner->init( thread_id, BATCH_SIZE, params, kSingleEnd ) == false)
        return;

    nvbio::cuda::check_error("cuda initializations");

    hipMemGetInfo(&free, &total);
    log_stats(stderr, "[%u]   ready to start processing: device has %ld MB free\n", thread_id, free/1024/1024);

    Timer global_timer;
    global_timer.start();

    UberScoringScheme& scoring_scheme = params.scoring_scheme;

    uint32 n_reads = 0;

    io::SequenceDataHost   local_read_data_host;
    io::HostOutputBatchSE  local_output_batch_host;

    // loop through the batches of reads
    while (1)
    {
        uint32 read_begin;

        Timer io_timer;
        io_timer.start();

        io::SequenceDataHost* read_data_host = input_thread->next( &read_begin );

        io_timer.stop();
        stats.read_io.add( read_data_host ? read_data_host->size() : 0u, io_timer.seconds() );

        if (read_data_host == NULL)
        {
            log_verbose(stderr, "[%u] end of input reached\n", thread_id);
            break;
        }

        if (read_data_host->max_sequence_len() > Aligner::MAX_READ_LEN)
        {
            log_error(stderr, "[%u] unsupported read length %u (maximum is %u)\n", thread_id,
                read_data_host->max_sequence_len(),
                Aligner::MAX_READ_LEN );
            break;
        }

        // make a local copy of the host batch
        local_read_data_host = *read_data_host;

        // mark this set as ready to be reused
        input_thread->release( read_data_host );

        Timer timer;
        timer.start();

        //aligner.output_file->start_batch( &local_read_data_host );
        local_output_batch_host.read_data = &local_read_data_host;

        io::SequenceDataDevice read_data( local_read_data_host );
        hipDeviceSynchronize();

        timer.stop();
        stats.read_HtoD.add( read_data.size(), timer.seconds() );

        const uint32 count = read_data.size();
        log_info(stderr, "[%u] aligning reads [%u, %u]\n", thread_id, read_begin, read_begin + count - 1u);
        log_verbose(stderr, "[%u]   %u reads\n", thread_id, count);
        log_verbose(stderr, "[%u]   %.3f M bps (%.1f MB)\n", thread_id, float(read_data.bps())/1.0e6f, float(read_data.words()*sizeof(uint32)+read_data.bps()*sizeof(char))/float(1024*1024));
        log_verbose(stderr, "[%u]   %.1f bps/read (min: %u, max: %u)\n", thread_id, float(read_data.bps())/float(read_data.size()), read_data.min_sequence_len(), read_data.max_sequence_len());

        if (params.mode == AllMapping)
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                all_ed(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    local_output_batch_host,
                    stats );
            }
            else
            {
                all_sw(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    local_output_batch_host,
                    stats );
            }
        }
        else
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                best_approx_ed(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    local_output_batch_host,
                    stats );
            }
            else
            {
                best_approx_sw(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    local_output_batch_host,
                    stats );
            }
        }

        global_timer.stop();
        stats.global_time += global_timer.seconds();
        global_timer.start();

        //aligner->output_file->end_batch();

        // increase the total reads counter
        n_reads += count;

        log_verbose(stderr, "[%u]   %.1f K reads/s\n", thread_id, 1.0e-3f * float(n_reads) / stats.global_time);
    }

    global_timer.stop();
    stats.global_time += global_timer.seconds();
    stats.n_reads = n_reads;

    if (params.report.length())
        nvbio::bowtie2::cuda::generate_device_report( thread_id, stats, stats.mate1, params.report.c_str() );

    log_visible(stderr, "[%u] nvBowtie cuda driver... done\n", thread_id);

    log_stats(stderr, "[%u]   total        : %.2f sec (avg: %.1fK reads/s).\n", thread_id, stats.global_time, 1.0e-3f * float(n_reads)/stats.global_time);
    log_stats(stderr, "[%u]   mapping      : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.map.time, 1.0e-6f * stats.map.avg_speed(), 1.0e-6f * stats.map.max_speed, stats.map.device_time);
    log_stats(stderr, "[%u]   selecting    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.select.time, 1.0e-6f * stats.select.avg_speed(), 1.0e-6f * stats.select.max_speed, stats.select.device_time);
    log_stats(stderr, "[%u]   sorting      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.sort.time, 1.0e-6f * stats.sort.avg_speed(), 1.0e-6f * stats.sort.max_speed, stats.sort.device_time);
    log_stats(stderr, "[%u]   scoring      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.score.time, 1.0e-6f * stats.score.avg_speed(), 1.0e-6f * stats.score.max_speed, stats.score.device_time);
    log_stats(stderr, "[%u]   locating     : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.locate.time, 1.0e-6f * stats.locate.avg_speed(), 1.0e-6f * stats.locate.max_speed, stats.locate.device_time);
    log_stats(stderr, "[%u]   backtracking : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.backtrack.time, 1.0e-6f * stats.backtrack.avg_speed(), 1.0e-6f * stats.backtrack.max_speed, stats.backtrack.device_time);
    log_stats(stderr, "[%u]   finalizing   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.finalize.time, 1.0e-6f * stats.finalize.avg_speed(), 1.0e-6f * stats.finalize.max_speed, stats.finalize.device_time);
    log_stats(stderr, "[%u]   results DtoH : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.alignments_DtoH.time, 1.0e-6f * stats.alignments_DtoH.avg_speed(), 1.0e-6f * stats.alignments_DtoH.max_speed);
    log_stats(stderr, "[%u]   results I/O  : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.io.time, 1.0e-6f * stats.io.avg_speed(), 1.0e-6f * stats.io.max_speed);
    log_stats(stderr, "[%u]   reads HtoD   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.read_HtoD.time, 1.0e-6f * stats.read_HtoD.avg_speed(), 1.0e-6f * stats.read_HtoD.max_speed);
    log_stats(stderr, "[%u]   reads I/O    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.read_io.time, 1.0e-6f * stats.read_io.avg_speed(), 1.0e-6f * stats.read_io.max_speed);
}


void ComputeThreadSE::run()
{
    try {
        do_run();
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
    }
}

ComputeThreadPE::ComputeThreadPE(
    const uint32                             _thread_id,
    const uint32                             _device_id,
    const io::SequenceData&                  _reference_data,
    const io::FMIndexData&                   _driver_data,
    const std::map<std::string,std::string>& _options,
    const Params&                            _params,
          Stats&                             _stats) :
    thread_id( _thread_id ),
    device_id( _device_id ),
    reference_data_host( _reference_data ),
    driver_data_host( _driver_data ),
    options( _options ),
    input_thread( NULL ),
    output_file( NULL ),
    params( _params ),
    stats( _stats )
{
    log_visible(stderr, "[%u] nvBowtie cuda driver created on device %u\n", thread_id, device_id);

    // initialize the selected device
    hipSetDevice( device_id );
    hipSetDeviceFlags( hipDeviceMapHost | hipDeviceLmemResizeToMax );

    aligner = SharedPointer<Aligner>( new Aligner() );

    Timer timer;
    timer.start();

    const bool need_reverse =
        (params.allow_sub == 0 && USE_REVERSE_INDEX) ||
        (params.allow_sub == 1 && params.subseed_len == 0 && params.mode == BestMappingApprox);

    reference_data_device.reset( new io::SequenceDataDevice( reference_data_host ) );

    driver_data_device.reset( new io::FMIndexDataDevice( driver_data_host,
                        io::FMIndexDataDevice::FORWARD |
        (need_reverse ? io::FMIndexDataDevice::REVERSE : 0u) |
                        io::FMIndexDataDevice::SA ) );

    timer.stop();

    log_stats(stderr, "[%u]   allocated device driver data (%.2f GB - %.1fs)\n", thread_id, float(driver_data_device->allocated()) / 1.0e9f, timer.seconds() );
}

// gauge the favourite batch size
//
uint32 ComputeThreadPE::gauge_batch_size()
{
    // switch to the selected device
    hipSetDevice( device_id );

    uint32 BATCH_SIZE;

    for (BATCH_SIZE = params.max_batch_size*1024; BATCH_SIZE >= 16*1024; BATCH_SIZE /= 2)
    {
        std::pair<uint64,uint64> mem_stats;

        // gauge how much memory we'd need
        if (aligner->init_alloc( BATCH_SIZE, params, kPairedEnds, false, &mem_stats ) == true)
        {
            log_stats(stderr, "[%u]   estimated allocation sizes: HOST %lu MB, DEVICE %lu MB)\n",
                thread_id,
                mem_stats.first / (1024*1024),
                mem_stats.second / (1024*1024) );
            break;
        }
    }

    return BATCH_SIZE;
}

void ComputeThreadPE::do_run()
{
    log_visible(stderr, "[%u] nvBowtie cuda driver... started\n", thread_id);

    // switch to the selected device
    hipSetDevice( device_id );

    // build an empty report
    FILE* html_output = (params.report != std::string("")) ? fopen( params.report.c_str(), "w" ) : NULL;
    if (html_output)
    {
        // encapsulate the document
        {
            html::html_object html( html_output );
            {
                const char* meta_list = "<meta http-equiv=\"refresh\" content=\"1\" />";

                { html::header_object hd( html_output, "Bowtie2 Report", html::style(), meta_list ); }
                { html::body_object body( html_output ); }
            }
        }
        fclose( html_output );
    }

    Timer timer;

    io::SequenceDataDevice& reference_data = *reference_data_device.get();
    io::FMIndexDataDevice&  driver_data    = *driver_data_device.get();

    typedef FMIndexDef::type fm_index_type;

    fm_index_type fmi  = driver_data.index();
    fm_index_type rfmi = driver_data.rindex();

    size_t free, total;
    hipMemGetInfo(&free, &total);
    log_stats(stderr, "[%u]   device has %ld of %ld MB free\n", thread_id, free/1024/1024, total/1024/1024);

    const uint32 BATCH_SIZE = input_thread->batch_size();

    log_stats(stderr, "[%u]   processing reads in batches of %uK\n", thread_id, BATCH_SIZE/1024);

    // setup the output file
    aligner->output_file = output_file;

    // initialize the aligner
    if (aligner->init( thread_id, BATCH_SIZE, params, kPairedEnds ) == false)
        return;

    nvbio::cuda::check_error("cuda initializations");

    hipMemGetInfo(&free, &total);
    log_stats(stderr, "[%u]   ready to start processing: device has %ld MB free\n", thread_id, free/1024/1024);

    size_t stack_size_limit;
    hipDeviceGetLimit( &stack_size_limit, hipLimitStackSize );
    log_debug(stderr, "[%u]   max cuda stack size: %u\n", thread_id, stack_size_limit);

    Timer global_timer;
    global_timer.start();

    UberScoringScheme& scoring_scheme = params.scoring_scheme;

    uint32 n_reads = 0;

    io::SequenceDataHost    local_read_data_host1;
    io::SequenceDataHost    local_read_data_host2;
    io::HostOutputBatchPE   local_output_batch_host;

    // loop through the batches of reads
    while (1)
    {
        uint32 read_begin;

        Timer io_timer;
        io_timer.start();

        std::pair<io::SequenceDataHost*,io::SequenceDataHost*> read_data_host_pair = input_thread->next( &read_begin );

        io::SequenceDataHost* read_data_host1 = read_data_host_pair.first;
        io::SequenceDataHost* read_data_host2 = read_data_host_pair.second;

        io_timer.stop();
        stats.read_io.add( read_data_host1 ? read_data_host1->size() : 0u, io_timer.seconds() );

        if (read_data_host1 == NULL ||
            read_data_host2 == NULL)
        {
            log_verbose(stderr, "[%u] end of input reached\n", thread_id);
            break;
        }

        if ((read_data_host1->max_sequence_len() > Aligner::MAX_READ_LEN) ||
            (read_data_host2->max_sequence_len() > Aligner::MAX_READ_LEN))
        {
            log_error(stderr, "[%u] unsupported read length %u (maximum is %u)\n",
                thread_id,
                nvbio::max(read_data_host1->max_sequence_len(), read_data_host2->max_sequence_len()),
                Aligner::MAX_READ_LEN );
            break;
        }

        // make a local copy of the host batch
        local_read_data_host1 = *read_data_host1;
        local_read_data_host2 = *read_data_host2;

        // mark this set as ready to be reused
        input_thread->release( read_data_host_pair );

        Timer timer;
        timer.start();

        //aligner.output_file->start_batch( &local_read_data_host1, &local_read_data_host2 );
        local_output_batch_host.read_data[0] = &local_read_data_host1;
        local_output_batch_host.read_data[1] = &local_read_data_host2;

        io::SequenceDataDevice read_data1( local_read_data_host1/*, io::ReadDataDevice::READS | io::ReadDataDevice::QUALS*/ );
        io::SequenceDataDevice read_data2( local_read_data_host2/*, io::ReadDataDevice::READS | io::ReadDataDevice::QUALS*/ );

        timer.stop();
        stats.read_HtoD.add( read_data1.size(), timer.seconds() );

        const uint32 count = read_data1.size();
        log_info(stderr, "[%u] aligning reads [%u, %u]\n", thread_id, read_begin, read_begin + count - 1u);
        log_verbose(stderr, "[%u]   %u reads\n", thread_id, count);
        log_verbose(stderr, "[%u]   %.3f M bps (%.1f MB)\n", thread_id,
            float(read_data1.bps() + read_data2.bps())/1.0e6f,
            float(read_data1.words()*sizeof(uint32)+read_data1.bps()*sizeof(char))/float(1024*1024)+
            float(read_data2.words()*sizeof(uint32)+read_data2.bps()*sizeof(char))/float(1024*1024));
        log_verbose(stderr, "[%u]   %.1f bps/read (min: %u, max: %u)\n", thread_id,
            float(read_data1.bps()+read_data2.bps())/float(read_data1.size()+read_data2.size()),
            nvbio::min( read_data1.min_sequence_len(), read_data2.min_sequence_len() ),
            nvbio::max( read_data1.max_sequence_len(), read_data2.max_sequence_len() ));

        if (params.mode == AllMapping)
        {
            log_error(stderr, "[%u] paired-end all-mapping is not yet supported!\n", thread_id);
            exit(1);
        }
        else
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                best_approx_ed(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data1,
                    read_data2,
                    local_output_batch_host,
                    stats );
            }
            else
            {
                best_approx_sw(
                    *aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data1,
                    read_data2,
                    local_output_batch_host,
                    stats );
            }
        }

        global_timer.stop();
        stats.global_time += global_timer.seconds();
        global_timer.start();

        //aligner.output_file->end_batch();

        // increase the total reads counter
        n_reads += count;

        log_verbose(stderr, "[%u]   %.1f K reads/s\n", thread_id, 1.0e-3f * float(n_reads) / stats.global_time);
    }

    global_timer.stop();
    stats.global_time += global_timer.seconds();

    if (params.report.length())
        nvbio::bowtie2::cuda::generate_device_report( thread_id, stats, stats.concordant, params.report.c_str() );

    log_visible(stderr, "[%u] nvBowtie cuda driver... done\n", thread_id);

    log_stats(stderr, "[%u]   total          : %.2f sec (avg: %.1fK reads/s).\n", thread_id, stats.global_time, 1.0e-3f * float(n_reads)/stats.global_time);
    log_stats(stderr, "[%u]   mapping        : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.map.time, 1.0e-6f * stats.map.avg_speed(), 1.0e-6f * stats.map.max_speed, stats.map.device_time);
    log_stats(stderr, "[%u]   scoring        : %.2f sec (avg: %.1fM reads/s, max: %.3fM reads/s, %.2f device sec).).\n", thread_id, stats.scoring_pipe.time, 1.0e-6f * stats.scoring_pipe.avg_speed(), 1.0e-6f * stats.scoring_pipe.max_speed, stats.scoring_pipe.device_time);
    log_stats(stderr, "[%u]     selecting    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.select.time, 1.0e-6f * stats.select.avg_speed(), 1.0e-6f * stats.select.max_speed, stats.select.device_time);
    log_stats(stderr, "[%u]     sorting      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.sort.time, 1.0e-6f * stats.sort.avg_speed(), 1.0e-6f * stats.sort.max_speed, stats.sort.device_time);
    log_stats(stderr, "[%u]     scoring(a)   : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.score.time, 1.0e-6f * stats.score.avg_speed(), 1.0e-6f * stats.score.max_speed, stats.score.device_time);
    log_stats(stderr, "[%u]     scoring(o)   : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.opposite_score.time, 1.0e-6f * stats.opposite_score.avg_speed(), 1.0e-6f * stats.opposite_score.max_speed, stats.opposite_score.device_time);
    log_stats(stderr, "[%u]     locating     : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", thread_id, stats.locate.time, 1.0e-6f * stats.locate.avg_speed(), 1.0e-6f * stats.locate.max_speed, stats.locate.device_time);
    log_stats(stderr, "[%u]   backtracing(a) : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.backtrack.time, 1.0e-6f * stats.backtrack.avg_speed(), 1.0e-6f * stats.backtrack.max_speed, stats.backtrack.device_time);
    log_stats(stderr, "[%u]   backtracing(o) : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.backtrack_opposite.time, 1.0e-6f * stats.backtrack_opposite.avg_speed(), 1.0e-6f * stats.backtrack_opposite.max_speed, stats.backtrack_opposite.device_time);
    log_stats(stderr, "[%u]   finalizing     : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", thread_id, stats.finalize.time, 1.0e-6f * stats.finalize.avg_speed(), 1.0e-6f * stats.finalize.max_speed, stats.finalize.device_time);
    log_stats(stderr, "[%u]   results DtoH   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.alignments_DtoH.time, 1.0e-6f * stats.alignments_DtoH.avg_speed(), 1.0e-6f * stats.alignments_DtoH.max_speed);
    log_stats(stderr, "[%u]   results I/O    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.io.time, 1.0e-6f * stats.io.avg_speed(), 1.0e-6f * stats.io.max_speed);
    log_stats(stderr, "[%u]   reads HtoD     : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.read_HtoD.time, 1.0e-6f * stats.read_HtoD.avg_speed(), 1.0e-6f * stats.read_HtoD.max_speed);
    log_stats(stderr, "[%u]   reads I/O      : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", thread_id, stats.read_io.time, 1.0e-6f * stats.read_io.avg_speed(), 1.0e-6f * stats.read_io.max_speed);
}

void ComputeThreadPE::run()
{
    try {
        do_run();
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
    }
}

} // namespace cuda
} // namespace bowtie2
} // namespace nvbio
