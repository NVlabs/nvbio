#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "filter-chains.h"
#include "mem-search.h"
#include "options.h"
#include "pipeline.h"
#include "util.h"

#include <nvbio/basic/numbers.h>
#include <nvbio/basic/algorithms.h>
#include <nvbio/basic/priority_queue.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/transform_iterator.h>
#include <nvbio/basic/vector_view.h>
#include <nvbio/basic/primitives.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>

using namespace nvbio;

// compute the coverage for each chain in a set
__global__
void chain_coverage_kernel(
    const uint32                                    n_chains,           // the number of chains
    const uint32*                                   chain_reads,        // the chain reads
    const uint32*                                   chain_offsets,      // the chain offsets
    const uint32*                                   chain_lengths,      // the chain lengths
    const mem_state::mem_type*                      mems,               // the MEMs for this chunk of reads
    const uint32*                                   mems_index,         // a sorting index into the MEMs specifying their processing order
          uint2*                                    chain_ranges,       // the output chain ranges
          uint64*                                   chain_weights)      // the output chain weights
{
    const uint32 chain_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (chain_id >= n_chains)
        return;

    const uint32 read  = chain_reads[ chain_id ];
    const uint32 begin = chain_offsets[ chain_id ];
    const uint32 end   = chain_lengths[ chain_id ] + begin;

    uint2  range  = make_uint2( uint32(-1), 0u );
    uint32 weight = 0;

    // NOTE: we assume here the MEMs of a chain appear sorted by their left coordinate
    for (uint32 i = begin; i < end; ++i)
    {
        const mem_state::mem_type seed = mems[ mems_index[i] ];

        const uint2 span = seed.span();

        if (span.x >= range.y)
            weight += span.y - span.x;
        else if (span.y > range.y)
            weight += span.y - range.y;

        range.x = nvbio::min( range.x, seed.span().x );
        range.y = nvbio::max( range.y, seed.span().y );
    }

    // write out the outputs
    chain_ranges[ chain_id ]  = range;
    chain_weights[ chain_id ] = uint64( weight ) | (uint64( read ) << 32);
}

// filter the chains belonging to each read
__global__
void chain_filter_kernel(
    const read_chunk                                chunk,              // the current sub-batch
    const uint32                                    n_chains,           // the number of chains
    const uint32*                                   chain_reads,        // the chain reads
    const uint32*                                   chain_index,        // the chain order
    const uint2*                                    chain_ranges,       // the chain ranges
    const uint64*                                   chain_weights,      // the chain weights
    const float                                     mask_level,         // input option
    const float                                     chain_drop_ratio,   // input option
    const uint32                                    min_seed_len,       // input option
          uint8*                                    chain_flags)        // the output flags
{
    const uint32 read_id = threadIdx.x + blockIdx.x * blockDim.x + chunk.read_begin;
    if (read_id >= chunk.read_end)
        return;

    const uint32 begin = uint32( nvbio::lower_bound( read_id, chain_reads, n_chains ) - chain_reads );
    const uint32 end   = uint32( nvbio::upper_bound( read_id, chain_reads, n_chains ) - chain_reads );

    // skip pathological cases
    if (begin == end)
        return;

    // keep the first chain
    chain_flags[ chain_index[begin] ] = 1u; // mark to keep

    // and loop through all the rest to decide which ones to keep
    uint32 n = 1;

    for (uint32 i = begin + 1; i < end; ++i)
    {
        const uint2  i_span = chain_ranges[ chain_index[i] ];
        const uint32 i_w    = chain_weights[ i ] & 0xFFFFFFFFu;               // already sorted as chain_index

        uint32 j;
        for (j = begin; j < begin + n; ++j)
        {
            const uint2  j_span = chain_ranges[ chain_index[j] ];
            const uint32 j_w    = chain_weights[ j ] & 0xFFFFFFFFu;           // already sorted as chain_index

            const uint32 max_begin = nvbio::max( i_span.x, j_span.x );
            const uint32 min_end   = nvbio::min( i_span.y, j_span.y );

            if (min_end > max_begin) // have overlap
            {
                const uint32 min_l = nvbio::min( i_span.y - i_span.x, j_span.y - j_span.x );
				if (min_end - max_begin >= min_l * mask_level) // significant overlap
                {
                    chain_flags[ chain_index[i] ] = 1u; // mark to keep

                    if (i_w < j_w * chain_drop_ratio &&
                        j_w - i_w >= min_seed_len * 2)
                        break;
				}
            }
        }
		if (j == n) // no significant overlap with better chains, keep it.
        {
            chain_flags[ chain_index[i] ] = 1u; // mark to keep

            ++n;
        }
    }
}

// filter chains for the current pipeline::chunk of reads
void filter_chains(pipeline_state *pipeline, const io::SequenceDataDevice *reads)
{
    const ScopedTimer<float> timer( &pipeline->stats.chain_time ); // keep track of the time spent here

    struct chains_state<device_tag> *chn = &pipeline->chn;

    const uint32 n_reads = pipeline->chunk.read_end - pipeline->chunk.read_begin;
    const uint32 n_mems  = pipeline->chunk.mem_end  - pipeline->chunk.mem_begin;

    // skip pathological cases
    if (n_mems == 0u)
        return;

    // extract the list of unique chain ids together with their counts, i.e. the chain lengths
    nvbio::vector<device_tag,uint64> unique_chains( n_mems );
    nvbio::vector<device_tag,uint32> unique_counts( n_mems );
    nvbio::vector<device_tag,uint8>  temp_storage;

    const uint32 n_chains = runlength_encode(
        n_mems,
        chn->mems_chain.begin(),                        // the input chain ids, one per seed
        unique_chains.begin(),                          // the output "unique" chain ids
        unique_counts.begin(),                          // the output repetition counts, i.e. the chain lengths
        temp_storage );                                 // some temp storage

    // resize the chain vectors if needed
    uint32 reserved_space = uint32( chn->chain_lengths.size() );
    if (n_chains > reserved_space)
    {
        chn->chain_lengths.clear();  chn->chain_lengths.resize( n_chains );
        chn->chain_offsets.clear();  chn->chain_offsets.resize( n_chains );
        chn->chain_reads.clear();    chn->chain_reads.resize( n_chains );

        reserved_space = n_chains;
    }

    // copy their lengths
    thrust::copy(
        unique_counts.begin(),
        unique_counts.begin() + n_chains,
        chn->chain_lengths.begin() );

    // find the offset to the beginning of each chain
    thrust::lower_bound(
        chn->mems_chain.begin(),                    // the beginning of the sorted list of keys to search in
        chn->mems_chain.begin() + n_mems,           // the end of the sorted list of keys to search in
        unique_chains.begin(),                      // the beginning of the sequence of values to search
        unique_chains.begin() + n_chains,           // the end of the sequence of values to search
        chn->chain_offsets.begin() );               // the output sequence

    // extract the read-id frome the chain ids
    thrust::transform(
        unique_chains.begin(),                      // the beginning of the input sequence to transform
        unique_chains.begin() + n_chains,           // the end of the input sequence to transform
        chn->chain_reads.begin(),                   // the beginning othe output sequence
        nvbio::hi_bits_functor<uint32,uint64>() );  // the functor to apply, in this case a 32-bit left shift

    // debug check: make sure the chain offsets are sorted
    if (is_sorted<device_tag>( n_chains, chn->chain_offsets.begin() ) == false)
    {
        log_error(stderr, "filter_chains: chain offsets are not sorted!\n");
        exit(0);
    }

    // debug check: make sure the chains are sorted by read
    if (is_sorted<device_tag>( n_chains, chn->chain_reads.begin() ) == false)
    {
        log_error(stderr, "filter_chains: chains are not sorted by read!\n");
        exit(0);
    }

    nvbio::vector<device_tag,uint2>  chain_ranges( n_chains );
    nvbio::vector<device_tag,uint64> chain_weights( n_chains );
    nvbio::vector<device_tag,uint32> chain_index( reserved_space ); // potentially a little bigger because we'll reuse
                                                                    // it for the final filtering...

    optional_device_synchronize();
    cuda::check_error("chain-coverage-init");

    // compute chain coverages
    {
        const uint32 block_dim = 128;
        const uint32 n_blocks  = util::divide_ri( n_chains, block_dim );

        chain_coverage_kernel<<<n_blocks, block_dim>>>(
            n_chains,
            nvbio::plain_view( chn->chain_reads ),
            nvbio::plain_view( chn->chain_offsets ),
            nvbio::plain_view( chn->chain_lengths ),
            nvbio::plain_view( chn->mems ),
            nvbio::plain_view( chn->mems_index ),
            nvbio::plain_view( chain_ranges ),
            nvbio::plain_view( chain_weights ) );

        optional_device_synchronize();
        cuda::check_error("chain-coverage kernel");
    }

    // sort the chains by weight
    thrust::copy(
        thrust::make_counting_iterator<uint32>(0u),
        thrust::make_counting_iterator<uint32>(0u) + n_chains,
        chain_index.begin() );

    thrust::sort_by_key(                            // TODO: this is slow, switch to nvbio::cuda::SortEnactor
        chain_weights.begin(),
        chain_weights.begin() + n_chains,
        chain_index.begin() );

    nvbio::vector<device_tag,uint8> chain_flags( n_chains );
    thrust::fill( chain_flags.begin(), chain_flags.begin() + n_chains, 0u );

    // filter chains: set the flags for the chains to be kept
    {
        const uint32 block_dim = 128;
        const uint32 n_blocks  = util::divide_ri( n_reads, block_dim );

        chain_filter_kernel<<<n_blocks, block_dim>>>(
            pipeline->chunk,
            n_chains,
            nvbio::plain_view( chn->chain_reads ),
            nvbio::plain_view( chain_index ),
            nvbio::plain_view( chain_ranges ),
            nvbio::plain_view( chain_weights ),
            command_line_options.mask_level,
            command_line_options.chain_drop_ratio,
            command_line_options.min_seed_len,
            nvbio::plain_view( chain_flags ) );

        optional_device_synchronize();
        cuda::check_error("chain-filter kernel");
    }

    // filter chain_reads
    const uint32 n_filtered_chains = copy_flagged(
        n_chains,                                   // the number of input elements
        chn->chain_reads.begin(),                   // the input sequence of flagged elements to copy
        chain_flags.begin(),                        // the input sequence of flags
        chain_index.begin(),                        // the output sequence of copied elements
        temp_storage );                             // some temporary storage

    chn->chain_reads.swap( chain_index );

    // debug check: make sure the chains are sorted by read
    if (is_sorted<device_tag>( n_filtered_chains, chn->chain_reads.begin() ) == false)
    {
        log_error(stderr, "filter_chains: filtered chains are not sorted by read!\n");
        exit(0);
    }

    // filter chain_offsets
    cuda::copy_flagged(
        n_chains,                                   // the number of input elements
        chn->chain_offsets.begin(),                 // the input sequence of flagged elements to copy
        chain_flags.begin(),                        // the input sequence of flags
        chain_index.begin(),                        // the output sequence of copied elements
        temp_storage );                             // some temporary storage

    chn->chain_offsets.swap( chain_index );

    // filter chain_lengths
    cuda::copy_flagged(
        n_chains,                                   // the number of input elements
        chn->chain_lengths.begin(),                 // the input sequence of flagged elements to copy
        chain_flags.begin(),                        // the input sequence of flags
        chain_index.begin(),                        // the output sequence of copied elements
        temp_storage );                             // some temporary storage

    chn->chain_lengths.swap( chain_index );

    // assign the output number of chains
    chn->n_chains = n_filtered_chains;

    // keep stats
    pipeline->stats.n_chains += n_filtered_chains;
}
