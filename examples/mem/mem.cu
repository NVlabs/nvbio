#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// mem.cu
//

#include <stdio.h>
#include <stdlib.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/dna.h>
#include <nvbio/strings/string_set.h>
#include <nvbio/strings/infix.h>
#include <nvbio/strings/seeds.h>
#include <nvbio/fmindex/mem.h>
#include <nvbio/io/reads/reads.h>
#include <nvbio/io/fmi.h>

using namespace nvbio;

// main test entry point
//
int main(int argc, char* argv[])
{
    //
    // perform some basic option parsing
    //

    const uint32 batch_reads   =   1*1024*1024;
    const uint32 batch_bps     = 100*1024*1024;

    const char* reads = argv[argc-1];
    const char* index = argv[argc-2];

    uint32 max_reads        = uint32(-1);
    uint32 min_intv         = 1u;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-max-reads" ) == 0)
            max_reads = uint32( atoi( argv[++i] ) );
        else if (strcmp( argv[i], "-min-intv" ) == 0)
            min_intv = int16( atoi( argv[++i] ) );
    }

    const uint32 fm_flags = io::FMIndexData::GENOME  |
                            io::FMIndexData::FORWARD |
                            io::FMIndexData::REVERSE |
                            io::FMIndexData::SA;

    io::FMIndexData *h_fmi = NULL;
    io::FMIndexDataMMAP mmap_loader;
    io::FMIndexDataRAM file_loader;

    if (mmap_loader.load( index ))
    {
        h_fmi = &mmap_loader;
    } else {
        if (!file_loader.load( index, fm_flags ))
        {
            log_error(stderr, "    failed loading index \"%s\"\n", index);
            return 1u;
        }

        h_fmi = &file_loader;
    }

    // build its device version
    const io::FMIndexDataDevice d_fmi( *h_fmi, fm_flags );

    typedef io::FMIndexDataDevice::stream_type genome_type;

    // fetch the genome string
    const uint32      genome_len = d_fmi.genome_length();
    const genome_type d_genome( d_fmi.genome_stream() );

    // open a read file
    log_info(stderr, "  opening reads file... started\n");

    SharedPointer<io::ReadDataStream> read_data_file(
        io::open_read_file(
            reads,
            io::Phred33,
            2*max_reads,
            uint32(-1),
            io::ReadEncoding( io::FORWARD | io::REVERSE_COMPLEMENT ) ) );

    // check whether the file opened correctly
    if (read_data_file == NULL || read_data_file->is_ok() == false)
    {
        log_error(stderr, "    failed opening file \"%s\"\n", reads);
        return 1u;
    }
    log_info(stderr, "  opening reads file... done\n");

    typedef io::FMIndexDataDevice::fm_index_type        fm_index_type;
    typedef MEMFilterDevice<fm_index_type>              mem_filter_type;

    // fetch the FM-index
    const fm_index_type f_index = d_fmi.index();
    const fm_index_type r_index = d_fmi.rindex();

    // create a MEM filter
    mem_filter_type mem_filter;

    const uint32 mems_batch = 16*1024*1024;
    nvbio::vector<device_tag,mem_filter_type::mem_type> mems( mems_batch );

    while (1)
    {
        // load a batch of reads
        SharedPointer<io::ReadData> h_read_data( read_data_file->next( batch_reads, batch_bps ) );
        if (h_read_data == NULL)
            break;

        log_info(stderr, "  loading reads... started\n");

        // copy it to the device
        const io::ReadDataDevice d_read_data( *h_read_data );

        const uint32 n_reads = d_read_data.size() / 2;

        log_info(stderr, "  loading reads... done\n");
        log_info(stderr, "    %u reads\n", n_reads);

        log_info(stderr, "  ranking MEMs... started\n");

        Timer timer;
        timer.start();

        mem_filter.rank(
            f_index,
            r_index,
            d_read_data.const_read_string_set(),
            min_intv );

        hipDeviceSynchronize();
        timer.stop();

        const uint64 n_mems = mem_filter.n_mems();

        log_info(stderr, "  ranking MEMs... done\n");
        log_info(stderr, "    %.1f avg ranges\n", float( mem_filter.n_ranges() ) / float( n_reads ) );
        log_info(stderr, "    %.1f avg MEMs\n", float( n_mems ) / float( n_reads ) );
        log_info(stderr, "    %.1f K reads/s\n", 1.0e-3f * float(n_reads) / timer.seconds());

        log_info(stderr, "  locating MEMs... started\n");

        float locate_time = 0.0f;

        // loop through large batches of hits and locate & merge them
        for (uint64 mems_begin = 0; mems_begin < n_mems; mems_begin += mems_batch)
        {
            const uint64 mems_end = nvbio::min( mems_begin + mems_batch, n_mems );

            timer.start();

            mem_filter.locate(
                mems_begin,
                mems_end,
                mems.begin() );

            hipDeviceSynchronize();
            timer.stop();
            locate_time += timer.seconds();

            log_verbose(stderr, "\r    %5.2f%% (%4.1f M MEMs/s)",
                 100.0f * float( mems_end ) / float( n_mems ),
                1.0e-6f * float( mems_end ) / locate_time );
        }

        log_info(stderr, "  locating MEMs... done\n");
    }
    return 0;
}
