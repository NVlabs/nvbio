#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// alignment_test.cu
//

#include <nvbio-test/alignment_test_utils.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/basic/cached_iterator.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/vector_view.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/dna.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <nvbio/alignment/sink.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

using namespace nvbio;

namespace nvbio {
namespace aln {

enum { CACHE_SIZE = 32 };
typedef nvbio::lmem_cache_tag<CACHE_SIZE>                                       lmem_cache_tag_type;
typedef nvbio::uncached_tag                                                     uncached_tag_type;

//
// An alignment stream class to be used in conjunction with the BatchAlignmentScore class
//
template <typename t_aligner_type, uint32 M, uint32 N, typename cache_type = lmem_cache_tag_type>
struct AlignmentStream
{
    typedef t_aligner_type                                                          aligner_type;

    typedef nvbio::cuda::ldg_pointer<uint32>                                        storage_iterator;

    typedef nvbio::PackedStringLoader<storage_iterator,4,false,cache_type>          pattern_loader_type;
    typedef typename pattern_loader_type::input_iterator                            uncached_pattern_iterator;
    typedef typename pattern_loader_type::iterator                                  pattern_iterator;
    typedef nvbio::vector_view<pattern_iterator>                                    pattern_string;

    typedef nvbio::PackedStringLoader<storage_iterator,2,false,cache_type>          text_loader_type;
    typedef typename text_loader_type::input_iterator                               uncached_text_iterator;
    typedef typename text_loader_type::iterator                                     text_iterator;
    typedef nvbio::vector_view<text_iterator>                                       text_string;

    // an alignment context
    struct context_type
    {
        int32                   min_score;
        aln::BestSink<int32>    sink;
    };
    // a container for the strings to be aligned
    struct strings_type
    {
        pattern_loader_type     pattern_loader;
        text_loader_type        text_loader;
        pattern_string          pattern;
        trivial_quality_string  quals;
        text_string             text;
    };

    // constructor
    AlignmentStream(
        aligner_type        _aligner,
        const uint32        _count,
        const uint32*       _patterns,
        const uint32*       _text,
               int16*       _scores) :
        m_aligner( _aligner ), m_count(_count), m_patterns(storage_iterator(_patterns)), m_text(storage_iterator(_text)), m_scores(_scores) {}

    // get the aligner
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    const aligner_type& aligner() const { return m_aligner; };

    // return the maximum pattern length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_pattern_length() const { return M; }

    // return the maximum text length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_text_length() const { return N; }

    // return the stream size
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 size() const { return m_count; }

    // return the i-th pattern's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 pattern_length(const uint32 i, context_type* context) const { return M; }

    // return the i-th text's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 text_length(const uint32 i, context_type* context) const { return N; }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    bool init_context(
        const uint32    i,
        context_type*   context) const
    {
        context->min_score = Field_traits<int32>::min();
        return true;
    }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void load_strings(
        const uint32        i,
        const uint32        window_begin,
        const uint32        window_end,
        const context_type* context,
              strings_type* strings) const
    {
        strings->pattern = pattern_string( M,
            strings->pattern_loader.load(
                m_patterns + i * M,
                M,
                make_uint2( window_begin, window_end ),
                false ) );

        strings->text = text_string( N, strings->text_loader.load( m_text + i * N, N ) );
    }

    // handle the output
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void output(
        const uint32        i,
        const context_type* context) const
    {
        // copy the output score
        m_scores[i] = context->sink.score;
    }

    aligner_type                m_aligner;
    uint32                      m_count;
    uncached_pattern_iterator   m_patterns;
    uncached_text_iterator      m_text;
    int16*                      m_scores;
};

// A simple kernel to test the speed of alignment without the possible overheads of the BatchAlignmentScore interface
//
template <uint32 BLOCKDIM, uint32 MAX_REF_LEN, typename aligner_type, typename score_type>
__global__ void alignment_test_kernel(const aligner_type aligner, const uint32 N_probs, const uint32 M, const uint32 N, const uint32* strptr, const uint32* refptr, score_type* score)
{
    const uint32 tid = blockIdx.x * BLOCKDIM + threadIdx.x;

    typedef lmem_cache_tag_type                                                 lmem_cache_type;
    typedef nvbio::cuda::ldg_pointer<uint32>                                    storage_iterator;

    typedef nvbio::PackedStringLoader<storage_iterator,4,false,lmem_cache_type>     pattern_loader_type;
    typedef typename pattern_loader_type::input_iterator                            uncached_pattern_iterator;
    typedef typename pattern_loader_type::iterator                                  pattern_iterator;
    typedef nvbio::vector_view<pattern_iterator>                                    pattern_string;

    typedef nvbio::PackedStringLoader<storage_iterator,2,false,lmem_cache_type>     text_loader_type;
    typedef typename text_loader_type::input_iterator                               uncached_text_iterator;
    typedef typename text_loader_type::iterator                                     text_iterator;
    typedef nvbio::vector_view<text_iterator>                                       text_string;

    pattern_loader_type pattern_loader;
    pattern_string pattern = pattern_string( M, pattern_loader.load( uncached_pattern_iterator( strptr ) + tid * M, tid < N_probs ? M : 0u ) );

    text_loader_type text_loader;
    text_string text = text_string( N, text_loader.load( uncached_text_iterator( refptr ) + tid * N, tid < N_probs ? N : 0u ) );

    aln::BestSink<int32> sink;

    aln::alignment_score<MAX_REF_LEN>(
        aligner,
        pattern,
        aln::trivial_quality_string(),
        text,
        Field_traits<int32>::min(),
        sink );

    score[tid] = sink.score;
}

//
// A class for making a single alignment test, testing both scoring and traceback
//
struct SingleTest
{
    thrust::host_vector<uint8>   str_hvec;
    thrust::host_vector<uint8>   ref_hvec;
    thrust::device_vector<uint8> str_dvec;
    thrust::device_vector<uint8> ref_dvec;
    thrust::device_vector<float> temp_dvec;
    thrust::device_vector<float> score_dvec;
    thrust::device_vector<uint2> sink_dvec;

    // test full DP alignment
    //
    // \param test              test name
    // \param aligner           alignment algorithm
    // \param ref_alignment     reference alignment string
    //
    template <uint32 BLOCKDIM, uint32 N, uint32 M, typename aligner_type>
    void full(const char* test, const aligner_type aligner, const char* ref_alignment)
    {
        NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 32u;

        typedef ScoreMatrices<N,M,typename aligner_type::aligner_tag> SWMatrices;

        SharedPointer<SWMatrices> mat = SharedPointer<SWMatrices>( new SWMatrices() );

        const uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        const uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        typename column_storage_type<aligner_type>::type column[N];

        const int32 ref_score = ref_sw<M,N>( str_hptr, ref_hptr, aligner, mat.get() );

        aln::BestSink<int32> sink;
        aln::alignment_score(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            sink,
            column );

        const int32 cpu_score = sink.score;

        if (cpu_score != ref_score)
        {
            log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
            exit(1);
        }

        TestBacktracker backtracker;
        backtracker.clear();

        const Alignment<int32> aln = aln::alignment_traceback<1024u,1024u,CHECKPOINTS>(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            backtracker );

        const int32 aln_score = backtracker.score( aligner, aln.source.x, str_hptr, ref_hptr );
        const std::string aln_string = rle( backtracker.aln ).c_str();
        if (aln_score != ref_score)
        {
            log_error(stderr, "    expected %s backtracking score %d, got %d\n", test, ref_score, aln_score);
            log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
            //mat->print();
            exit(1);
        }
        fprintf(stderr, "    %15s : ", test);
        fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
        if (strcmp( ref_alignment, aln_string.c_str() ) != 0)
        {
            log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
            exit(1);
        }
    }

    // test banded alignment
    //
    // \param test              test name
    // \param aligner           alignment algorithm
    // \param ref_alignment     reference alignment string
    //
    template <uint32 BLOCKDIM, uint32 BAND_LEN, const uint32 N, const uint32 M, typename aligner_type>
    void banded(const char* test, const aligner_type aligner, const char* ref_alignment)
    {
        NVBIO_VAR_UNUSED const uint32 CHECKPOINTS = 32u;

        const uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        const uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        const int32 ref_score = ref_banded_sw<M,N,BAND_LEN>( str_hptr, ref_hptr, 0u, aligner );

        aln::BestSink<int32> sink;
        aln::banded_alignment_score<BAND_LEN>(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            sink );

        const int32 cpu_score = sink.score;
        if (cpu_score != ref_score)
        {
            log_error(stderr, "    expected %s score %d, got: %d\n", test, ref_score, cpu_score);
            exit(1);
        }

        TestBacktracker backtracker;
        backtracker.clear();

        const Alignment<int32> aln = aln::banded_alignment_traceback<BAND_LEN,1024u,CHECKPOINTS>(
            aligner,
            vector_view<const uint8*>( M, str_hptr ),
            trivial_quality_string(),
            vector_view<const uint8*>( N, ref_hptr ),
            -1000,
            backtracker );

        const int32 aln_score = backtracker.score( aligner, aln.source.x, str_hptr, ref_hptr );
        const std::string aln_string = rle( backtracker.aln ).c_str();
        if (aln_score != ref_score)
        {
            log_error(stderr, "    expected %s backtracking score %d, got %d\n", ref_score, aln_score);
            log_error(stderr, "    %s - %d - [%u, %u] x [%u, %u]\n", aln_string.c_str(), aln.score, aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
            exit(1);
        }
        fprintf(stderr, "    %15s : ", test);
        fprintf(stderr, "%d - %s - [%u:%u] x [%u:%u]\n", aln.score, aln_string.c_str(), aln.source.x, aln.sink.x, aln.source.y, aln.sink.y);
        if (strcmp( ref_alignment, aln_string.c_str() ) != 0)
        {
            log_error(stderr, "    expected %s, got %s\n", ref_alignment, aln_string.c_str());
            exit(1);
        }
    }
};

// execute a given batch alignment type on a given stream
//
// \tparam batch_type               a \ref BatchAlignment "Batch Alignment"
// \tparam stream_type              a stream compatible to the given batch_type
//
// \return                          average time
//
template <typename batch_type, typename stream_type>
float enact_batch(
          batch_type&               batch,
    const stream_type&              stream,
    const uint32                    n_tests,
    const uint32                    n_tasks)
{
    // alloc all the needed temporary storage
    const uint64 temp_size = batch_type::max_temp_storage(
        stream.max_pattern_length(),
        stream.max_text_length(),
        stream.size() );

    thrust::device_vector<uint8> temp_dvec( temp_size );

    Timer timer;
    timer.start();

    for (uint32 i = 0; i < n_tests; ++i)
    {
        // enact the batch
        batch.enact( stream, temp_size, nvbio::raw_pointer( temp_dvec ) );

        hipDeviceSynchronize();
    }

    timer.stop();

    return timer.seconds() / float(n_tests);
}

// execute and time a batch of full DP alignments using BatchAlignmentScore
//
template <bool supported, typename scheduler_type, uint32 N, uint32 M, typename stream_type>
struct batch_score_profile_dispatch
{
    static void run(
        const stream_type               stream,
        const uint32                    n_tests,
        const uint32                    n_tasks)
    {}
};

// execute and time a batch of full DP alignments using BatchAlignmentScore
//
template <typename scheduler_type, uint32 N, uint32 M, typename stream_type>
struct batch_score_profile_dispatch<true,scheduler_type,N,M,stream_type>
{
    static void run(
        const stream_type               stream,
        const uint32                    n_tests,
        const uint32                    n_tasks)
    {
        typedef aln::BatchedAlignmentScore<stream_type, scheduler_type> batch_type;  // our batch type

        // setup a batch
        batch_type batch;

        const float time = enact_batch(
            batch,
            stream,
            n_tests,
            n_tasks );

        fprintf(stderr,"  %5.1f", 1.0e-9f * float(n_tasks*uint64(N*M))/time );
    }
};

// execute and time a batch of full DP alignments using BatchAlignmentScore
//
template <typename scheduler_type, uint32 N, uint32 M, typename stream_type>
void batch_score_profile(
    const stream_type               stream,
    const uint32                    n_tests,
    const uint32                    n_tasks)
{
    NVBIO_VAR_UNUSED const bool is_supported = aln::supports_scheduler<typename stream_type::aligner_type,scheduler_type>::pred;

    batch_score_profile_dispatch<is_supported,scheduler_type,N,M,stream_type>::run(
        stream,
        n_tests,
        n_tasks );
}

// execute and time the batch_score<scheduler> algorithm for all possible schedulers
//
template <uint32 N, uint32 M, typename aligner_type>
void batch_score_profile_all(
    const aligner_type              aligner,
    const uint32                    n_tests,
    const uint32                    n_tasks,
    thrust::device_vector<uint32>&  pattern_dvec,
    thrust::device_vector<uint32>&  text_dvec,
    thrust::device_vector<int16>&   score_dvec)
{
    {
        typedef AlignmentStream<aligner_type,M,N> stream_type;

        // create a stream
        stream_type stream(
            aligner,
            n_tasks,
            nvbio::raw_pointer( pattern_dvec ),
            nvbio::raw_pointer( text_dvec ),
            nvbio::raw_pointer( score_dvec ) );

        // test the DeviceThreadScheduler
        batch_score_profile<DeviceThreadScheduler,N,M>(
            stream,
            n_tests,
            n_tasks );

        // test the DeviceStagedThreadScheduler
        batch_score_profile<DeviceStagedThreadScheduler,N,M>(
            stream,
            n_tests,
            n_tasks );
    }
    {
        typedef AlignmentStream<aligner_type,M,N,uncached_tag_type> stream_type;

        // create a stream
        stream_type stream(
            aligner,
            n_tasks,
            nvbio::raw_pointer( pattern_dvec ),
            nvbio::raw_pointer( text_dvec ),
            nvbio::raw_pointer( score_dvec ) );

        // test the DeviceWarpScheduler
        batch_score_profile<DeviceWarpScheduler,N,M>(
            stream,
            n_tests,
            n_tasks );
    }
    {
        const uint32 BLOCKDIM = 128;
        const uint32 N_BLOCKS = (n_tasks + BLOCKDIM-1) / BLOCKDIM;

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // enact the batch
            alignment_test_kernel<BLOCKDIM,N> <<<N_BLOCKS,BLOCKDIM>>>(
                aligner,
                n_tasks,
                M,
                N,
                nvbio::raw_pointer( pattern_dvec ),
                nvbio::raw_pointer( text_dvec ),
                nvbio::raw_pointer( score_dvec ) );

            hipDeviceSynchronize();
        }

        timer.stop();

        const float time = timer.seconds();

        fprintf(stderr,"  %5.1f", 1.0e-9f * float(n_tasks*uint64(N*M))*(float(n_tests)/time) );
    }
    fprintf(stderr, " GCUPS\n");
}

// execute and time a batch of banded alignments using BatchBandedAlignmentScore
//
template <uint32 BAND_LEN, typename scheduler_type, uint32 N, uint32 M, typename stream_type>
void batch_banded_score_profile(
    const stream_type               stream,
    const uint32                    n_tests,
    const uint32                    n_tasks)
{
    typedef aln::BatchedBandedAlignmentScore<BAND_LEN,stream_type, scheduler_type> batch_type;  // our batch type

    // setup a batch
    batch_type batch;

    const float time = enact_batch(
        batch,
        stream,
        n_tests,
        n_tasks );

    fprintf(stderr,"  %5.1f", 1.0e-9f * float(n_tasks*uint64(BAND_LEN*M))*(float(n_tests)/time) );
}
// execute and time the batch_banded_score<scheduler> algorithm for all possible schedulers
//
template <uint32 BAND_LEN, uint32 N, uint32 M, typename aligner_type>
void batch_banded_score_profile_all(
    const aligner_type              aligner,
    const uint32                    n_tests,
    const uint32                    n_tasks,
    thrust::device_vector<uint32>&  pattern_dvec,
    thrust::device_vector<uint32>&  text_dvec,
    thrust::device_vector<int16>&   score_dvec)
{
    typedef AlignmentStream<aligner_type,M,N> stream_type;

    // create a stream
    stream_type stream(
        aligner,
        n_tasks,
        nvbio::raw_pointer( pattern_dvec ),
        nvbio::raw_pointer( text_dvec ),
        nvbio::raw_pointer( score_dvec ) );

    // test the DeviceThreadScheduler
    batch_banded_score_profile<BAND_LEN,DeviceThreadScheduler,N,M>(
        stream,
        n_tests,
        n_tasks );

    // test the DeviceStagedThreadScheduler
    batch_banded_score_profile<BAND_LEN,DeviceStagedThreadScheduler,N,M>(
        stream,
        n_tests,
        n_tasks );

    // TODO: test DeviceWarpScheduler
    fprintf(stderr, " GCUPS\n");
}

// a simple banded edit distance test
//
template <typename string_type>
void banded_edit_distance_test(
    const uint32      test_id,
    const string_type pattern,
    const string_type text,
    const int32       ref_score)
{
    const int32 ed = banded_alignment_score<5>(
        make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
        pattern,
        text,
        -255 );

    if (ed != ref_score)
    {
        log_error(stderr, "  synthetic Edit Distance test %u... failed\n", test_id);
        log_error(stderr, "    expected %d, got: %d - pattern: %s text: %s\n", ref_score, ed, pattern.begin(), text.begin());
        exit(1);
    }
    else
        fprintf(stderr, "  synthetic Edit Distance test %u... passed!\n", test_id);
}

void test(int argc, char* argv[])
{
                     uint32 n_tests          = 1;
    NVBIO_VAR_UNUSED uint32 N_WARP_TASKS     = 4096;
                     uint32 N_THREAD_TASKS   = 128*1024;
                     uint32 TEST_MASK        = 0xFFFFFFFFu;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-N-thread-tasks" ) == 0)
            N_THREAD_TASKS = atoi( argv[++i] );
        else if (strcmp( argv[i], "-N-warp-tasks" ) == 0)
            N_WARP_TASKS = atoi( argv[++i] );
        else if (strcmp( argv[i], "-N-tests" ) == 0)
            n_tests = atoi( argv[++i] );
        else if (strcmp( argv[i], "-tests" ) == 0)
        {
            const std::string tests_string( argv[++i] );

            char temp[256];
            const char* begin = tests_string.c_str();
            const char* end   = begin;

            TEST_MASK = 0u;

            while (1)
            {
                while (*end != ':' && *end != '\0')
                {
                    temp[end - begin] = *end;
                    end++;
                }

                temp[end - begin] = '\0';

                if (strcmp( temp, "functional" ) == 0)
                    TEST_MASK |= FUNCTIONAL;
                else if (strcmp( temp, "ed" ) == 0)
                    TEST_MASK |= ED;
                else if (strcmp( temp, "ed-banded" ) == 0)
                    TEST_MASK |= ED_BANDED;
                else if (strcmp( temp, "sw" ) == 0)
                    TEST_MASK |= SW;
                else if (strcmp( temp, "sw-banded" ) == 0)
                    TEST_MASK |= SW_BANDED;
                else if (strcmp( temp, "sw-warp" ) == 0)
                    TEST_MASK |= SW_WARP;
                else if (strcmp( temp, "sw-striped" ) == 0)
                    TEST_MASK |= SW_STRIPED;
                else if (strcmp( temp, "gotoh" ) == 0)
                    TEST_MASK |= GOTOH;
                else if (strcmp( temp, "gotoh-banded" ) == 0)
                    TEST_MASK |= GOTOH_BANDED;

                if (*end == '\0')
                    break;

                ++end; begin = end;
            }
        }
    }

    fprintf(stderr,"testing alignment... started\n");

    if (TEST_MASK & FUNCTIONAL)
    {
        typedef vector_view<const char*> const_string;

        // right aligned, no gaps
        {
            const_string  text = make_string("AAAAGGGTGCTCAA");
            const_string  pattern  = make_string("GGGTGCTCAA");

            banded_edit_distance_test(
                1u,         // test id
                pattern,    // pattern
                text,       // text
                0 );        // expected score
        }
        // right aligned, 2 insertions
        {
            const_string  text = make_string("AAAAGGGTGCTCAA");
            const_string  pattern  = make_string("GGGTAAGCTC");

            banded_edit_distance_test(
                2u,         // test id
                pattern,    // pattern
                text,       // text
                -2 );       // expected score
        }
        // right aligned, 2 deletions
        {
            const_string   text = make_string("AAAAGGGTGCAATC");
            const_string pattern  = make_string("AAGGGTGCTC");

            banded_edit_distance_test(
                3u,         // test id
                pattern,    // pattern
                text,       // text
                -2 );       // expected score
        }
        // left aligned, zero gaps
        {
            const_string     text = make_string("AAAAGGGTGCTCAA");
            const_string pattern  = make_string("AAAAGGGTGC");

            banded_edit_distance_test(
                4u,         // test id
                pattern,    // pattern
                text,       // text
                0 );        // expected score
        }
        // left aligned, 2 deletions
        {
            const_string     text = make_string("AAAAGGAAGTGCTC");
            const_string pattern  = make_string("AAAAGGGTG");

            banded_edit_distance_test(
                5u,         // test id
                pattern,    // pattern
                text,       // text
                -2 );       // expected score
        }
        // centrally aligned, 2 insertions
        {
            const_string   text = make_string("AACAGGGTGCTC");
            const_string pattern  = make_string("CACCGGGT");

            banded_edit_distance_test(
                6u,         // test id
                pattern,    // pattern
                text,       // text
                -2 );       // expected score
        }
    }

    if (TEST_MASK & FUNCTIONAL)
    {
        NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
        const uint32 M = 7;
        const uint32 N = 20;

        thrust::host_vector<uint8> str_hvec( M );
        thrust::host_vector<uint8> ref_hvec( N );

        uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        string_to_dna("ACAACTA", str_hptr);
        string_to_dna("AAACACCCTAACACACTAAA", ref_hptr);

        SingleTest test;
        nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
        nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

        {
            fprintf(stderr,"  testing Smith-Waterman scoring...\n");
            aln::SimpleSmithWatermanScheme scoring;
            scoring.m_match     =  2;
            scoring.m_mismatch  = -1;
            scoring.m_deletion  = -1;
            scoring.m_insertion = -1;

            test.full<BLOCKDIM,N,M>(      "global",  make_smith_waterman_aligner<aln::GLOBAL>( scoring ),      "1M2D3M1D3M10D" );
            test.full<BLOCKDIM,N,M>(       "local",  make_smith_waterman_aligner<aln::LOCAL>( scoring ),       "4M1D3M" );
            test.full<BLOCKDIM,N,M>( "semi-global",  make_smith_waterman_aligner<aln::SEMI_GLOBAL>( scoring ), "4M1D3M" );
        }
        {
            fprintf(stderr,"  testing Gotoh scoring...\n");
            aln::SimpleGotohScheme scoring;
            scoring.m_match    =  2;
            scoring.m_mismatch = -1;
            scoring.m_gap_open = -1;
            scoring.m_gap_ext  = -1;

            test.full<BLOCKDIM,N,M>(      "global", make_gotoh_aligner<aln::GLOBAL>( scoring ),      "1M2D3M1D3M10D" );
            test.full<BLOCKDIM,N,M>(       "local", make_gotoh_aligner<aln::LOCAL>( scoring ),       "4M1D3M" );
            test.full<BLOCKDIM,N,M>( "semi-global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "4M1D3M" );
            test.banded<BLOCKDIM, 7u, N, M>( "banded-semi-global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "4M1D3M" );
        }
    }

    if (TEST_MASK & FUNCTIONAL)
    {
        fprintf(stderr,"  testing real banded Gotoh problem...\n");
        NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
        NVBIO_VAR_UNUSED const uint32 BAND_LEN = 31;
        NVBIO_VAR_UNUSED const uint32 M = 150;
        NVBIO_VAR_UNUSED const uint32 N = 150 + 31;

        thrust::host_vector<uint8> str_hvec( M );
        thrust::host_vector<uint8> ref_hvec( N );

        uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );
        string_to_dna("TTATGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTAT", str_hptr);
        string_to_dna("ATCGGATTCTTTCTTACTTGTAGGTGGTCTGGTTTTTGCCTTTTAAGCTTCTGCAAAAAACAACAACAAACTTGTGGTATTACACTGACTCTACAGATCAATTTGGGGACAACTTCCATGTGTTCCACCACCAATACTGAATCTTTCAATCGACTGACGTGGTATCTCTCTCTCCATCTAT", ref_hptr);

        aln::SimpleGotohScheme scoring;
        scoring.m_match    =  0;
        scoring.m_mismatch = -5;
        scoring.m_gap_open = -8;
        scoring.m_gap_ext  = -3;

        SingleTest test;
        nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
        nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

        test.banded<BLOCKDIM, BAND_LEN, N, M>( "banded-semi-global", make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ), "147M2D3M" );
    }

    // This code is for debugging purposes, useful to plug-in and analyze real problems coming from an app
    if (TEST_MASK & FUNCTIONAL)
    {
        fprintf(stderr,"  testing real full-matrix Gotoh problem...\n");
        NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
        NVBIO_VAR_UNUSED const uint32 M = 144;
        NVBIO_VAR_UNUSED const uint32 N = 500;

        thrust::host_vector<uint8> str_hvec( M );
        thrust::host_vector<uint8> ref_hvec( N );

        uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        const char* str_ascii =
            "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";
        const char* ref_ascii =
            "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
            "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
            "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
            "AAATTAAGTATTGAAAAAATGC";

        string_to_dna( str_ascii, str_hptr );
        string_to_dna( ref_ascii, ref_hptr );

        aln::SimpleGotohScheme scoring;
        scoring.m_match    =  0;
        scoring.m_mismatch = -5;
        scoring.m_gap_open = -8;
        scoring.m_gap_ext  = -3;

        aln::GotohAligner<aln::SEMI_GLOBAL, aln::SimpleGotohScheme> aligner( scoring );

        SingleTest test;
        nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
        nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

        test.full<BLOCKDIM,N,M>( "semi-global", aligner, "6I138M" );
    }

    // This code is for debugging purposes, useful to plug-in and analyze real problems coming from an app
    if (TEST_MASK & FUNCTIONAL)
    {
        fprintf(stderr,"  testing real full-matrix Edit Distance problem...\n");
        NVBIO_VAR_UNUSED const uint32 BLOCKDIM = 128;
        NVBIO_VAR_UNUSED const uint32 M = 144;
        NVBIO_VAR_UNUSED const uint32 N = 500;

        thrust::host_vector<uint8> str_hvec( M );
        thrust::host_vector<uint8> ref_hvec( N );

        uint8* str_hptr = nvbio::raw_pointer( str_hvec );
        uint8* ref_hptr = nvbio::raw_pointer( ref_hvec );

        const char* str_ascii =
            "TAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAGCAGCACCTGGTAAATTAAGTATTGAAAAAATGCAGATCG";
        const char* ref_ascii =
            "CAGCACTGACCGGTGAGCATAAACCCTGGGGATGCCCAGAGCTGGTACAGCCAGGAGCTCCAGAAGCGTGGGATTCTCAGAGGGAAGTGGAGCTCACTGCTCTACAGGTCCTATTCAAGTTAGAAAGTAAGATACAATGCACACAAAGCCAAATTGTC"
            "ATCATTCAGCTCCTATTACAGGGGAACTAAGAGCTGCATTGAAAATTATTTGCAAAGCTTGTAAGTGGTTCTGCCACTTATTAGCCGTGTGAACCTTAGCAAATTACCTAGCGTCTCTGAGTTTCAACTTCCTCATCTACAAAATAGAAATGATAATAAT"
            "AACCGCATCGCAAGAGTTGTTGGAAAAATGAAAATGAGGTATCATAGGAGGTAACATGTATGGAGCATTTACCATAGGCCAAGCACTGTTCTAAGAACTTCGGACATGTTATCTCACTTGTATAAGTACTTAGGTGCCTACAACATAAACAGCACCTGGT"
            "AAATTAAGTATTGAAAAAATGC";

        string_to_dna( str_ascii, str_hptr );
        string_to_dna( ref_ascii, ref_hptr );

        aln::EditDistanceAligner<aln::SEMI_GLOBAL> aligner;

        SingleTest test;
        nvbio::cuda::thrust_copy_vector(test.str_hvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_hvec, ref_hvec);
        nvbio::cuda::thrust_copy_vector(test.str_dvec, str_hvec);
        nvbio::cuda::thrust_copy_vector(test.ref_dvec, ref_hvec);

        test.full<BLOCKDIM,N,M>( "semi-global", aligner, "1I1M2I1M3I136M" );
    }

    // do a larger speed test of the Gotoh alignment
    if (TEST_MASK & (ED | SW | GOTOH))
    {
        const uint32 N_TASKS = N_THREAD_TASKS;
        const uint32 M = 150;
        const uint32 N = 500;

        const uint32 M_WORDS = (M + 7)  >> 3;
        const uint32 N_WORDS = (N + 15) >> 4;

        thrust::host_vector<uint32> str( M_WORDS * N_TASKS );
        thrust::host_vector<uint32> ref( N_WORDS * N_TASKS );

        LCG_random rand;
        fill_packed_stream<4u>( rand, 4u, M * N_TASKS, nvbio::raw_pointer( str ) );
        fill_packed_stream<2u>( rand, 4u, N * N_TASKS, nvbio::raw_pointer( ref ) );

        thrust::device_vector<uint32> str_dvec( str );
        thrust::device_vector<uint32> ref_dvec( ref );
        thrust::device_vector<int16>  score_dvec( N_TASKS );

        if (TEST_MASK & ED)
        {
            fprintf(stderr,"  testing Edit Distance scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_score_profile_all<N,M>(
                    make_edit_distance_aligner<aln::GLOBAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all<N,M>(
                    make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_score_profile_all<N,M>(
                    make_edit_distance_aligner<aln::LOCAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
        if (TEST_MASK & ED)
        {
            aln::SimpleSmithWatermanScheme scoring;
            scoring.m_match     =  2;
            scoring.m_mismatch  = -1;

            fprintf(stderr,"  testing Hamming Distance scoring speed...\n");
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all<N,M>(
                    make_hamming_distance_aligner<aln::SEMI_GLOBAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_score_profile_all<N,M>(
                    make_hamming_distance_aligner<aln::LOCAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
        if (TEST_MASK & SW)
        {
            aln::SimpleSmithWatermanScheme scoring;
            scoring.m_match     =  2;
            scoring.m_mismatch  = -1;
            scoring.m_deletion  = -1;
            scoring.m_insertion = -1;

            fprintf(stderr,"  testing Smith-Waterman scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_score_profile_all<N,M>(
                    make_smith_waterman_aligner<aln::GLOBAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all<N,M>(
                    make_smith_waterman_aligner<aln::SEMI_GLOBAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_score_profile_all<N,M>(
                    make_smith_waterman_aligner<aln::LOCAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
        if (TEST_MASK & GOTOH)
        {
            aln::SimpleGotohScheme scoring;
            scoring.m_match    =  2;
            scoring.m_mismatch = -1;
            scoring.m_gap_open = -1;
            scoring.m_gap_ext  = -1;

            fprintf(stderr,"  testing Gotoh scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_score_profile_all<N,M>(
                    make_gotoh_aligner<aln::GLOBAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all<N,M>(
                    make_gotoh_aligner<aln::SEMI_GLOBAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_score_profile_all<N,M>(
                    make_gotoh_aligner<aln::LOCAL>( scoring ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
    }
    // do a larger speed test of the banded SW alignment
    if (TEST_MASK & (ED_BANDED | SW_BANDED | GOTOH_BANDED))
    {
        const uint32 BAND_LEN = 15u;
        const uint32 N_TASKS  = N_THREAD_TASKS;
        const uint32 M = 150;
        const uint32 N = M+BAND_LEN;

        const uint32 M_WORDS = (M + 7)  >> 3;
        const uint32 N_WORDS = (N + 15) >> 4;

        thrust::host_vector<uint32> str( M_WORDS * N_TASKS );
        thrust::host_vector<uint32> ref( N_WORDS * N_TASKS );

        LCG_random rand;
        fill_packed_stream<4u>( rand, 4u, M * N_TASKS, nvbio::raw_pointer( str ) );
        fill_packed_stream<2u>( rand, 4u, N * N_TASKS, nvbio::raw_pointer( ref ) );

        thrust::device_vector<uint32> str_dvec( str );
        thrust::device_vector<uint32> ref_dvec( ref );
        thrust::device_vector<int16>  score_dvec( N_TASKS );

        if (TEST_MASK & ED_BANDED)
        {
            fprintf(stderr,"  testing banded Edit Distance scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_edit_distance_aligner<aln::GLOBAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_edit_distance_aligner<aln::SEMI_GLOBAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_edit_distance_aligner<aln::LOCAL>(),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
        if (TEST_MASK & SW_BANDED)
        {
            fprintf(stderr,"  testing banded Smith-Waterman scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_smith_waterman_aligner<aln::GLOBAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_smith_waterman_aligner<aln::SEMI_GLOBAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_smith_waterman_aligner<aln::LOCAL>( aln::SimpleSmithWatermanScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
        if (TEST_MASK & GOTOH_BANDED)
        {
            fprintf(stderr,"  testing banded Gotoh scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_gotoh_aligner<aln::GLOBAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_gotoh_aligner<aln::SEMI_GLOBAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_banded_score_profile_all<BAND_LEN,N,M>(
                    make_gotoh_aligner<aln::LOCAL>( aln::SimpleGotohScheme(2,-1,-1,-1) ),
                    n_tests,
                    N_TASKS,
                    str_dvec,
                    ref_dvec,
                    score_dvec );
            }
        }
    }
    fprintf(stderr,"testing alignment... done\n");
}

} // namespace sw
} // namespace nvbio
