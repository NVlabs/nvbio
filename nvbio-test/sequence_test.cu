#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// sequence_test.cu
//

#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/dna.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/io/sequence/sequence_mmap.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

namespace nvbio {


int sequence_test(int argc, char* argv[])
{
    char* index_name = NULL;
    char* reads_name = NULL;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-map" ) == 0)
            index_name = argv[++i];
        else if (strcmp( argv[i], "-reads" ) == 0)
            reads_name = argv[++i];
    }

    log_info(stderr,"testing sequence-data... started\n");

    try
    {
        if (index_name != NULL)
        {
            log_verbose(stderr, "  loading sequence file %s\n", index_name );

            // try to load the index in memory
            io::SequenceDataHost index;
            if (io::load_sequence_file(
                DNA,
                &index,
                index_name ) == false)
            {
                log_error(stderr,"  loading file %s failed\n", index_name);
                return 0;
            }

            log_verbose(stderr, "  sequences : %u\n", index.size() );
            log_verbose(stderr, "  bps       : %u\n", index.bps() );
            log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                index.avg_sequence_len(),
                index.min_sequence_len(),
                index.max_sequence_len() );

            // try to load the index in mapped-memory
            io::SequenceDataMMAPServer server;
            if (server.load( DNA, index_name, "test", io::SequenceFlags( io::SEQUENCE_DATA | io::SEQUENCE_NAMES ) ) == false)
            {
                log_error(stderr,"  server mapping of file %s failed\n", index_name);
                return 0;
            }

            // scope the client so as to make sure it's destroyed before the server
            {
                // and map it into a client
                io::SequenceDataMMAP client;
                if (client.load( "test" ) == false)
                {
                    log_error(stderr,"  client mapping of file %s failed\n", index_name);
                    return 0;
                }

                log_verbose(stderr, "  sequences : %u\n", client.size() );
                log_verbose(stderr, "  bps       : %u\n", client.bps() );
                log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                    client.avg_sequence_len(),
                    client.min_sequence_len(),
                    client.max_sequence_len() );

                // check whether the stats match
                if (static_cast<const io::SequenceDataInfo&>( index ) !=
                    static_cast<const io::SequenceDataInfo&>( client ))
                {
                    log_error(stderr,"  loaded and mapped versions of file %s do not match!\n", index_name);
                    return 0;
                }
            }
        }
        if (reads_name != NULL)
        {
            SharedPointer<io::SequenceDataStream> read_file( io::open_sequence_file( reads_name ) );
            if (read_file == NULL || read_file->is_ok() == false)
            {
                log_error(stderr,"  failed opening reads file %s\n", reads_name);
                return 0;
            }

            io::SequenceDataHost read_data;

            io::next( DNA_N, &read_data, read_file.get(), 10000 );

            log_verbose(stderr, "  sequences : %u\n", read_data.size() );
            log_verbose(stderr, "  bps       : %u\n", read_data.bps() );
            log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
                read_data.avg_sequence_len(),
                read_data.min_sequence_len(),
                read_data.max_sequence_len() );
        }
    }
    catch (...)
    {
        log_error(stderr, "caught an unknown exception!\n");
        return 0;
    }

    log_info(stderr,"testing sequence-data... done\n");
    return 1;
}

} // namespace nvbio
