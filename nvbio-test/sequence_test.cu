#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// alignment_test.cu
//

#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/dna.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/io/sequence/sequence_mmap.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

namespace nvbio {

int sequence_test(int argc, char* argv[])
{
    char* index_name = NULL;
    char* reads_name = NULL;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-map" ) == 0)
            index_name = argv[++i];
        else if (strcmp( argv[i], "-reads" ) == 0)
            reads_name = argv[++i];
    }

    log_info(stderr,"testing sequence-data... started\n");

    if (index_name != NULL)
    {
        io::SequenceDataMMAPServer server;
        if (server.load( DNA, index_name, "test", io::SequenceFlags( io::SEQUENCE_DATA | io::SEQUENCE_NAMES ) ) == false)
        {
            log_error(stderr,"  server mapping of file %s failed\n", index_name);
            return 0;
        }

        io::SequenceDataMMAP client;
        if (client.load( "test" ) == false)
        {
            log_error(stderr,"  client mapping of file %s failed\n", index_name);
            return 0;
        }

        log_verbose(stderr, "  sequences : %u\n", client.size() );
        log_verbose(stderr, "  bps       : %u\n", client.bps() );
    }
    if (reads_name != NULL)
    {
        SharedPointer<io::SequenceDataStream> read_file( io::open_sequence_file( reads_name ) );
        if (read_file == NULL || read_file->is_ok() == false)
        {
            log_error(stderr,"  failed opening reads file %s\n", reads_name);
            return 0;
        }

        io::SequenceDataHost read_data;

        io::next( DNA_N, &read_data, read_file.get(), 10000 );

        log_verbose(stderr, "  sequences : %u\n", read_data.size() );
        log_verbose(stderr, "  bps       : %u\n", read_data.bps() );
        log_verbose(stderr, "  avg bps   : %u (min: %u, max: %u)\n",
            read_data.avg_sequence_len(),
            read_data.min_sequence_len(),
            read_data.max_sequence_len() );
    }

    log_info(stderr,"testing sequence-data... done\n");
    return 1;
}

} // namespace nvbio
