#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// qgram_test.cu
//
//#define CUFMI_CUDA_DEBUG
//#define CUFMI_CUDA_ASSERTS

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/vector_wrapper.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/io/reads/reads.h>
#include <nvbio/io/fmi.h>
#include <nvbio/qgram/qgram.h>
#include <nvbio/qgram/qgroup.h>

namespace nvbio {

// return the size of a given range
struct range_size
{
    typedef uint2  argument_type;
    typedef uint32 result_type;

    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 operator() (const uint2 range) const { return range.y - range.x; }
};

// return 1 for non-empty ranges, 0 otherwise
struct valid_range
{
    typedef uint2  argument_type;
    typedef uint32 result_type;

    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 operator() (const uint2 range) const { return range.y - range.x > 0 ? 1u : 0u; }
};

int qgram_test(int argc, char* argv[])
{
    uint32 len       = 10000000;
    uint32 n_queries = 10000000;
    char*  reads = "./data/SRR493095_1.fastq.gz";
    char*  index = "./data/human.NCBI36/Homo_sapiens.NCBI36.53.dna.toplevel.fa";

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-length" ) == 0)
            len = atoi( argv[++i] )*1000;
        if (strcmp( argv[i], "-reads" ) == 0)
            reads = argv[++i];
        if (strcmp( argv[i], "-index" ) == 0)
            index = argv[++i];
    }

    log_info(stderr, "q-gram test... started\n");

    const io::QualityEncoding qencoding = io::Phred33;

    log_info(stderr, "  loading reads... started\n");

    SharedPointer<io::ReadDataStream> read_data_file(
        io::open_read_file(
            reads,
            qencoding,
            uint32(-1),
            uint32(-1) ) );

    if (read_data_file == NULL || read_data_file->is_ok() == false)
    {
        log_error(stderr, "    failed opening file \"%s\"\n", reads);
        return 1u;
    }

    const uint32 batch_size = uint32(-1);
    const uint32 batch_bps  = len;

    // load a batch of reads
    SharedPointer<io::ReadData> h_read_data( read_data_file->next( batch_size, batch_bps ) );
    
    // build its device version
    io::ReadDataCUDA d_read_data( *h_read_data );

    log_info(stderr, "  loading reads... done\n");

    // fetch the actual string
    typedef io::ReadData::const_read_stream_type string_type;

    const uint32      string_len = d_read_data.bps();
    const string_type string     = string_type( d_read_data.read_stream() );

    log_info(stderr, "    symbols: %.1f M symbols\n", 1.0e-6f * float(string_len));

    io::FMIndexDataRAM fmi;
    if (!fmi.load( index, io::FMIndexData::GENOME ))
    {
        log_error(stderr, "    failed loading index \"%s\"\n", index);
        return 1u;
    }

    // build its device version
    const io::FMIndexDataCUDA fmi_cuda( fmi, io::FMIndexDataCUDA::GENOME );

    typedef io::FMIndexData::stream_type genome_type;

    const uint32      genome_len = fmi_cuda.genome_length();
    const genome_type genome( fmi_cuda.genome_stream() );

    // prepare a vector to store the query results
    thrust::device_vector<uint2> d_ranges( n_queries );

    // and start testing...
    {
        log_info(stderr, "  building q-gram index... started\n");

        // build the q-gram index
        QGramIndexDevice qgram_index;

        Timer timer;
        timer.start();

        qgram_index.build<2u>(     // implicitly convert N to A
            16u,
            string_len,
            string );

        hipDeviceSynchronize();
        timer.stop();
        float time = timer.seconds();

        log_info(stderr, "  building q-gram index... done\n");
        log_info(stderr, "    unique q-grams : %.2f M q-grams\n", 1.0e-6f * float( qgram_index.n_unique_qgrams ));
        log_info(stderr, "    throughput     : %.1f M q-grams/s\n", 1.0e-6f * float( string_len ) / time);
        log_info(stderr, "    memory usage   : %.1f MB\n", float( qgram_index.used_device_memory() ) / float(1024*1024) );

        log_info(stderr, "  querying q-gram index... started\n");

        timer.start();

        // build a q-gram search functor
        const string_qgram_search_functor<2u,QGramIndexDevice::view_type,genome_type> qgram_search(
            nvbio::plain_view( qgram_index ), genome_len, genome );

        // and search the genome q-grams in the index
        thrust::transform(
            thrust::make_counting_iterator<uint32>(0u),
            thrust::make_counting_iterator<uint32>(0u) + n_queries,
            d_ranges.begin(),
            qgram_search );

        hipDeviceSynchronize();
        timer.stop();

        time = timer.seconds();

        const uint32 n_occurrences = thrust::reduce(
            thrust::make_transform_iterator( d_ranges.begin(), range_size() ),
            thrust::make_transform_iterator( d_ranges.begin(), range_size() ) + n_queries );

        const uint32 n_matches = thrust::reduce(
            thrust::make_transform_iterator( d_ranges.begin(), valid_range() ),
            thrust::make_transform_iterator( d_ranges.begin(), valid_range() ) + n_queries );

        log_info(stderr, "  querying q-gram index... done\n");
        log_info(stderr, "    throughput     : %.2f B q-grams/s\n", (1.0e-9f * float( n_queries )) / time);
        log_info(stderr, "    matches        : %.2f M\n", 1.0e-6f * float( n_matches ) );
        log_info(stderr, "    occurrences    : %.2f M\n", 1.0e-6f * float( n_occurrences ) );
    }
    {
        log_info(stderr, "  building q-gram index... started\n");

        // build the q-group index
        QGroupIndexDevice qgroup_index;

        Timer timer;
        timer.start();

        qgroup_index.build<2u>(     // implicitly convert N to A
            16u,
            string_len,
            string );

        hipDeviceSynchronize();
        timer.stop();
        float time = timer.seconds();

        log_info(stderr, "  building q-group index... done\n");
        log_info(stderr, "    unique q-grams : %.2f M q-grams\n", 1.0e-6f * float( qgroup_index.n_unique_qgrams ));
        log_info(stderr, "    throughput     : %.1f M q-grams/s\n", 1.0e-6f * float( string_len ) / time);
        log_info(stderr, "    memory usage   : %.1f MB\n", float( qgroup_index.used_device_memory() ) / float(1024*1024) );

        log_info(stderr, "  querying q-group index... started\n");

        timer.start();

        // build a q-gram search functor
        const string_qgram_search_functor<2u,QGroupIndexDevice::view_type,genome_type> qgram_search(
            nvbio::plain_view( qgroup_index ), genome_len, genome );

        // and search the genome q-grams in the index
        thrust::transform(
            thrust::make_counting_iterator<uint32>(0u),
            thrust::make_counting_iterator<uint32>(0u) + n_queries,
            d_ranges.begin(),
            qgram_search );

        hipDeviceSynchronize();
        timer.stop();

        time = timer.seconds();

        const uint32 n_occurrences = thrust::reduce(
            thrust::make_transform_iterator( d_ranges.begin(), range_size() ),
            thrust::make_transform_iterator( d_ranges.begin(), range_size() ) + n_queries );

        const uint32 n_matches = thrust::reduce(
            thrust::make_transform_iterator( d_ranges.begin(), valid_range() ),
            thrust::make_transform_iterator( d_ranges.begin(), valid_range() ) + n_queries );

        log_info(stderr, "  querying q-group index... done\n");
        log_info(stderr, "    throughput     : %.2f B q-grams/s\n", (1.0e-9f * float( n_queries )) / time);
        log_info(stderr, "    matches        : %.2f M\n", 1.0e-6f * float( n_matches ) );
        log_info(stderr, "    occurrences    : %.2f M\n", 1.0e-6f * float( n_occurrences ) );
    }

    log_info(stderr, "q-gram test... done\n" );
    return 0;
}

} // namespace nvbio
