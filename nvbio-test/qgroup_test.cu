/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// qgroup_test.cu
//
//#define CUFMI_CUDA_DEBUG
//#define CUFMI_CUDA_ASSERTS

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/vector_wrapper.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/io/reads/reads.h>
#include <nvbio/qgroup/qgroup.h>

namespace nvbio {

int qgroup_test(int argc, char* argv[])
{
    uint32 len   = 10000000;
    char*  reads = "./data/SRR493095_1.fastq.gz";

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-length" ) == 0)
            len = atoi( argv[++i] )*1000;
        if (strcmp( argv[i], "-reads" ) == 0)
            reads = argv[++i];
    }

    log_info(stderr, "q-group test... started\n");

    const io::QualityEncoding qencoding = io::Phred33;

    log_info(stderr, "  loading reads... started\n");

    SharedPointer<io::ReadDataStream> read_data_file(
        io::open_read_file(
            reads,
            qencoding,
            uint32(-1),
            uint32(-1) ) );

    if (read_data_file == NULL || read_data_file->is_ok() == false)
    {
        log_error(stderr, "    failed opening file \"%s\"\n", reads);
        return 1u;
    }

    const uint32 batch_size = uint32(-1);
    const uint32 batch_bps  = len;

    // load a batch of reads
    SharedPointer<io::ReadData> h_read_data( read_data_file->next( batch_size, batch_bps ) );
    
    // build its device version
    io::ReadDataCUDA d_read_data( *h_read_data );

    log_info(stderr, "  loading reads... done\n");

    // fetch the actual string
    typedef io::ReadData::const_read_stream_type string_type;

    const uint32      string_len = d_read_data.bps();
    const string_type string     = string_type( d_read_data.read_stream() );

    log_info(stderr, "  building q-group... started\n");
    log_info(stderr, "    symbols: %.1f M symbols\n", 1.0e-6f * float(string_len));

    // build the Q-Group
    QGroupDevice qgroup;

    Timer timer;
    timer.start();

    qgroup.build<io::ReadData::READ_BITS>(
        8u,
        string_len,
        string );

    hipDeviceSynchronize();
    timer.stop();
    const float time = timer.seconds();

    log_info(stderr, "  building q-group... done\n");
    log_info(stderr, "    unique q-grams : %.1f M qgrams\n", 1.0e-6f * float( qgroup.n_unique_qgrams ));
    log_info(stderr, "    throughput     : %.1f M qgrams/s\n", 1.0e-6f * float( string_len ) / time);
    log_info(stderr, "    memory usage   : %.1f MB\n", float( qgroup.used_device_memory() ) / float(1024*1024) );

    log_info(stderr, "q-group test... done\n" );
    return 0;
}

} // namespace nvbio
