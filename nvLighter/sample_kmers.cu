#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// sample_kmers.h
//

#include "sample_kmers.h"
#include "utils.h"
#include <nvbio/basic/pipeline_context.h>
#include <nvbio/basic/numbers.h>
#include <nvbio/basic/bloom_filter.h>
#include <nvbio/basic/primitives.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/threads.h>
#include <nvbio/basic/system.h>
#include <nvbio/basic/exceptions.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/strings/prefetcher.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

///
/// A functor to sample kmers and insert them in a Bloom filter
///
template <typename string_set_type, typename filter_type>
struct SampleKmersFunctor
{
    /// constructor
    ///
    ///\param _k                kmer length
    ///\param _alpha            the sampling frequency
    ///\param _string_set       the input string set to sample
    ///\param _filter           the kmer Bloom filter
    ///
    NVBIO_HOST_DEVICE
    SampleKmersFunctor(
        const uint32          _k,
        const float           _alpha,
        const string_set_type _string_set,
              filter_type     _filter) :
    k(_k), kmask( (uint64(1u) << (k*2))-1u ), alpha( _alpha ), string_set( _string_set ), filter(_filter) {}

    /// functor operator
    ///
    ///\param i     input string index
    ///
    NVBIO_HOST_DEVICE
    void operator() (const uint32 i) const
    {
        typedef typename string_set_type::string_type                   string_type;
        typedef typename string_traits<string_type>::forward_iterator   forward_iterator;

        // fetch the i-th string
        const string_type string = string_set[i];

        const uint32 len = length( string );
        if (len < k)
            return;

        // build a forward string iterator
        forward_iterator it( string.begin() );

        // start with an empty kmer
        uint64 kmer     = 0u;
        uint32 kmer_len = 0u;

         // initialie a random number generator
        LCG_random random( hash(i) );

        for (uint32 j = 0; j < len; ++j)
        {
            // fetch the next character
            const uint8 c = *it; ++it;

            if (c < 4) // make sure this is not an N
            {
                kmer |= c; // insert the new character at the end of the kmer (in a big-endian encoding)
                if (kmer_len < k)
                    kmer_len++;

                if (kmer_len >= k) // check whether we have an actual 'k'-mer
                {
                    if (float( random.next() ) / float(LCG_random::MAX) < alpha)
                    {
                        // insert the kmer
                        filter.insert( kmer );
                    }
                }

                // shift the kmer to the right, dropping the last symbol
                kmer <<= 2;
                kmer &= kmask;
            }
            else
            {
                // an N, skip all k-mers containing it
                it += k-1;
                j  += k-1;

                // and reset the kmer
                kmer     = 0u;
                kmer_len = 0u;
            }
        }
    }

    const uint32            k;
    const uint64            kmask;
    const float             alpha;
    string_set_type         string_set;
    mutable filter_type     filter;
};


// process the next batch
//
bool SampleKmersStage::process(PipelineContext& context)
{
    typedef nvbio::io::SequenceDataAccess<DNA_N>::sequence_string_set_type string_set_type;

    // declare the Bloom filter type
    typedef nvbio::blocked_bloom_filter<hash_functor1, hash_functor2, uint64_2*> filter_type;

    typedef SampleKmersFunctor<string_set_type,filter_type> functor_type;

    // fetch the input
    nvbio::io::SequenceDataHost* h_read_data = context.input<nvbio::io::SequenceDataHost>( 0 );

    float time = 0.0f;

    // introduce a timing scope
    try
    {
        const nvbio::ScopedTimer<float> timer( &time );

        if (device >= 0)
        {
            //
            // Device (GPU) path
            //

            // set the device
            hipSetDevice( device );

            // copy it to the device
            nvbio::io::SequenceDataDevice d_read_data( *h_read_data );

            // build a view
            const nvbio::io::SequenceDataAccess<DNA_N> d_read_view( d_read_data );

            // build the Bloom filter
            filter_type filter( SAMPLED_KMERS_FILTER_K, filter_size, (uint64_2*)filter_storage );
            //filter_type filter( filter_size, filter_storage );

            // build the kmer sampling functor
            const functor_type kmer_filter(
                k,
                alpha,
                d_read_view.sequence_string_set(),
                filter );

            device_for_each( d_read_view.size(), kmer_filter );

            hipDeviceSynchronize();
            cuda::check_error("sample-kmers");
        }
        else
        {
            //
            // Host (CPU) path
            //

            omp_set_num_threads( -device );

            // build a view
            const io::SequenceDataAccess<DNA_N> h_read_view( *h_read_data );

            // build the Bloom filter
            filter_type filter( SAMPLED_KMERS_FILTER_K, filter_size, (uint64_2*)filter_storage );

            // build the kmer sampling functor
            const functor_type kmer_filter(
                k,
                alpha,
                h_read_view.sequence_string_set(),
                filter );

            host_for_each(
                h_read_view.size(),
                kmer_filter );
        }
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "[SampleKmersStage] caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (thrust::system::system_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a thrust::system_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "[SampleKmersStage] caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "[SampleKmersStage] caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (...)
    {
        log_error(stderr, "[SampleKmersStage] caught an unknown exception!\n");
        exit(1);
    }

    // update the time stats
    stats->m_mutex.lock();
    stats->m_time += time;

    log_info(stderr, "\r  processed reads [%llu, %llu] (%.1fM / %.2fG bps, %.1fK reads/s, %.1fM bps/s - %s<%d>)        ",
        stats->m_reads,
        stats->m_reads + h_read_data->size(),
        1.0e-6f * (h_read_data->bps()),
        1.0e-9f * (stats->m_bps + h_read_data->bps()),
        stats->m_time ? (1.0e-3f * (stats->m_reads + h_read_data->size())) / stats->m_time : 0.0f,
        stats->m_time ? (1.0e-6f * (stats->m_bps   + h_read_data->bps() )) / stats->m_time : 0.0f,
        device >= 0 ? "gpu" : "cpu",
        device >= 0 ? device : -device );

    log_debug_cont(stderr, "\n");
    log_debug(stderr,"  peak memory : %.1f GB\n", float( peak_resident_memory() ) / float(1024*1024*1024));

    stats->m_reads += h_read_data->size();
    stats->m_bps   += h_read_data->bps();
    stats->m_mutex.unlock();
    return true;
}


///
/// A functor to sample kmers and insert them in a Bloom filter
///
template <typename string_set_type, typename sampled_filter_type, typename trusted_filter_type, typename threshold_type>
struct TrustedKmersFunctor
{
    /// constructor
    ///
    ///\param _k                kmer length
    ///\param _alpha            the sampling frequency
    ///\param _string_set       the input string set to sample
    ///\param _filter           the kmer Bloom filter
    ///
    NVBIO_HOST_DEVICE
    TrustedKmersFunctor(
        const uint32                _k,
        const string_set_type       _string_set,
        const sampled_filter_type   _sampled_filter,
              trusted_filter_type   _trusted_filter,
        const threshold_type        _threshold) :
        k(_k), kmask( (uint64(1u) << (k*2))-1u ),
        string_set( _string_set ),
        sampled_filter(_sampled_filter),
        trusted_filter(_trusted_filter),
        threshold(_threshold) {}

    /// functor operator
    ///
    ///\param i     input string index
    ///
    NVBIO_HOST_DEVICE
    void operator() (const uint32 i) const
    {
        typedef typename string_set_type::string_type                                           string_type;
        typedef nvbio::StringPrefetcher< string_type, nvbio::lmem_cache_tag<MAX_READ_LENGTH> >  string_prefetcher_type;
        typedef typename string_prefetcher_type::string_type                                    local_string_type;
        typedef typename nvbio::string_traits<local_string_type>::forward_iterator              forward_iterator;

        //bool occur[MAX_READ_LENGTH];
        uint32 occur_storage[MAX_READ_LENGTH/32];

        nvbio::PackedStream<uint32*,uint8,1u,false> occur( occur_storage );

        // instantiate a prefetcher
        string_prefetcher_type string_prefetcher;

        // fetch the i-th string
        //const string_type string = string_set[i];
        const local_string_type string = string_prefetcher.load( string_set[i] );

        const uint32 len = length( string );
        if (len < k)
            return;

        // build a forward string iterator
        forward_iterator it( string.begin() );

        // start with an empty kmer
        uint64 kmer     = 0u;
        uint32 kmer_len = 0u;

        const int32 occur_cnt = len - k + 1;

        // initialize all to false
        for (uint32 j = 0; j < (occur_cnt+31)/32; ++j)
            occur_storage[j] = 0u;

        // mark occurring kmers
        for (uint32 j = 0; j < len; ++j)
        {
            // fetch the next character
            const uint8 c = *it; ++it;

            if (c < 4) // make sure this is not an N
            {
                kmer |= c; // insert the new character at the end of the kmer (in a big-endian encoding)
                if (kmer_len < k)
                    kmer_len++;

                if (kmer_len >= k) // check whether we have an actual 'k'-mer
                {
                    if (sampled_filter[ kmer ])
                        occur[j - k + 1] = true;
                }

                // shift the kmer to the right, dropping the last symbol
                kmer <<= 2;
                kmer &= kmask;
            }
            else
            {
                // an N, skip all kmers containing it
                it += k-1;
                j  += k-1;

                // and reset the kmer
                kmer     = 0u;
                kmer_len = 0u;
            }
        }

        // mark trusted kmers
        int32 zero_cnt = 0;
        int32 one_cnt  = 0;

        // reset the forward iterator
        it = forward_iterator( string.begin() );

        // start with an empty kmer
        kmer     = 0u;
        kmer_len = 0u;

        // keep a k-bits mask of trusted positions
        const uint64 trusted_mask = (uint64(1u) << k) - 1u;
              uint64 trusted      = 0u;

        for (uint32 j = 0; j < len; ++j)
        {
            if (j >= k)
            {
                if (occur[j - k]) --one_cnt;
                else              --zero_cnt;
            }

            if (j < occur_cnt)
            {
                if (occur[j]) ++one_cnt;
                else          ++zero_cnt;
            }

            const int32 sum = one_cnt + zero_cnt;

            //if (qual[j] <= bad_quality)
            //{
            //    trusted[j] = false;
            //    continue ;
            //}

            trusted |= (one_cnt > threshold[sum]) ? 1u : 0u;

            // fetch the next character
            const uint8 c = *it; ++it;

            if (c < 4) // if an N, skip it (the kmers containing it will be marked as untrusted and skipped as well)
            {
                kmer |= c; // insert the new character at the end of the kmer (in a big-endian encoding)

                if (popc( trusted ) == k) // check whether we have an actual 'k'-mer - i.e. k trusted positions in a row
                    trusted_filter.insert( kmer );
            }

            // shift the kmer to the right, dropping the last symbol
            kmer <<= 2;
            kmer &= kmask;

            // shift the trusted bits by one to the right, dropping the last symbol
            trusted <<= 1;
            trusted &= trusted_mask;
        }
    }

    const uint32                k;
    const uint64                kmask;
    string_set_type             string_set;
    const sampled_filter_type   sampled_filter;
    mutable trusted_filter_type trusted_filter;
    const threshold_type        threshold;
};

// process the next batch
//
bool TrustedKmersStage::process(PipelineContext& context)
{
    typedef nvbio::io::SequenceDataAccess<DNA_N>::sequence_string_set_type string_set_type;

    // fetch the input
    nvbio::io::SequenceDataHost* h_read_data = context.input<nvbio::io::SequenceDataHost>( 0 );

    float time = 0.0f;

    // introduce a timing scope
    try
    {
        const nvbio::ScopedTimer<float> timer( &time );

        if (device >= 0)
        {
            //
            // Device (GPU) path
            //

            // declare the Bloom filter types
            typedef nvbio::blocked_bloom_filter<hash_functor1, hash_functor2, nvbio::cuda::ldg_pointer<uint4> > sampled_filter_type;
            typedef nvbio::blocked_bloom_filter<hash_functor1, hash_functor2, uint64_2*>                        trusted_filter_type;

            typedef TrustedKmersFunctor<string_set_type,sampled_filter_type,trusted_filter_type, cuda::ldg_pointer<uint32> > functor_type;

            // set the device
            hipSetDevice( device );

            // copy it to the device
            io::SequenceDataDevice d_read_data( *h_read_data );

            // build a view
            const io::SequenceDataAccess<DNA_N> d_read_view( d_read_data );

            // build the Bloom filter
            sampled_filter_type sampled_filter( SAMPLED_KMERS_FILTER_K, sampled_filter_size, (const uint4*)sampled_filter_storage );
            trusted_filter_type trusted_filter( TRUSTED_KMERS_FILTER_K, trusted_filter_size,    (uint64_2*)trusted_filter_storage );

            // build the kmer sampling functor
            const functor_type kmer_filter(
                k,
                d_read_view.sequence_string_set(),
                sampled_filter,
                trusted_filter,
                cuda::make_ldg_pointer(threshold) );

            // and apply the functor to all reads in the batch
            device_for_each(
                d_read_view.size(),
                kmer_filter );

            hipDeviceSynchronize();
            cuda::check_error("mark-trusted-kmers");
        }
        else
        {
            //
            // Host (CPU) path
            //

            omp_set_num_threads( -device );

            // declare the Bloom filter types
            typedef nvbio::blocked_bloom_filter<hash_functor1, hash_functor2, const uint64_2*>         sampled_filter_type;
            typedef nvbio::blocked_bloom_filter<hash_functor1, hash_functor2,       uint64_2*>         trusted_filter_type;

            typedef TrustedKmersFunctor<string_set_type,sampled_filter_type,trusted_filter_type,const uint32*> functor_type;

            // build a view
            const nvbio::io::SequenceDataAccess<DNA_N> h_read_view( *h_read_data );

            // build the Bloom filter
            sampled_filter_type sampled_filter( SAMPLED_KMERS_FILTER_K, sampled_filter_size, (const uint64_2*)sampled_filter_storage );
            trusted_filter_type trusted_filter( TRUSTED_KMERS_FILTER_K, trusted_filter_size,       (uint64_2*)trusted_filter_storage );

            // build the kmer sampling functor
            const TrustedKmersFunctor<string_set_type,sampled_filter_type,trusted_filter_type,const uint32*> kmer_filter(
                k,
                h_read_view.sequence_string_set(),
                sampled_filter,
                trusted_filter,
                threshold );

            // and apply the functor to all reads in the batch
            host_for_each(
                h_read_view.size(),
                kmer_filter );
        }
    }
    catch (nvbio::cuda_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a nvbio::cuda_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::bad_alloc e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a nvbio::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::logic_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a nvbio::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (nvbio::runtime_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a nvbio::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (thrust::system::system_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a thrust::system_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::bad_alloc e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a std::bad_alloc exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::logic_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a std::logic_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (std::runtime_error e)
    {
        log_error(stderr, "[TrustedKmersStage] caught a std::runtime_error exception:\n");
        log_error(stderr, "  %s\n", e.what());
        exit(1);
    }
    catch (...)
    {
        log_error(stderr, "[TrustedKmersStage] caught an unknown exception!\n");
        exit(1);
    }

    // update the time stats
    stats->m_mutex.lock();
    stats->m_time += time;

    log_info(stderr, "\r  processed reads [%llu, %llu] (%.1fM / %.2fG bps, %.1fK reads/s, %.1fM bps/s - %s<%d>)        ",
        stats->m_reads,
        stats->m_reads + h_read_data->size(),
        1.0e-6f * (h_read_data->bps()),
        1.0e-9f * (stats->m_bps + h_read_data->bps()),
        stats->m_time ? (1.0e-3f * (stats->m_reads + h_read_data->size())) / stats->m_time : 0.0f,
        stats->m_time ? (1.0e-6f * (stats->m_bps   + h_read_data->bps() )) / stats->m_time : 0.0f,
        device >= 0 ? "gpu" : "cpu",
        device >= 0 ? device : -device );

    log_debug_cont(stderr, "\n");
    log_debug(stderr,"  peak memory : %.1f GB\n", float( peak_resident_memory() ) / float(1024*1024*1024));

    stats->m_reads += h_read_data->size();
    stats->m_bps   += h_read_data->bps();
    stats->m_mutex.unlock();
    return true;
}
