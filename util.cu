#include <stdio.h>

#include "util.h"

// pick the best GPU to run on
void gpu_init(void)
{
    int device_count = 0;
    hipError_t err;

    hipSetDeviceFlags(hipDeviceMapHost | hipDeviceLmemResizeToMax);

    err = hipGetDeviceCount(&device_count);
    if (err != hipSuccess)
    {
        fprintf(stderr, "gpu_init: error enumerating GPUs (%d)\n", err);
        exit(1);
    }

    if (device_count == 0)
    {
        fprintf(stderr, "gpu_init: no devices found\n");
        exit(1);
    }

    // pick the best device to run on
    hipDeviceProp_t best_dev_prop;
    int best_dev = 0;
    int dev;

    hipGetDeviceProperties(&best_dev_prop, 0);

    for(dev = 0; dev < device_count; dev++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev);

        if (prop.major >= best_dev_prop.major &&
            prop.minor >= best_dev_prop.minor)
        {
            best_dev_prop = prop;
            best_dev = dev;
        }
    }

    hipSetDevice(best_dev);
    fprintf(stderr, "Running on %s (%d MB)\n", best_dev_prop.name, best_dev_prop.totalGlobalMem / 1024 / 1024);
}
