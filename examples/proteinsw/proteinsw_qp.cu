#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// seeding.cu
//

#include <nvbio/basic/console.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/strings/string.h>
#include <nvbio/strings/alphabet.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <thrust/sequence.h>
#include <stdio.h>
#include <stdlib.h>

using namespace nvbio;

int8 s_blosum62[] =
{
 4,  0, -2, -1, -2,  0, -2, -1, -1, -1, -1, -2, -4, -1, -1, -1,  1,  0,  0, -3, -2, -2, -1,  0,
 0,  9, -3, -4, -2, -3, -3, -1, -3, -1, -1, -3, -4, -3, -3, -3, -1, -1, -1, -2, -2, -3, -3, -2,
-2, -3,  6,  2, -3, -1, -1, -3, -1, -4, -3,  1, -4, -1,  0, -2,  0, -1, -3, -4, -3,  4,  1, -1,
-1, -4,  2,  5, -3, -2,  0, -3,  1, -3, -2,  0, -4, -1,  2,  0,  0, -1, -2, -3, -2,  1,  4, -1,
-2, -2, -3, -3,  6, -3, -1,  0, -3,  0,  0, -3, -4, -4, -3, -3, -2, -2, -1,  1,  3, -3, -3, -1,
 0, -3, -1, -2, -3,  6, -2, -4, -2, -4, -3,  0, -4, -2, -2, -2,  0, -2, -3, -2, -3, -1, -2, -1,
-2, -3, -1,  0, -1, -2,  8, -3, -1, -3, -2,  1, -4, -2,  0,  0, -1, -2, -3, -2,  2,  0,  0, -1,
-1, -1, -3, -3,  0, -4, -3,  4, -3,  2,  1, -3, -4, -3, -3, -3, -2, -1,  3, -3, -1, -3, -3, -1,
-1, -3, -1,  1, -3, -2, -1, -3,  5, -2, -1,  0, -4, -1,  1,  2,  0, -1, -2, -3, -2,  0,  1, -1,
-1, -1, -4, -3,  0, -4, -3,  2, -2,  4,  2, -3, -4, -3, -2, -2, -2, -1,  1, -2, -1, -4, -3, -1,
-1, -1, -3, -2,  0, -3, -2,  1, -1,  2,  5, -2, -4, -2,  0, -1, -1, -1,  1, -1, -1, -3, -1, -1,
-2, -3,  1,  0, -3,  0,  1, -3,  0, -3, -2,  6, -4, -2,  0,  0,  1,  0, -3, -4, -2,  3,  0, -1,
-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,
-1, -3, -1, -1, -4, -2, -2, -3, -1, -3, -2, -2, -4,  7, -1, -2, -1, -1, -2, -4, -3, -2, -1, -2,
-1, -3,  0,  2, -3, -2,  0, -3,  1, -2,  0,  0, -4, -1,  5,  1,  0, -1, -2, -2, -1,  0,  3, -1,
-1, -3, -2,  0, -3, -2,  0, -3,  2, -2, -1,  0, -4, -2,  1,  5, -1, -1, -3, -3, -2, -1,  0, -1,
 1, -1,  0,  0, -2,  0, -1, -2,  0, -2, -1,  1, -4, -1,  0, -1,  4,  1, -2, -3, -2,  0,  0,  0,
 0, -1, -1, -1, -2, -2, -2, -1, -1, -1, -1,  0, -4, -1, -1, -1,  1,  5,  0, -2, -2, -1, -1,  0,
 0, -1, -3, -2, -1, -3, -3,  3, -2,  1,  1, -3, -4, -2, -2, -3, -2,  0,  4, -3, -1, -3, -2, -1,
-3, -2, -4, -3,  1, -2, -2, -3, -3, -2, -1, -4, -4, -4, -2, -3, -3, -2, -3, 11,  2, -4, -3, -2,
-2, -2, -3, -2,  3, -3,  2, -1, -2, -1, -1, -2, -4, -3, -1, -2, -2, -2, -1,  2,  7, -3, -2, -1,
-2, -3,  4,  1, -3, -1,  0, -3,  0, -4, -3,  3, -4, -2,  0, -1,  0, -1, -3, -4, -3,  4,  1, -1,
-1, -3,  1,  4, -3, -2,  0, -3,  1, -3, -1,  0, -4, -1,  3,  0,  0, -1, -2, -3, -2,  1,  4, -1,
 0, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -4, -2, -1, -1,  0,  0, -1, -2, -1, -1, -1, -1,
};

template <typename matrix_iterator>
struct BlosumScheme
{
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE BlosumScheme(const matrix_iterator m, const int32 gap_open, const int32 gap_ext) :
        m_matrix(m), m_gap_open(gap_open), m_gap_ext(gap_ext) {}

    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 match(const uint8 q = 0)      const { return 11; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 substitution(const uint32 r_i, const uint32 q_j, const uint8 r, const uint8 q, const uint8 qq = 0)   const { return int8( m_matrix[ r + q*24 ] ); };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_open()            const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_extension()       const { return m_gap_ext; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_open()               const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_extension()          const { return m_gap_ext; };

    const matrix_iterator   m_matrix;
    const int32             m_gap_open;
    const int32             m_gap_ext;
};

template <typename matrix_iterator>
struct QueryProfile
{
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE QueryProfile(const matrix_iterator m, const int32 gap_open, const int32 gap_ext) :
        m_matrix(m), m_gap_open(gap_open), m_gap_ext(gap_ext) {}

    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 match(const uint8 q = 0)      const { return 11; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 substitution(const uint32 r_i, const uint32 q_j, const uint8 r, const uint8 q, const uint8 qq = 0)   const { return int8( m_matrix[ r + q_j*24 ] ); };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_open()            const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 pattern_gap_extension()       const { return m_gap_ext; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_open()               const { return m_gap_open; };
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE int32 text_gap_extension()          const { return m_gap_ext; };

    const matrix_iterator   m_matrix;
    const int32             m_gap_open;
    const int32             m_gap_ext;
};

// main test entry point
//
int main(int argc, char* argv[])
{
    log_info(stderr, "protein SW... started\n");
    const uint32 n_tests   = 10;
    const uint32 n_strings = 50000;
    const uint32 P         = 100;
    const uint32 T         = 1000;

    // alloc a device vector for holding the BLOSUM62 scoring matrix
    nvbio::vector<device_tag,uint8> d_blosum62( 24*24 );

    // copy the matrix to the device
    thrust::copy( s_blosum62, s_blosum62 + 24*24, d_blosum62.begin() );

    // alloc the storage for the host strings
    nvbio::vector<host_tag,uint8>  h_pattern_strings( P * n_strings );
    nvbio::vector<host_tag,uint8>  h_pattern_profiles( P * n_strings * 24 );
    nvbio::vector<host_tag,uint8>  h_text_strings( T * n_strings );

    // fill the strings with random characters
    LCG_random rand;
    for (uint32 i = 0; i < n_strings; ++i)
    {
        for (uint32 j = 0; j < P; ++j)
        {
            h_pattern_strings[i*P + j] = nvbio::min( uint8( rand.next() * 24.0f ), (uint8)23u );

            const uint8 q = h_pattern_strings[i*P + j];
            for (uint32 r = 0; r < 24; ++r)
                h_pattern_profiles[i*P + j*24 + r] = s_blosum62[ r + q*24 ];
        }
    }
    for (uint32 i = 0; i < T * n_strings; ++i)
        h_text_strings[i] = nvbio::min( uint8( rand.next() * 24.0f ), (uint8)23u );

    // copy to the device
    nvbio::vector<device_tag,uint8>  d_pattern_strings( h_pattern_strings );
    nvbio::vector<device_tag,uint8>  d_pattern_profiles( h_pattern_profiles );
    nvbio::vector<device_tag,uint8>  d_text_strings( h_text_strings );
    nvbio::vector<device_tag,uint32> d_pattern_offsets( n_strings + 1 );
    nvbio::vector<device_tag,uint32> d_text_offsets( n_strings + 1 );

    // build the string offsets
    thrust::sequence( d_pattern_offsets.begin(), d_pattern_offsets.end(), 0u, P );
    thrust::sequence( d_text_offsets.begin(),    d_text_offsets.end(),    0u, T );

    // prepare a vector of alignment sinks
    nvbio::vector< device_tag, aln::BestSink< uint32 > > sinks( n_strings );

    {
        const aln::SimpleGotohScheme scoring( 1, -1, -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadScheduler() );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Constant): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }
    {
        const BlosumScheme< cuda::ldg_pointer<uint8> > scoring( cuda::make_ldg_pointer( raw_pointer( d_blosum62 ) ), -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_pattern_strings ), (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadBlockScheduler<128,1>() );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Blosum62): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }
    {
        //const QueryProfile< cuda::ldg_pointer<uint8> > scoring( cuda::make_ldg_pointer( raw_pointer( d_pattern_profiles ) ), -5, -3 );
        const QueryProfile<const uint8*> scoring( raw_pointer( d_pattern_profiles ), -5, -3 );

        Timer timer;
        timer.start();

        for (uint32 i = 0; i < n_tests; ++i)
        {
            // and execute the batch alignment, on a GPU device
            aln::batch_alignment_score(
                aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                make_concatenated_string_set( n_strings, thrust::make_constant_iterator<uint32>(0),      (const uint32*)raw_pointer( d_pattern_offsets ) ),
                make_concatenated_string_set( n_strings, (const uint8*)raw_pointer( d_text_strings ),    (const uint32*)raw_pointer( d_text_offsets ) ),
                sinks.begin(),
                aln::DeviceThreadBlockScheduler<128,1>() );
        }
        hipDeviceSynchronize();

        timer.stop();
        log_info(stderr, "  GCUPS (Blosum62): %.1f\n", (1.0e-9f * float(P*T) * float(n_strings) * float(n_tests))/timer.seconds());
    }

    log_info(stderr, "protein SW... done\n");
    return 0;
}
