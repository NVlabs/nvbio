#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <nvBowtie/bowtie2/cuda/mapping.h>
#include <nvBowtie/bowtie2/cuda/mapping_impl.h>

namespace nvbio {
namespace bowtie2 {
namespace cuda {

//
// For all i in [0, #seed hit ranges[, output the seed hit range size in
// out_ranges[i].
//
__global__ 
void gather_ranges_kernel(
    const uint32                        count,
    const uint32                        n_reads,
    const SeedHitDequeArrayDeviceView   hits,
    const uint32*                       hit_counts_scan,
          uint64*                       out_ranges)
{
    const uint32 thread_id = threadIdx.x + BLOCKDIM*blockIdx.x;
    if (thread_id >= count) return;

    // do a binary search, looking for thread_id in hit_counts_scan,
    // to find the corresponding read id.
    const uint32 read_id = upper_bound_index( thread_id, hit_counts_scan, n_reads );

    // at this point we can figure out which seed hit / SA range this thread is
    // responsible of
    const uint32 count_offset = read_id ? hit_counts_scan[read_id-1] : 0u;

    const uint32 range_id = thread_id - count_offset;

    const SeedHit* hits_data = hits.get_data( read_id );

    const uint2 range = hits_data[ range_id ].get_range();
 
    // and we can compute the corresponding range size
    out_ranges[ thread_id ] = range.y - range.x;
}

//
// dispatch the call to gather_ranges_kernel
//
void gather_ranges(
    const uint32                        count,
    const uint32                        n_reads,
    const SeedHitDequeArrayDeviceView   hits,
    const uint32*                       hit_counts_scan,
          uint64*                       out_ranges)
{
    const int blocks = (count + BLOCKDIM-1) / BLOCKDIM;

    gather_ranges_kernel<<<blocks, BLOCKDIM>>>( count, n_reads, hits, hit_counts_scan, out_ranges );
}


//
// perform exact read mapping
//
void map_whole_read(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    uint8*                                          reseed,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_whole_read_t( read_batch, fmi, rfmi, queues, reseed, hits, params, fw, rc );
}

//
// perform one run of exact seed mapping for all the reads in the input queue,
// writing reads that need another run in the output queue
//
void map_exact(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    uint8*                                          reseed,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_exact_t( read_batch, fmi, rfmi, retry, queues, reseed, hits, params, fw, rc );
}

//
// perform multiple runs of exact seed mapping in one go and keep the best
//
void map_exact(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    SeedHitDequeArrayDeviceView                     hits,
    const uint2                                     seed_range,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_exact_t( read_batch, fmi, rfmi, hits, seed_range, params, fw, rc );
}

//
// perform one run of approximate seed mapping for all the reads in the input queue,
// writing reads that need another run in the output queue
//
void map_approx(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    uint8*                                          reseed,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_approx_t( read_batch, fmi, rfmi, retry, queues, reseed, hits, params, fw, rc );
}

//
// perform multiple runs of approximate seed mapping in one go and keep the best
//
void map_approx(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    SeedHitDequeArrayDeviceView                     hits,
    const uint2                                     seed_range,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_approx_t( read_batch, fmi, rfmi, hits, seed_range, params, fw, rc );
}

//
// perform one run of seed mapping
//
void map(
    const ReadsDef::type&                           read_batch,
    const FMIndexDef::type                          fmi,
    const FMIndexDef::type                          rfmi,
    const uint32                                    retry,
    const nvbio::cuda::PingPongQueuesView<uint32>   queues,
    uint8*                                          reseed,
    SeedHitDequeArrayDeviceView                     hits,
    const ParamsPOD                                 params,
    const bool                                      fw,
    const bool                                      rc)
{
    map_t( read_batch, fmi, rfmi, retry, queues, reseed, hits, params, fw, rc );
}

} // namespace cuda
} // namespace bowtie2
} // namespace nvbio
