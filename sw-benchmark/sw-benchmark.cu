#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// sw-benchmark.cu
//

//
// This project supports comparisons against multi-core SSE-enabled CPUs using
// conditional compilation of the SSW library:
//
//  https://github.com/mengyao/Complete-Striped-Smith-Waterman-Library
//
// In order to perform these additional tests, the user must download ssw.h and ssw.c
// from the above repository, copy them in the sw-benchmark directory, and run cmake with
// the option -DSSWLIB=ON.
//
#if defined(SSWLIB)
#include "ssw.h"
#include <omp.h>
#endif

#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/cuda/ldg.h>
#include <nvbio/basic/packedstream.h>
#include <nvbio/basic/packedstream_loader.h>
#include <nvbio/basic/vector_view.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/io/sequence/sequence.h>
#include <nvbio/fasta/fasta.h>
#include <nvbio/basic/dna.h>
#include <nvbio/alignment/alignment.h>
#include <nvbio/alignment/batched.h>
#include <nvbio/alignment/sink.h>
#include <thrust/device_vector.h>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

enum { MAX_READ_LENGTH = 1024 };

using namespace nvbio;

enum { CACHE_SIZE = 64 };
typedef nvbio::lmem_cache_tag<CACHE_SIZE>                                       lmem_cache_tag_type;
typedef nvbio::uncached_tag                                                     uncached_tag_type;

enum { REF_BITS       = 2 };
enum { REF_BIG_ENDIAN = false };

//
// An alignment stream class to be used in conjunction with the BatchAlignmentScore class
//
template <typename t_aligner_type, typename cache_type = lmem_cache_tag_type>
struct AlignmentStream
{
    typedef t_aligner_type                                                          aligner_type;

    typedef nvbio::cuda::ldg_pointer<uint32>                                        storage_iterator;

    typedef nvbio::PackedStringLoader<
        storage_iterator,
        io::SequenceDataTraits<DNA_N>::SEQUENCE_BITS,
        io::SequenceDataTraits<DNA_N>::SEQUENCE_BIG_ENDIAN,cache_type>                          pattern_loader_type;
    typedef typename pattern_loader_type::input_iterator                                        uncached_pattern_iterator;
    typedef typename pattern_loader_type::iterator                                              pattern_iterator;
    typedef nvbio::vector_view<pattern_iterator>                                                pattern_string;

    typedef nvbio::PackedStringLoader<
        storage_iterator,
        REF_BITS,
        REF_BIG_ENDIAN,uncached_tag_type>                                                       text_loader_type;
    typedef typename text_loader_type::input_iterator                                           uncached_text_iterator;
    typedef typename text_loader_type::iterator                                                 text_iterator;
    typedef nvbio::vector_view<text_iterator>                                                   text_string;

    // an alignment context
    struct context_type
    {
        int32                   min_score;
        aln::BestSink<int32>    sink;
    };
    // a container for the strings to be aligned
    struct strings_type
    {
        pattern_loader_type         pattern_loader;
        text_loader_type            text_loader;
        pattern_string              pattern;
        aln::trivial_quality_string quals;
        text_string                 text;
    };

    // constructor
    AlignmentStream(
        aligner_type        _aligner,
        const uint32        _count,
        const uint32*       _offsets,
        const uint32*       _patterns,
        const uint32        _max_pattern_len,
        const uint32        _total_pattern_len,
        const uint32*       _text,
        const uint32        _text_len,
               int16*       _scores) :
        m_aligner           ( _aligner ),
        m_count             (_count),
        m_max_pattern_len   (_max_pattern_len),
        m_total_pattern_len (_total_pattern_len),
        m_text_len          (_text_len),
        m_offsets           (_offsets),
        m_patterns          (storage_iterator(_patterns)),
        m_text              (storage_iterator(_text)),
        m_scores            (_scores) {}

    // get the aligner
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    const aligner_type& aligner() const { return m_aligner; };

    // return the maximum pattern length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_pattern_length() const { return m_max_pattern_len; }

    // return the maximum text length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 max_text_length() const { return m_text_len; }

    // return the stream size
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 size() const { return m_count; }

    // return the i-th pattern's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 pattern_length(const uint32 i, context_type* context) const { return m_offsets[i+1] - m_offsets[i]; }

    // return the i-th text's length
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    uint32 text_length(const uint32 i, context_type* context) const { return m_text_len; }

    // return the total number of cells
    uint64 cells() const { return uint64( m_total_pattern_len ) * uint64( m_text_len ); }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    bool init_context(
        const uint32    i,
        context_type*   context) const
    {
        context->min_score = Field_traits<int32>::min();
        return true;
    }

    // initialize the i-th context
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void load_strings(
        const uint32        i,
        const uint32        window_begin,
        const uint32        window_end,
        const context_type* context,
              strings_type* strings) const
    {
        const uint32 offset = m_offsets[i];
        const uint32 length = m_offsets[i+1] - offset;

        strings->text = text_string( m_text_len,
            strings->text_loader.load(
                m_text,
                m_text_len,
                make_uint2( window_begin, window_end ),
                false ) );

        strings->pattern = pattern_string( length,
            strings->pattern_loader.load( m_patterns + offset, length ) );
    }

    // handle the output
    NVBIO_FORCEINLINE NVBIO_HOST_DEVICE
    void output(
        const uint32        i,
        const context_type* context) const
    {
        // copy the output score
        m_scores[i] = context->sink.score;
    }

    aligner_type                m_aligner;
    uint32                      m_count;
    uint32                      m_max_pattern_len;
    uint32                      m_total_pattern_len;
    uint32                      m_text_len;
    const uint32*               m_offsets;
    uncached_pattern_iterator   m_patterns;
    uncached_text_iterator      m_text;
    int16*                      m_scores;
};

// A simple kernel to test the speed of alignment without the possible overheads of the BatchAlignmentScore interface
//
template <uint32 BLOCKDIM, uint32 MAX_PATTERN_LEN, typename aligner_type, typename score_type>
__global__ void alignment_test_kernel(
    const aligner_type aligner,
    const uint32        N_probs,
    const uint32*       offsets,
    const uint32*       pattern_ptr,
    const uint32        text_len,
    const uint32*       text_ptr,
    score_type*         score)
{
    const uint32 tid = blockIdx.x * BLOCKDIM + threadIdx.x;

    typedef lmem_cache_tag_type                                                 lmem_cache_type;
    typedef nvbio::cuda::ldg_pointer<uint32>                                    storage_iterator;

    typedef nvbio::PackedStringLoader<
        storage_iterator,
        io::SequenceDataTraits<DNA_N>::SEQUENCE_BITS,
        io::SequenceDataTraits<DNA_N>::SEQUENCE_BIG_ENDIAN,lmem_cache_type>                     pattern_loader_type;
    typedef typename pattern_loader_type::input_iterator                                        uncached_pattern_iterator;
    typedef typename pattern_loader_type::iterator                                              pattern_iterator;
    typedef nvbio::vector_view<pattern_iterator>                                                pattern_string;

    typedef nvbio::PackedStringLoader<
        storage_iterator,
        REF_BITS,
        REF_BIG_ENDIAN,uncached_tag_type>                                                       text_loader_type;
    typedef typename text_loader_type::input_iterator                                           uncached_text_iterator;
    typedef typename text_loader_type::iterator                                                 text_iterator;
    typedef nvbio::vector_view<text_iterator>                                                   text_string;

    if (tid >= N_probs)
        return;

    const uint32 pattern_off = offsets[tid];
    const uint32 pattern_len = offsets[tid+1] - pattern_off;

    pattern_loader_type pattern_loader;
    pattern_string pattern = pattern_string( pattern_len, pattern_loader.load( uncached_pattern_iterator( pattern_ptr ) + pattern_off, pattern_len ) );

    text_loader_type text_loader;
    text_string text = text_string( text_len, text_loader.load( uncached_text_iterator( text_ptr ), text_len ) );

    aln::BestSink<int32> sink;

    aln::alignment_score<MAX_PATTERN_LEN>(
        aligner,
        pattern,
        aln::trivial_quality_string(),
        text,
        Field_traits<int32>::min(),
        sink );

    score[tid] = sink.score;
}

unsigned char nst_nt4_table[256] = {
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 5 /*'-'*/, 4, 4,
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 0, 4, 1,  4, 4, 4, 2,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  3, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4, 
	4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4,  4, 4, 4, 4
};

struct ReferenceCounter
{
    ReferenceCounter() : m_size(0) {}

    void begin_read() {}
    void end_read() {}

    void id(const uint8 c) {}
    void read(const uint8 c) { ++m_size; }

    uint32 m_size;
};
struct ReferenceCoder
{
    typedef PackedStream<uint32*,uint8,2,false> stream_type;

    ReferenceCoder(uint32* storage) :
        m_size(0), m_stream( storage )
    {}

    void begin_read() {}
    void end_read() {}

    void id(const uint8 c) {}

    void read(const uint8 s)
    {
        const uint8 c = nst_nt4_table[s];

        m_stream[ m_size++ ] = c < 4 ? c : 0;
    }

    uint32      m_size;
    stream_type m_stream;
};


// execute a given batch alignment type on a given stream
//
// \tparam batch_type               a \ref BatchAlignment "Batch Alignment"
// \tparam stream_type              a stream compatible to the given batch_type
//
// \return                          average time
//
template <typename batch_type, typename stream_type>
float enact_batch(
          batch_type&               batch,
    const stream_type&              stream)
{
    // allow to alloc all the needed temporary storage
    const uint64 temp_size = batch_type::max_temp_storage(
        stream.max_pattern_length(),
        stream.max_text_length(),
        stream.size() );

    Timer timer;
    timer.start();

    // enact the batch
    batch.enact( stream, temp_size, NULL );

    hipDeviceSynchronize();

    timer.stop();

    return timer.seconds();
}

// execute and time a batch of full DP alignments using BatchAlignmentScore
//
template <typename scheduler_type, typename stream_type>
void batch_score_profile(
    const stream_type               stream)
{
    typedef aln::BatchedAlignmentScore<stream_type, scheduler_type> batch_type;  // our batch type

    // setup a batch
    batch_type batch;

    const float time = enact_batch(
        batch,
        stream );

    fprintf(stderr,"  %5.1f", 1.0e-9f * float(stream.cells())/time );
}

// execute and time the batch_score<scheduler> algorithm for all possible schedulers
//
template <typename aligner_type>
void batch_score_profile_all(
    const aligner_type                      aligner,
    const uint32                            n_tasks,
    const uint32*                           offsets_dvec,
    const uint32*                           pattern_dvec,
    const uint32                            max_pattern_len,
    const uint32                            total_pattern_len,
    const uint32*                           text_dvec,
    const uint32                            text_len,
    int16*                                  score_dvec)
{
    {
        typedef AlignmentStream<aligner_type> stream_type;

        // create a stream
        stream_type stream(
            aligner,
            n_tasks,
            offsets_dvec,
            pattern_dvec,
            max_pattern_len,
            total_pattern_len,
            text_dvec,
            text_len,
            score_dvec );

        // test the DeviceThreadScheduler
        batch_score_profile<aln::DeviceThreadScheduler>( stream );

        // test the DeviceStagedThreadScheduler
        //batch_score_profile<aln::DeviceStagedThreadScheduler>( stream );
    }
    {
        const uint32 BLOCKDIM = 128;
        const uint32 N_BLOCKS = (n_tasks + BLOCKDIM-1) / BLOCKDIM;

        Timer timer;
        timer.start();

        // enact the batch
        alignment_test_kernel<BLOCKDIM,MAX_READ_LENGTH> <<<N_BLOCKS,BLOCKDIM>>>(
            aligner,
            n_tasks,
            offsets_dvec,
            pattern_dvec,
            text_len,
            text_dvec,
            score_dvec );

        hipDeviceSynchronize();

        timer.stop();

        const float time = timer.seconds();

        fprintf(stderr,"  %5.1f", 1.0e-9f * float(uint64(total_pattern_len)*uint64(text_len))/time );
    }
    fprintf(stderr, " GCUPS\n");
}

enum AlignmentTest
{
    ALL                 = 0xFFFFFFFFu,
    ED                  = 1u,
    SW                  = 2u,
    GOTOH               = 4u,
    ED_BANDED           = 8u,
    SW_BANDED           = 16u,
    GOTOH_BANDED        = 32u,
    SSW                 = 64u
};

int main(int argc, char* argv[])
{
    uint32 TEST_MASK        = 0xFFFFFFFFu;

    const char* reads_name  = argv[argc-2];
    const char* ref_name    = argv[argc-1];
    uint32      threads     = omp_get_num_procs();
    io::QualityEncoding qencoding = io::Phred33;

    for (int i = 0; i < argc-2; ++i)
    {
        if (strcmp( argv[i], "-tests" ) == 0)
        {
            const std::string tests_string( argv[++i] );

            char temp[256];
            const char* begin = tests_string.c_str();
            const char* end   = begin;

            TEST_MASK = 0u;

            while (1)
            {
                while (*end != ':' && *end != '\0')
                {
                    temp[end - begin] = *end;
                    end++;
                }

                temp[end - begin] = '\0';

                if (strcmp( temp, "ed" ) == 0)
                    TEST_MASK |= ED;
                else if (strcmp( temp, "sw" ) == 0)
                    TEST_MASK |= SW;
                else if (strcmp( temp, "gotoh" ) == 0)
                    TEST_MASK |= GOTOH;
                else if (strcmp( temp, "ssw" ) == 0)
                    TEST_MASK |= SSW;

                if (*end == '\0')
                    break;

                ++end; begin = end;
            }
        }
        else if (strcmp( argv[i], "-threads" ) == 0)
            threads = atoi( argv[++i] );
    }

    fprintf(stderr,"sw-benchmark... started\n");

    log_visible(stderr, "opening read file \"%s\"\n", reads_name);
    SharedPointer<nvbio::io::SequenceDataStream> read_data_file(
        nvbio::io::open_sequence_file(reads_name,
                                  qencoding)
    );

    log_visible(stderr, "reading reference file \"%s\"... started\n", ref_name);

    // read the reference
    thrust::host_vector<uint32> h_ref_storage;
    uint32                      ref_length;
    uint32                      ref_words;
    {
        ReferenceCounter counter;

        FASTA_inc_reader fasta( ref_name );
        if (fasta.valid() == false)
        {
            fprintf(stderr, "  error: unable to open reference file \"%s\"\n", ref_name);
            exit(1);
        }
        while (fasta.read( 1024, counter ) == 1024);

        ref_length = counter.m_size;
        ref_words  = (ref_length + 15)/16; // # of words at 2 bits per symbol
    }
    {
        h_ref_storage.resize( ref_words );
        ReferenceCoder coder( &h_ref_storage[0] );

        FASTA_inc_reader fasta( ref_name );
        if (fasta.valid() == false)
        {
            fprintf(stderr, "  error: unable to open reference file \"%s\"\n", ref_name);
            exit(1);
        }
        while (fasta.read( 1024, coder ) == 1024);
    }
    log_visible(stderr, "reading reference file \"%s\"... done (%u bps)\n", ref_name, ref_length);

    typedef PackedStream<uint32*,uint8,REF_BITS,REF_BIG_ENDIAN> ref_stream_type;

    thrust::device_vector<uint32> d_ref_storage( h_ref_storage );
    ref_stream_type d_ref_stream( nvbio::raw_pointer( d_ref_storage ) );

    const uint32 batch_size = 256*1024;

    thrust::device_vector<int16> score_dvec( batch_size, 0 );

  #if defined(SSWLIB)
    std::vector<int8_t> unpacked_ref( ref_length );
    {
        ref_stream_type h_ref_stream( nvbio::raw_pointer( h_ref_storage ) );
        for (uint32 i = 0; i < ref_length; ++i)
            unpacked_ref[i] = h_ref_stream[i];
    }

    // Now set the number of threads
    omp_set_num_threads( threads );

    #pragma omp parallel
    {
        fprintf(stderr, "  running on multiple threads\n");
    }
  #endif

    io::SequenceDataHost h_read_data;

    while (io::next( DNA_N, &h_read_data, read_data_file.get(), batch_size ))
    {
        // build the device side representation
        const io::SequenceDataDevice d_read_data( h_read_data );

        const uint32 n_read_symbols = h_read_data.bps();

        fprintf(stderr,"  %u reads, avg: %u bps, max: %u bps\n",
            h_read_data.size(),
            h_read_data.avg_sequence_len(),
            h_read_data.max_sequence_len());

        if (TEST_MASK & GOTOH)
        {
            aln::SimpleGotohScheme scoring;
            scoring.m_match    =  2;
            scoring.m_mismatch = -1;
            scoring.m_gap_open = -2;
            scoring.m_gap_ext  = -1;

            fprintf(stderr,"  testing Gotoh scoring speed...\n");
            fprintf(stderr,"    %15s : ", "global");
            {
                batch_score_profile_all(
                    aln::make_gotoh_aligner<aln::GLOBAL,aln::TextBlockingTag>( scoring ),
                    d_read_data.size(),
                    nvbio::plain_view( d_read_data ).sequence_index(),
                    nvbio::plain_view( d_read_data ).sequence_storage(),
                    d_read_data.max_sequence_len(),
                    n_read_symbols,
                    nvbio::raw_pointer( d_ref_storage ),
                    ref_length,
                    nvbio::raw_pointer( score_dvec ) );
            }

            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all(
                    aln::make_gotoh_aligner<aln::SEMI_GLOBAL,aln::TextBlockingTag>( scoring ),
                    d_read_data.size(),
                    nvbio::plain_view( d_read_data ).sequence_index(),
                    nvbio::plain_view( d_read_data ).sequence_storage(),
                    d_read_data.max_sequence_len(),
                    n_read_symbols,
                    nvbio::raw_pointer( d_ref_storage ),
                    ref_length,
                    nvbio::raw_pointer( score_dvec ) );
            }
            fprintf(stderr,"    %15s : ", "local");
            {
                batch_score_profile_all(
                    aln::make_gotoh_aligner<aln::LOCAL,aln::TextBlockingTag>( scoring ),
                    d_read_data.size(),
                    nvbio::plain_view( d_read_data ).sequence_index(),
                    nvbio::plain_view( d_read_data ).sequence_storage(),
                    d_read_data.max_sequence_len(),
                    n_read_symbols,
                    nvbio::raw_pointer( d_ref_storage ),
                    ref_length,
                    nvbio::raw_pointer( score_dvec ) );
            }
        }
        if (TEST_MASK & ED)
        {
            fprintf(stderr,"  testing Edit Distance scoring speed...\n");
            fprintf(stderr,"    %15s : ", "semi-global");
            {
                batch_score_profile_all(
                    aln::make_edit_distance_aligner<aln::SEMI_GLOBAL,aln::TextBlockingTag>(),
                    d_read_data.size(),
                    nvbio::plain_view( d_read_data ).sequence_index(),
                    nvbio::plain_view( d_read_data ).sequence_storage(),
                    d_read_data.max_sequence_len(),
                    n_read_symbols,
                    nvbio::raw_pointer( d_ref_storage ),
                    ref_length,
                    nvbio::raw_pointer( score_dvec ) );
            }
        }

        #if defined(SSWLIB)
        if (TEST_MASK & SSW)
        {
            fprintf(stderr,"  testing SSW scoring speed...\n");
            fprintf(stderr,"    %15s : ", "local");

            const int8_t mat[4*4] = {2, -1, -1, -1, -1, 2, -1, -1, -1, -1, 2, -1, -1, -1, -1, 2};

            std::vector<int8_t> unpacked_reads( n_read_symbols );

            typedef io::SequenceDataAccess<DNA_N>           read_access_type;
            typedef read_access_type::sequence_stream_type  read_stream_type;

            const read_access_type reads_access( h_read_data );

            const read_stream_type packed_reads( reads_access.sequence_stream() );

            #pragma omp parallel for
            for (int i = 0; i < int( n_read_symbols ); ++i)
                unpacked_reads[i] = packed_reads[i];

            Timer timer;
            timer.start();

            #pragma omp parallel for
            for (int i = 0; i < int( h_read_data.size() ); ++i)
            {
                const uint32 read_off = reads_access.sequence_index()[i];
                const uint32 read_len = reads_access.sequence_index()[i+1] - read_off;

                s_profile* prof = ssw_init( &unpacked_reads[read_off], read_len, mat, 4, 2 );

                s_align* align = ssw_align(
                    prof, 
					&unpacked_ref[0], 
					ref_length, 
					2, 
					2,
					0u,	
					0u,
					0,
					15 );

                align_destroy( align );

                init_destroy( prof );
            }

            timer.stop();
            const float time = timer.seconds();

            fprintf(stderr,"  %5.1f", 1.0e-9f * float(uint64(n_read_symbols)*uint64(ref_length))/time );
            fprintf(stderr, " GCUPS\n");
        }
        #endif
    }
    fprintf(stderr,"sw-benchmark... done\n");
    return 0;
}
