#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// qmap.cu
//

#include <stdio.h>
#include <stdlib.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/vector.h>
#include <nvbio/basic/shared_pointer.h>
#include <nvbio/basic/dna.h>
#include <nvbio/strings/string_set.h>
#include <nvbio/strings/infix.h>
#include <nvbio/strings/seeds.h>
#include <nvbio/qgram/qgram.h>
#include <nvbio/qgram/filter.h>
#include <nvbio/io/sequence/sequence.h>

#include "alignment.h"
#include "util.h"

using namespace nvbio;

// query stats
//
struct Stats
{
    Stats() :
        time(0),
        build_time(0),
        extract_time(0),
        rank_time(0),
        locate_time(0),
        align_time(0),
        reads(0),
        aligned(0),
        queries(0),
        matches(0),
        occurrences(0),
        merged(0) {}

    float   time;
    float   build_time;
    float   extract_time;
    float   rank_time;
    float   locate_time;
    float   align_time;
    uint64  reads;
    uint64  aligned;
    uint64  queries;
    uint64  matches;
    uint64  occurrences;
    uint64  merged;
};

// build a set of q-grams from a given string, together with their sorted counterpart
//
template <typename genome_string, typename qgram_vector_type, typename index_vector_type>
void build_qgrams(
    const uint32                    Q,
    const uint32                    genome_len,
    const uint32                    genome_offset,
    const genome_string             genome,
    const uint32                    n_queries,
    qgram_vector_type&              qgrams,
    qgram_vector_type&              sorted_qgrams,
    index_vector_type&              sorted_indices)
{
    // build the q-grams
    qgrams.resize( n_queries );
    generate_qgrams( Q, 2u, genome_len, genome, n_queries, thrust::make_counting_iterator<uint32>(genome_offset), qgrams.begin() );

    // sort the q-grams
    sorted_qgrams = qgrams;
    sorted_indices.resize( n_queries );
    thrust::copy(
        thrust::make_counting_iterator<uint32>(genome_offset),
        thrust::make_counting_iterator<uint32>(genome_offset) + n_queries,
        sorted_indices.begin() );

    thrust::sort_by_key( sorted_qgrams.begin(), sorted_qgrams.end(), sorted_indices.begin() );
}

// build a q-gram set-index from a string-set
//
template <typename string_set_type>
void qgram_set_index_build(
    const uint32            Q,
    const uint32            seed_interval,
    const string_set_type   string_set,
    QGramSetIndexDevice&    qgram_index)
{
    log_verbose(stderr, "  building q-gram set-index... started\n");

    Timer timer;
    timer.start();

    // build the q-gram set index
    qgram_index.build(
        Q,              // q-gram size
        2u,             // implicitly convert N to A
        string_set,
        uniform_seeds_functor<>( Q, seed_interval ),
        12u );

    hipDeviceSynchronize();
    timer.stop();
    const float time = timer.seconds();

    log_verbose(stderr, "  building q-gram set-index... done\n");
    log_verbose(stderr, "    indexed q-grams : %6.2f M q-grams\n", 1.0e-6f * float( qgram_index.n_qgrams ));
    log_verbose(stderr, "    unique q-grams  : %6.2f M q-grams\n", 1.0e-6f * float( qgram_index.n_unique_qgrams ));
    log_verbose(stderr, "    throughput      : %5.1f M q-grams/s\n", 1.0e-6f * float( qgram_index.n_qgrams ) / time);
    log_verbose(stderr, "    memory usage    : %5.1f MB\n", float( qgram_index.used_device_memory() ) / float(1024*1024) );
}

// perform q-gram index mapping
//
template <typename qgram_index_type, typename qgram_filter_type, typename genome_string>
void map(
          qgram_index_type&                 qgram_index,
          qgram_filter_type&                qgram_filter,
    const uint32                            merge_intv,
    const io::SequenceDataDevice&           reads,
    const uint32                            n_queries,
    const uint32                            genome_len,
    const uint32                            genome_offset,
    const genome_string                     genome,
    nvbio::vector<device_tag,int16>&        best_scores,
          Stats&                            stats)
{
    typedef typename qgram_index_type::system_tag system_tag;

    // prepare some vectors to store the query qgrams
    nvbio::vector<system_tag,uint64>  qgrams( n_queries );
    nvbio::vector<system_tag,uint64>  sorted_qgrams( n_queries );
    nvbio::vector<system_tag,uint32>  sorted_indices( n_queries );

    const uint32 Q = qgram_index.Q;

    Timer timer;
    timer.start();

    build_qgrams(
        Q,
        genome_len,
        genome_offset,
        genome,
        n_queries,
        qgrams,
        sorted_qgrams,
        sorted_indices );

    hipDeviceSynchronize();
    timer.stop();
    const float extract_time = timer.seconds();

    stats.queries       += n_queries;
    stats.extract_time  += extract_time;

    //
    // search the sorted query q-grams with a q-gram filter
    //

    const uint32 batch_size = 32*1024*1024;

    typedef typename qgram_filter_type::hit_type        hit_type;
    typedef typename qgram_filter_type::diagonal_type   diagonal_type;

    // prepare storage for the output hits
    nvbio::vector<system_tag,hit_type>      hits( batch_size );
    nvbio::vector<system_tag,diagonal_type> merged_hits( batch_size );
    nvbio::vector<system_tag,uint16>        merged_counts( batch_size );
    nvbio::vector<system_tag,int16>         scores( batch_size );
    nvbio::vector<system_tag,uint32>        out_reads( batch_size );
    nvbio::vector<system_tag,int16>         out_scores( batch_size );
    nvbio::vector<system_tag,uint8>         temp_storage;

    timer.start();

    // first step: rank the query q-grams
    const uint64 n_hits = qgram_filter.rank(
        qgram_index,
        n_queries,
        nvbio::raw_pointer( sorted_qgrams ),
        nvbio::raw_pointer( sorted_indices ) );

    hipDeviceSynchronize();
    timer.stop();
    stats.rank_time   += timer.seconds();
    stats.occurrences += n_hits;

    nvbio::vector<device_tag, aln::BestSink<int16> >  sinks( batch_size );
    nvbio::vector<device_tag,string_infix_coord_type> genome_infix_coords( batch_size );
    nvbio::vector<device_tag,string_infix_coord_type> read_infix_coords( batch_size );

    const static uint32 BAND_LEN = 31;

    // loop through large batches of hits and locate & merge them
    for (uint64 hits_begin = 0; hits_begin < n_hits; hits_begin += batch_size)
    {
        typedef io::SequenceDataAccess<DNA_N>                               read_access_type;
        typedef read_access_type::sequence_string_set_type                  read_string_set_type;
        typedef read_access_type::sequence_stream_type                      read_stream;

        // build an access pointer to the sequence data
        const read_access_type reads_access( reads );

        const uint64 hits_end = nvbio::min( hits_begin + batch_size, n_hits );

        timer.start();

        qgram_filter.locate(
            hits_begin,
            hits_end,
            hits.begin() );

        const uint32 n_merged = qgram_filter.merge(
            merge_intv,
            hits_end - hits_begin,
            hits.begin(),
            merged_hits.begin(),
            merged_counts.begin() );

        hipDeviceSynchronize();
        timer.stop();
        stats.locate_time += timer.seconds();
        stats.merged      += n_merged;

        timer.start();

        // build the set of read infixes
        thrust::transform(
            merged_hits.begin(),
            merged_hits.begin() + hits_end - hits_begin,
            read_infix_coords.begin(),
            read_infixes( nvbio::plain_view( reads ) ) );

        // build the set of genome infixes
        thrust::transform(
            merged_hits.begin(),
            merged_hits.begin() + hits_end - hits_begin,
            genome_infix_coords.begin(),
            genome_infixes<BAND_LEN>( genome_len, nvbio::plain_view( reads ) ) );

        typedef nvbio::vector<device_tag,string_infix_coord_type>::const_iterator infix_iterator;

        // build a view of the reads
        const SparseStringSet<read_stream,infix_iterator> read_infix_set(
            hits_end - hits_begin,
            reads_access.sequence_stream(),
            read_infix_coords.begin() );

        const SparseStringSet<genome_string,infix_iterator> genome_infix_set(
            hits_end - hits_begin,
            genome,
            genome_infix_coords.begin() );

        typedef aln::MyersTag<5u> myers_dna5_tag;
        aln::batch_banded_alignment_score<BAND_LEN>(
            aln::make_edit_distance_aligner<aln::SEMI_GLOBAL, myers_dna5_tag>(),
            read_infix_set,
            genome_infix_set,
            sinks.begin(),
            aln::DeviceThreadScheduler(),
            reads.max_sequence_len(),
            reads.max_sequence_len() + BAND_LEN );

        hipDeviceSynchronize();
        timer.stop();
        stats.align_time += timer.seconds();

        // compute the best score for each read in this batch;
        // note that we divide the string-id by 2 to merge results coming from the forward
        // and reverse-complemented strands
        cuda::reduce_by_key(
            n_merged,
            thrust::make_transform_iterator(
                merged_hits.begin(),
                make_composition_functor( divide_by_two(), component_functor<diagonal_type>( 1u ) ) ), // take the second component divided by 2
            thrust::make_transform_iterator( sinks.begin(), sink_score() ),
            out_reads.begin(),
            out_scores.begin(),
            thrust::maximum<int16>(),
            temp_storage );

        // and keep track of the global best
        update_scores(
            n_merged,
            nvbio::plain_view( out_reads ),
            nvbio::plain_view( out_scores ),
            nvbio::plain_view( best_scores ) );
    }
}

// main test entry point
//
int main(int argc, char* argv[])
{
    //
    // perform some basic option parsing
    //

    const uint32 batch_reads   =   1*1024*1024;
    const uint32 batch_bps     = 100*1024*1024;
    const uint32 queries_batch =  16*1024*1024;

    const char* reads = argv[argc-1];
    const char* index = argv[argc-2];

    uint32 Q                = 20;
    uint32 Q_intv           = 10;
    uint32 merge_intv       = 16;
    uint32 max_reads        = uint32(-1);
    int16  score_threshold  = -20;

    for (int i = 0; i < argc; ++i)
    {
        if (strcmp( argv[i], "-q" ) == 0)
        {
            Q      = uint32( atoi( argv[++i] ) );
            Q_intv = uint32( atoi( argv[++i] ) );
        }
        if (strcmp( argv[i], "-m" ) == 0)
            merge_intv = uint32( atoi( argv[++i] ) );
        else if (strcmp( argv[i], "-max-reads" ) == 0)
            max_reads = uint32( atoi( argv[++i] ) );
        else if (strcmp( argv[i], "-t" ) == 0)
            score_threshold = int16( atoi( argv[++i] ) );
    }

    log_info(stderr, "qmap... started\n");

    // load a genome archive...
    log_visible(stderr, "  loading reference index ... started\n");
    log_info(stderr, "  file: \"%s\"\n", index);

    io::SequenceDataHost h_genome_data;
    if (io::load_sequence_file( DNA, &h_genome_data, index ) == false)
    {
        log_error(stderr, "    failed loading index \"%s\"\n", index);
        return 1u;
    }

    log_visible(stderr, "  loading reference index ... done\n");
    log_verbose(stderr, "    sequences : %u\n", h_genome_data.size() );
    log_verbose(stderr, "    bps       : %u\n", h_genome_data.bps() );
    log_verbose(stderr, "    avg bps   : %u (min: %u, max: %u)\n",
        h_genome_data.avg_sequence_len(),
        h_genome_data.min_sequence_len(),
        h_genome_data.max_sequence_len() );

    // build its device version
    const io::SequenceDataDevice      d_genome_data( h_genome_data );
    const io::SequenceDataAccess<DNA> d_genome_access( d_genome_data );

    typedef io::SequenceDataAccess<DNA>::sequence_stream_type genome_type;

    // fetch the genome string
    const uint32      genome_len = d_genome_data.bps();
    const genome_type d_genome( d_genome_access.sequence_stream() );

    // open a read file
    log_info(stderr, "  opening reads file... started\n");

    SharedPointer<io::SequenceDataStream> read_data_file(
        io::open_sequence_file(
            reads,
            io::Phred33,
            2*max_reads,
            uint32(-1),
            io::SequenceEncoding( io::FORWARD | io::REVERSE_COMPLEMENT ) ) );

    // check whether the file opened correctly
    if (read_data_file == NULL || read_data_file->is_ok() == false)
    {
        log_error(stderr, "    failed opening file \"%s\"\n", reads);
        return 1u;
    }
    log_info(stderr, "  opening reads file... done\n");

    // keep stats
    Stats stats;

    io::SequenceDataHost h_read_data;

    while (1)
    {
        // load a batch of reads
        if (io::next( DNA_N, &h_read_data, read_data_file.get(), batch_reads, batch_bps ) == 0)
            break;

        log_info(stderr, "  loading reads... started\n");

        // copy it to the device
        const io::SequenceDataDevice d_read_data( h_read_data );
        const io::SequenceDataAccess<DNA_N> d_read_access( d_read_data );

        const uint32 n_reads = d_read_data.size() / 2;

        log_info(stderr, "  loading reads... done\n");
        log_info(stderr, "    %u reads\n", n_reads);

        // prepare some typedefs for the involved string-sets and infixes
        typedef io::SequenceDataAccess<DNA_N>                                   read_access_type;     // the read view type
        typedef read_access_type::sequence_string_set_type                      string_set_type;    // the read string-set
        typedef string_set_infix_coord_type                                     infix_coord_type;   // the infix coordinate type, for string-sets
        typedef nvbio::vector<device_tag,infix_coord_type>                      infix_vector_type;  // the device vector type for infix coordinates
        typedef InfixSet<string_set_type, const string_set_infix_coord_type*>   seed_set_type;      // the infix-set type for representing seeds

        // fetch the actual read string-set
        const string_set_type d_read_string_set = d_read_access.sequence_string_set();

        // build the q-gram index
        QGramSetIndexDevice qgram_index;

        qgram_set_index_build(
            Q,
            Q_intv,
            d_read_string_set,
            qgram_index );

        typedef QGramFilterDevice<QGramSetIndexDevice,const uint64*,const uint32*> qgram_filter_type;
        qgram_filter_type qgram_filter;

        float time = 0.0f;

        const int16 worst_score = Field_traits<int16>::min();
        nvbio::vector<device_tag,int16> best_scores( n_reads, worst_score );
        nvbio::vector<device_tag,uint8> temp_storage;

        // stream through the genome
        for (uint32 genome_begin = 0; genome_begin < genome_len; genome_begin += queries_batch)
        {
            const uint32 genome_end = nvbio::min( genome_begin + queries_batch, genome_len );

            Timer timer;
            timer.start();

            map(
                qgram_index,
                qgram_filter,
                merge_intv,
                d_read_data,
                genome_end - genome_begin,
                genome_len,
                genome_begin,
                d_genome,
                best_scores,
                stats );

            hipDeviceSynchronize();
            timer.stop();
            time += timer.seconds();

            const float genome_ratio = float( genome_end ) / float( genome_len );

            log_verbose(stderr, "\r  aligned %5.2f%% of genome (%6.2f K reads/s)", 100.0f * genome_ratio, (1.0e-3f * n_reads) * genome_ratio / time  );
        }
        log_verbose_cont(stderr, "\n");

        // accumulate the number of aligned reads
        stats.reads += n_reads;
        stats.time  += time;

        // count how many reads have a score >= score_threshold
        const uint32 n_aligned = cuda::reduce(
            n_reads,
            thrust::make_transform_iterator( best_scores.begin(), above_threshold( score_threshold ) ),
            thrust::plus<uint32>(),
            temp_storage );

        stats.aligned += n_aligned;

        log_info(stderr, "  aligned %6.2f %% reads (%6.2f K reads/s)\n", 100.0f * float( stats.aligned ) / float( stats.reads ), (1.0e-3f * float( stats.reads )) / stats.time);
        log_verbose(stderr, "  breakdown:\n");
        log_verbose(stderr, "    extract throughput : %.2f B q-grams/s\n", (1.0e-9f * float( stats.queries )) / stats.extract_time);
        log_verbose(stderr, "    rank throughput    : %6.2f K reads/s\n", (1.0e-3f * float( stats.reads )) / stats.rank_time);
        log_verbose(stderr, "                       : %6.2f B seeds/s\n", (1.0e-9f * float( stats.queries )) / stats.rank_time);
        log_verbose(stderr, "    locate throughput  : %6.2f K reads/s\n", (1.0e-3f * float( stats.reads )) / stats.locate_time);
        log_verbose(stderr, "    align throughput   : %6.2f K reads/s\n", (1.0e-3f * float( stats.reads )) / stats.align_time);
        log_verbose(stderr, "                       : %6.2f M hits/s\n",  (1.0e-6f * float( stats.merged )) / stats.align_time);
        log_verbose(stderr, "    occurrences        : %.3f B\n", 1.0e-9f * float( stats.occurrences ) );
        log_verbose(stderr, "    merged occurrences : %.3f B (%.1f %%)\n", 1.0e-9f * float( stats.merged ), 100.0f * float(stats.merged)/float(stats.occurrences));
    }

    log_info(stderr, "qmap... done\n");
    return 0;
}
