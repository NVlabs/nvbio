/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// alloc_test.cu
//

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/cuda/arch.h>

namespace nvbio {

int alloc_test()
{
    log_info( stderr, "alloc test... started\n" );
    const uint32 N_TESTS = 32;

    for (size_t size = 1024*1024; size <= size_t(1u << 30); size *= 4)
    {
        Timer timer;

        float cuda_malloc_time = 0.0f;
        float cuda_free_time   = 0.0f;

        float malloc_time = 0.0f;
        float free_time   = 0.0f;

        for (uint32 i = 0; i < N_TESTS; ++i)
        {
            void* ptr;

            // cuda
            timer.start();
            hipMalloc( &ptr, size );
            timer.stop();

            cuda_malloc_time += timer.seconds();

            timer.start();
            hipFree( ptr );
            timer.stop();

            cuda_free_time += timer.seconds();

            // cpu
            timer.start();
            ptr = malloc( size );
            timer.stop();

            malloc_time += timer.seconds();

            timer.start();
            free( ptr );
            timer.stop();

            free_time += timer.seconds();
        }

        const float GB = float(1024*1024*1024);

        cuda_malloc_time /= N_TESTS;
        cuda_free_time /= N_TESTS;
        malloc_time /= N_TESTS;
        free_time /= N_TESTS;

        log_info( stderr, "  %u MB:\n", size/(1024*1024) );
        log_info( stderr, "    cuda malloc : %.2f ms, %.3f GB/s\n", cuda_malloc_time*1000.0f, (float(size)/(cuda_malloc_time)) / GB );
        log_info( stderr, "    cuda free   : %.2f ms, %.3f GB/s\n", cuda_free_time*1000.0f,   (float(size)/(cuda_free_time)) / GB );
        log_info( stderr, "    malloc      : %.2f ms, %.3f GB/s\n", malloc_time*1000.0f,      (float(size)/(malloc_time)) / GB );
        log_info( stderr, "    free        : %.2f ms, %.3f GB/s\n", free_time*1000.0f,        (float(size)/(free_time)) / GB );
    }
    log_info( stderr, "alloc test... done\n" );
    return 0;
}

} // namespace nvbio
