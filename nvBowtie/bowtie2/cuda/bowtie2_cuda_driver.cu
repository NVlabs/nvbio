#include "hip/hip_runtime.h"
/*
 * nvbio
 * Copyright (c) 2011-2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *    * Redistributions of source code must retain the above copyright
 *      notice, this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA CORPORATION nor the
 *      names of its contributors may be used to endorse or promote products
 *      derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//#define NVBIO_ENABLE_PROFILING

#define MOD_NAMESPACE
#define MOD_NAMESPACE_BEGIN namespace bowtie2 { namespace driver {
#define MOD_NAMESPACE_END   }}
#define MOD_NAMESPACE_NAME bowtie2::driver

#include <nvBowtie/bowtie2/cuda/bowtie2_cuda_driver.h>
#include <nvBowtie/bowtie2/cuda/defs.h>
#include <nvBowtie/bowtie2/cuda/fmindex_def.h>
#include <nvBowtie/bowtie2/cuda/params.h>
#include <nvBowtie/bowtie2/cuda/stats.h>
#include <nvBowtie/bowtie2/cuda/persist.h>
#include <nvBowtie/bowtie2/cuda/scoring.h>
#include <nvBowtie/bowtie2/cuda/mapq.h>
#include <nvBowtie/bowtie2/cuda/input_thread.h>
#include <nvBowtie/bowtie2/cuda/aligner.h>
#include <nvBowtie/bowtie2/cuda/aligner_inst.h>
#include <nvbio/basic/cuda/arch.h>
#include <nvbio/basic/timer.h>
#include <nvbio/basic/console.h>
#include <nvbio/basic/options.h>
#include <nvbio/basic/threads.h>
#include <nvbio/basic/atomics.h>
#include <nvbio/basic/html.h>
#include <nvbio/basic/dna.h>
#include <nvbio/basic/version.h>
#include <nvbio/fmindex/bwt.h>
#include <nvbio/fmindex/ssa.h>
#include <nvbio/fmindex/fmindex.h>
#include <nvbio/fmindex/fmindex_device.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <algorithm>
#include <numeric>
#include <functional>

namespace nvbio {
namespace bowtie2 {
namespace cuda {

std::map<std::string,std::string> load_options(const char* name)
{
    std::map<std::string,std::string> options;

    FILE* file = fopen( name, "r" );
    if (file == NULL)
    {
        log_warning( stderr, "failed opening \"%s\"\n", name );
        return options;
    }

    char key[1024];
    char value[1024];

    while (fscanf( file, "%s %s", key, value ) == 2)
        options[ key ] = std::string( value );

    fclose( file );

    return options;
}

// bogus implementation of a function to check if a string is a number
bool is_number(const char* str, uint32 len = uint32(-1))
{
    if (str[0] == '-')
        ++str;

    for (uint32 l = 0; *str != '\0' && l < len; ++l)
    {
        const char c = *str; ++str;
        if (c == '.')             continue;
        if (c >= '0' && c <= '9') continue;
        return false;
    }
    return true;
}

// bogus implementation of a function to check if an option is a function
SimpleFunc parse_function(const char* str, const SimpleFunc def)
{
    if (str[1] != ',')
        return def;

    if (!(str[0] == 'C' ||
          str[0] == 'L' ||
          str[0] == 'G' ||
          str[0] == 'S'))
          return def;

    SimpleFunc ret;
    ret.type = (str[0] == 'C') ? SimpleFunc::LinearFunc :
               (str[0] == 'L') ? SimpleFunc::LinearFunc :
               (str[0] == 'G') ? SimpleFunc::LogFunc    :
                                 SimpleFunc::SqrtFunc;

    std::string nums = std::string( str + 2 );
    const size_t c = nums.find(',');
    if (c == std::string::npos)
        return def;

    if (is_number( nums.c_str(), (uint32)c )      == false) return def;
    if (is_number( nums.c_str() + c + 1 ) == false) return def;

    const std::string num1 = nums.substr( 0, c );
    const std::string num2 = std::string( nums.c_str() + c + 1 );

    ret.k = (float)atof( num1.c_str() );
    ret.m = (float)atof( nums.c_str() + c + 1 );

    // take care of transforming constant functions in linear ones
    if (str[0] == 'C')
    {
        //ret.k += ret.m;
        ret.m = 0.0f;
    }
    return ret;
}

template <typename options_type>
SimpleFunc func_option(const options_type& options, const char* name, const SimpleFunc func)
{
    return (options.find( std::string(name) ) != options.end()) ?
        parse_function( options.find(std::string(name))->second.c_str(), func ) :
        func;
}

template <typename options_type>
SimpleFunc func_option(const options_type& options, const char* name1, const char* name2, const SimpleFunc func)
{
    return
        (options.find( std::string(name1) ) != options.end()) ?
            parse_function( options.find(std::string(name1))->second.c_str(), func ) :
        (options.find( std::string(name2) ) != options.end()) ?
            parse_function( options.find(std::string(name2))->second.c_str(), func ) :
            func;
}

void parse_options(Params& params, const std::map<std::string,std::string>& options, bool init)
{
    const bool   old_local        = params.alignment_type == LocalAlignment;
    const uint32 old_scoring_mode = params.scoring_mode;

    params.mode             = mapping_mode( string_option(options, "mode",    init ? "best" : mapping_mode( params.mode )).c_str() ); // mapping mode
    params.scoring_mode     = scoring_mode( string_option(options, "scoring", init ? "sw"   : scoring_mode( params.scoring_mode )).c_str() ); // scoring mode
    params.alignment_type   = uint_option(options, "local",                 init ? 0u      : params.alignment_type == LocalAlignment ) ? LocalAlignment : EndToEndAlignment;           // local alignment
    params.keep_stats       = (bool)uint_option(options, "stats",           init ? 1u      : params.keep_stats);           // keep stats
    params.max_hits         = uint_option(options, "max-hits",              init ? 100u    : params.max_hits);             // too big = memory exhaustion 
    params.max_dist         = uint_option(options, "max-dist",              init ? 15u     : params.max_dist);             // must be <= MAX_BAND_LEN/2
    params.max_effort_init  = uint_option(options, "max-effort-init",       init ? 15u     : params.max_effort_init);      // initial scoring effort limit
    params.max_effort       = uint_option(options, "max-effort",    "D",    init ? 15u     : params.max_effort);           // scoring effort limit
    params.min_ext          = uint_option(options, "min-ext",               init ? 30u     : params.min_ext);              // min # of extensions
    params.max_ext          = uint_option(options, "max-ext",               init ? 400u    : params.max_ext);              // max # of extensions
    params.max_reseed       = uint_option(options, "max-reseed",    "R",    init ? 2u      : params.max_reseed);           // max # of reseeding rounds
    params.rep_seeds        = uint_option(options, "rep-seeds",             init ? 1000u   : params.rep_seeds);            // reseeding threshold
    params.allow_sub        = uint_option(options, "N",                     init ? 0u      : params.allow_sub);            // allow substitution in seed
    params.mapq_filter      = uint_option(options, "mapQ-filter",   "Q",    init ? 0u      : params.mapq_filter);          // filter anything below this
    params.report           = string_option(options, "report",              init ? ""      : params.report.c_str());       // generate a report file
    params.scoring_file     = string_option(options, "scoring-scheme",      init ? ""      : params.scoring_file.c_str());
    params.randomized       =(bool) uint_option(options, "rand",                  init ? 1u      : params.randomized);           // use randomized selection
    params.randomized       =(bool)!uint_option(options, "no-rand",                               !params.randomized);           // don't use randomized selection
    params.top_seed         = uint_option(options, "top",                   init ? 0u      : params.top_seed);             // explore top seed entirely
    params.min_read_len     = uint_option(options, "min-read-len",          init ? 12u     : params.min_read_len);         // minimum read length
    params.ungapped_mates   = uint_option(options, "ungapped-mates", "ug",  init ? 0u      : params.ungapped_mates);       // ungapped mate alignment

    // force the all-mapping mode with the '--all|-a' option
    if (uint_option(options, "all", "a", params.mode == AllMapping))
        params.mode = AllMapping;

    // force Edit-Distance scoring with the '--ed' option
    if (uint_option(options, "ed", params.scoring_mode == EditDistanceMode))
        params.scoring_mode = EditDistanceMode;

    // force Smith-Waterman scoring with the '--sw' option
    if (uint_option(options, "sw", params.scoring_mode == SmithWatermanMode))
        params.scoring_mode = SmithWatermanMode;

    const bool local = params.alignment_type == LocalAlignment;

    // set the default seeding values, or reset them if the alignment type has been changed
    if (init || (local != old_local))
    {
        params.seed_len  = local ? 20 : 22u;
        params.seed_freq = SimpleFunc( SimpleFunc::SqrtFunc, 1.0f, (local ? 0.75f : 1.15) );
    }

    params.seed_len         = uint_option(options,  "seed-len",      "L",                    params.seed_len);      // no greater than 32
    params.seed_freq        = func_option( options, "seed-freq",     "i",                    params.seed_freq );    // seed interval
    params.subseed_len      = uint_option(options,  "subseed-len",          init ? 0u      : params.subseed_len);   // no greater than 32

    params.pe_overlap    = (bool) uint_option(options, "overlap",          init ? 1u      : params.pe_overlap);            // paired-end overlap
    params.pe_overlap    = (bool)!uint_option(options, "no-overlap",                       !params.pe_overlap);            // paired-end overlap
    params.pe_dovetail   = (bool) uint_option(options, "dovetail",         init ? 0u      : params.pe_dovetail);           // paired-end dovetail
    params.pe_unpaired   = (bool)!uint_option(options, "no-mixed",         init ? 0u      :!params.pe_unpaired);           // paired-end no-mixed
    params.min_frag_len  = uint_option(options, "minins", "I",      init ? 0u      : params.min_frag_len);          // paired-end minimum fragment length
    params.max_frag_len  = uint_option(options, "maxins", "X",      init ? 500u    : params.max_frag_len);          // paired-end maximum fragment length

    // the maximum batch of reads processed in parallel
    params.max_batch_size = uint_option(options, "batch-size",  init ? 1024u : params.max_batch_size );   // maximum batch size

    // internal controls
    params.scoring_window   =       uint_option(options, "scoring-window",   init ? 32u        : params.scoring_window);       // scoring window size
    params.debug.read_id    = (uint32)int_option(options, "debug-read",      init ? -1         : (int32)params.debug.read_id); // debug read id
    params.debug.select     = (bool)uint_option(options, "debug-select",     init ? 0u         : params.debug.select);       // debug select kernel
    params.debug.locate     = (bool)uint_option(options, "debug-locate",     init ? 0u         : params.debug.locate);       // debug locate kernel
    params.debug.score      = (bool)uint_option(options, "debug-score",      init ? 1u         : params.debug.score);        // debug score kernel
    params.debug.score_bad  = (bool)uint_option(options, "debug-score-bad",  init ? 0u         : params.debug.score_bad);    // debug score bad
    params.debug.score_info = (bool)uint_option(options, "debug-score-info", init ? 0u         : params.debug.score_info);   // debug score info
    params.debug.reduce     = (bool)uint_option(options, "debug-reduce",     init ? 1u         : params.debug.reduce);       // debug reduce kernel
    params.debug.traceback  = (bool)uint_option(options, "debug-traceback",  init ? 1u         : params.debug.traceback);    // debug traceback kernel
    params.debug.asserts    = (bool)uint_option(options, "debug-asserts",    init ? 1u         : params.debug.asserts);      // debug asserts

    params.persist_batch     =  int_option(options, "persist-batch",         init ? -1         : params.persist_batch);         // persist pass
    params.persist_seeding   =  int_option(options, "persist-seeding",       init ? -1         : params.persist_seeding);       // persist pass
    params.persist_extension =  int_option(options, "persist-extension",     init ? -1         : params.persist_extension);     // persist pass
    params.persist_file      =  string_option(options, "persist-file",       init ? ""         : params.persist_file.c_str() ); // persist file

    params.no_multi_hits     =  int_option(options, "no-multi-hits",      init ? 0          : params.no_multi_hits ); // disable multi-hit selection

    params.max_effort_init = nvbio::max( params.max_effort_init, params.max_effort );
    params.max_ext         = nvbio::max( params.max_ext,         params.max_effort );

    UberScoringScheme& sc = params.scoring_scheme;

    // set the default ED values, or reset them if the scoring mode has been changed
    if (init || (params.scoring_mode != old_scoring_mode))
        sc.ed.m_score_min = SimpleFunc( SimpleFunc::LinearFunc, -(float)params.max_dist, 0.0f );

    // set the default SW values, or reset them if the alignment type has been changed
    if (init || (local != old_local))
    {
        sc.sw = local ? 
            SmithWatermanScoringScheme<>::local() :
            SmithWatermanScoringScheme<>();
    }

    // load scoring scheme from file
    if (params.scoring_file != "")
        sc.sw = load_scoring_scheme( params.scoring_file.c_str(), AlignmentType( params.alignment_type ) );

    // score-min
    sc.ed.m_score_min = func_option( options, "score-min", sc.ed.m_score_min );
    sc.sw.m_score_min = func_option( options, "score-min", sc.sw.m_score_min );

    // match bonus
    sc.sw.m_match.m_val = int_option( options, "ma", sc.sw.m_match.m_val );

    // mismatch penalties
    const int2 mp = int2_option( options, "mp", make_int2( sc.sw.m_mmp.m_max_val, sc.sw.m_mmp.m_min_val ) );
    sc.sw.m_mmp.m_max_val = mp.x;
    sc.sw.m_mmp.m_min_val = mp.y;

    // np
    sc.sw.m_np.m_val = int_option( options, "np", sc.sw.m_np.m_val );

    // read gaps
    const int2 rdg         = int2_option( options, "rdg", make_int2( sc.sw.m_read_gap_const, sc.sw.m_read_gap_coeff ) );
    sc.sw.m_read_gap_const = rdg.x;
    sc.sw.m_read_gap_coeff = rdg.y;

    // reference gaps
    const int2 rfg        = int2_option( options, "rfg", make_int2( sc.sw.m_ref_gap_const, sc.sw.m_ref_gap_coeff ) );
    sc.sw.m_ref_gap_const = rfg.x;
    sc.sw.m_ref_gap_coeff = rfg.y;
}

//
// single-end driver
//
int driver(
    const char*                              output_name, 
    const io::SequenceData&                  reference_data_host,
    const io::FMIndexData&                   driver_data_host,
          io::SequenceDataStream&            read_data_stream,
    const std::map<std::string,std::string>& options,
    const std::string&                       cmdline,
    const std::string&                       rg_id,
    const std::string&                       rg_string)
{
    log_visible(stderr, "Bowtie2 cuda driver... started\n");

    // WARNING: we don't do any error checking on passed parameters!
    Params params;
    {
        bool init = true;
        std::string config = string_option(options, "config", "" );
        if (config != "") { parse_options( params, load_options( config.c_str() ), init ); init = false; }
                            parse_options( params, options,                        init );

    }
    if (params.alignment_type == LocalAlignment &&
        params.scoring_mode == EditDistanceMode)
    {
        log_warning(stderr, "edit-distance scoring is incompatible with local alignment, switching to Smith-Waterman\n");
        params.scoring_mode = SmithWatermanMode;
    }

    // build an empty report
    FILE* html_output = (params.report != std::string("")) ? fopen( params.report.c_str(), "w" ) : NULL;
    if (html_output)
    {
        // encapsulate the document
        {
            html::html_object html( html_output );
            {
                const char* meta_list = "<meta http-equiv=\"refresh\" content=\"1\" />";

                { html::header_object hd( html_output, "Bowtie2 Report", html::style(), meta_list ); }
                { html::body_object body( html_output ); }
            }
        }
        fclose( html_output );
    }

    // compute band length
    const uint32 band_len = Aligner::band_length( params.max_dist );
    const SimpleFunc& score_min = params.scoring_mode == EditDistanceMode ? params.scoring_scheme.ed.m_score_min : params.scoring_scheme.sw.m_score_min;
    
    // print command line options
    log_visible(stderr, "  mode           = %s\n", mapping_mode( params.mode ));
    log_visible(stderr, "  scoring        = %s\n", scoring_mode( params.scoring_mode ));
    log_visible(stderr, "  score-min      = %s:%.2f:%.2f\n", score_min.type_string(), score_min.k, score_min.m);
    log_visible(stderr, "  alignment type = %s\n", params.alignment_type == LocalAlignment ? "local" : "end-to-end");
    log_visible(stderr, "  seed length    = %u\n", params.seed_len);
    log_visible(stderr, "  seed interval  = (%s, %.3f, %.3f)\n", params.seed_freq.type_symbol(), params.seed_freq.k, params.seed_freq.m);
    log_visible(stderr, "  seed rounds    = %u\n", params.max_reseed);
    log_visible(stderr, "  max hits       = %u\n", params.max_hits);
    log_visible(stderr, "  max edit dist  = %u (band len %u)\n", params.max_dist, band_len);
    log_visible(stderr, "  max effort     = %u\n", params.max_effort);
    log_visible(stderr, "  substitutions  = %u\n", params.allow_sub);
    log_visible(stderr, "  mapQ filter    = %u\n", params.mapq_filter);
    log_visible(stderr, "  randomized     = %s\n", params.randomized ? "yes" : "no");
    if (params.allow_sub)
        log_visible(stderr, "  subseed length = %u\n", params.subseed_len);

    const bool need_reverse =
        (params.allow_sub == 0 && USE_REVERSE_INDEX) ||
        (params.allow_sub == 1 && params.subseed_len == 0 && params.mode == BestMappingApprox);

    Timer timer;

    timer.start();

    io::SequenceDataDevice reference_data( reference_data_host );

    io::FMIndexDataDevice driver_data( driver_data_host,
                        io::FMIndexDataDevice::FORWARD |
        (need_reverse ? io::FMIndexDataDevice::REVERSE : 0u) |
                        io::FMIndexDataDevice::SA );

    timer.stop();

    log_stats(stderr, "  allocated device driver data (%.2f GB - %.1fs)\n", float(driver_data.allocated()) / 1.0e9f, timer.seconds() );

    typedef FMIndexDef::type fm_index_type;

    fm_index_type fmi  = driver_data.index();
    fm_index_type rfmi = driver_data.rindex();

    Aligner aligner;

    size_t free, total;
    hipMemGetInfo(&free, &total);
    log_stats(stderr, "  device has %ld of %ld MB free\n", free/1024/1024, total/1024/1024);

    uint32 BATCH_SIZE;

    for (BATCH_SIZE = params.max_batch_size*1024; BATCH_SIZE >= 16*1024; BATCH_SIZE /= 2)
    {
        // leave some guard band of free memory
        const uint32 guard_band = 600*1024*1024;

        // gauge how much memory we'd need
        const std::pair<uint64,uint64> mem_stats = aligner.init_alloc( BATCH_SIZE, params, kSingleEnd, false );
        if (mem_stats.second < free - guard_band)
            break;
    }
    log_stats(stderr, "  processing reads in batches of %uK\n", BATCH_SIZE/1024);

    if (aligner.init( BATCH_SIZE, params, kSingleEnd ) == false)
        return 1;

    nvbio::cuda::check_error("cuda initializations");

    hipMemGetInfo(&free, &total);
    log_stats(stderr, "  ready to start processing: device has %ld MB free\n", free/1024/1024);

    float polling_time = 0.0f;
    Timer global_timer;
    global_timer.start();

    UberScoringScheme& scoring_scheme = params.scoring_scheme;

    Stats stats( params );

    aligner.output_file = io::OutputFile::open(output_name,
                                               io::SINGLE_END,
                                               io::BNT(reference_data_host));

    aligner.output_file->set_rg( rg_id.c_str(), rg_string.c_str() );
    aligner.output_file->set_program(
        "nvBowtie",
        "nvBowtie",
        NVBIO_VERSION_STRING,
        cmdline.c_str() );

    aligner.output_file->configure_mapq_evaluator(params.mapq_filter);

    aligner.output_file->header();

    // setup the input thread
    InputThread input_thread( &read_data_stream, stats, BATCH_SIZE );
    input_thread.create();

    uint32 input_set  = 0;
    uint32 n_reads    = 0;

    io::SequenceDataHost local_read_data_host;

    // loop through the batches of reads
    for (uint32 read_begin = 0; true; read_begin += BATCH_SIZE)
    {
        Timer polling_timer;
        polling_timer.start();

        // poll until the current input set is loaded...
        while (input_thread.read_data[ input_set ] == NULL) { yield(); }

        // make sure the other writes are seen
        host_acquire_fence();

        polling_timer.stop();
        polling_time += polling_timer.seconds();

        io::SequenceDataHost* read_data_host = input_thread.read_data[ input_set ];
        if (read_data_host == (io::SequenceDataHost*)InputThread::INVALID)
        {
            log_verbose(stderr, "end of input reached\n");
            break;
        }

        if (read_data_host->max_sequence_len() > Aligner::MAX_READ_LEN)
        {
            log_error(stderr, "unsupported read length %u (maximum is %u)\n",
                read_data_host->max_sequence_len(),
                Aligner::MAX_READ_LEN );
            break;
        }

        // make a local copy of the host batch
        local_read_data_host = *read_data_host;

        // mark this set as ready to be reused
        input_thread.read_data[ input_set ] = NULL;

        // make sure the other threads see this change
        host_release_fence();

        // advance input set pointer
        input_set = (input_set + 1) % InputThread::BUFFERS;

        Timer timer;
        timer.start();

        aligner.output_file->start_batch( &local_read_data_host );

        io::SequenceDataDevice read_data( local_read_data_host );
        hipDeviceSynchronize();

        timer.stop();
        stats.read_HtoD.add( read_data.size(), timer.seconds() );

        const uint32 count = read_data.size();
        log_info(stderr, "aligning reads [%u, %u]\n", read_begin, read_begin + count - 1u);
        log_verbose(stderr, "  %u reads\n", count);
        log_verbose(stderr, "  %.3f M bps (%.1f MB)\n", float(read_data.bps())/1.0e6f, float(read_data.words()*sizeof(uint32)+read_data.bps()*sizeof(char))/float(1024*1024));
        log_verbose(stderr, "  %.1f bps/read (min: %u, max: %u)\n", float(read_data.bps())/float(read_data.size()), read_data.min_sequence_len(), read_data.max_sequence_len());

        if (params.mode == AllMapping)
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                all_ed(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    stats );
            }
            else
            {
                all_sw(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    stats );
            }
        }
        else
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                best_approx_ed(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    stats );
            }
            else
            {
                best_approx_sw(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data,
                    stats );
            }
        }

        global_timer.stop();
        stats.global_time += global_timer.seconds();
        global_timer.start();

        aligner.output_file->end_batch();

        // increase the total reads counter
        n_reads += count;

        log_verbose(stderr, "  %.1f K reads/s\n", 1.0e-3f * float(n_reads) / stats.global_time);
    }

    input_thread.join();

    io::IOStats iostats;

    aligner.output_file->close();

    // transfer I/O statistics to the old stats struct
    iostats = aligner.output_file->get_aggregate_statistics();

    stats.alignments_DtoH.add(iostats.alignments_DtoH_count, iostats.alignments_DtoH_time);
    stats.io = iostats.output_process_timings;
    stats.n_reads           = n_reads;
    stats.n_mapped          = stats.mate1.n_mapped;
    stats.n_ambiguous       = stats.mate1.n_ambiguous;
    stats.n_nonambiguous    = stats.mate1.n_unambiguous;
    stats.n_unique          = stats.mate1.n_unique;
    stats.n_multiple        = stats.mate1.n_multiple;
    stats.mapped            = stats.mate1.mapped_ed_histogram;
    stats.f_mapped          = stats.mate1.mapped_ed_histogram_fwd;
    stats.r_mapped          = stats.mate1.mapped_ed_histogram_rev;
    memcpy(stats.mapq_bins, stats.mate1.mapq_bins,             sizeof(stats.mate1.mapq_bins));
    memcpy(stats.mapped2,   stats.mate1.mapped_ed_correlation, sizeof(stats.mate1.mapped_ed_correlation));

    delete aligner.output_file;

    global_timer.stop();
    stats.global_time += global_timer.seconds();

    nvbio::bowtie2::cuda::generate_report(stats, params.report.c_str());

    log_stats(stderr, "  total        : %.2f sec (avg: %.1fK reads/s).\n", stats.global_time, 1.0e-3f * float(n_reads)/stats.global_time);
    log_stats(stderr, "  mapping      : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.map.time, 1.0e-6f * stats.map.avg_speed(), 1.0e-6f * stats.map.max_speed, stats.map.device_time);
    log_stats(stderr, "  selecting    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.select.time, 1.0e-6f * stats.select.avg_speed(), 1.0e-6f * stats.select.max_speed, stats.select.device_time);
    log_stats(stderr, "  sorting      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.sort.time, 1.0e-6f * stats.sort.avg_speed(), 1.0e-6f * stats.sort.max_speed, stats.sort.device_time);
    log_stats(stderr, "  scoring      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.score.time, 1.0e-6f * stats.score.avg_speed(), 1.0e-6f * stats.score.max_speed, stats.score.device_time);
    log_stats(stderr, "  locating     : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.locate.time, 1.0e-6f * stats.locate.avg_speed(), 1.0e-6f * stats.locate.max_speed, stats.locate.device_time);
    log_stats(stderr, "  backtracking : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.backtrack.time, 1.0e-6f * stats.backtrack.avg_speed(), 1.0e-6f * stats.backtrack.max_speed, stats.backtrack.device_time);
    log_stats(stderr, "  finalizing   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.finalize.time, 1.0e-6f * stats.finalize.avg_speed(), 1.0e-6f * stats.finalize.max_speed, stats.finalize.device_time);
    log_stats(stderr, "  results DtoH : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.alignments_DtoH.time, 1.0e-6f * stats.alignments_DtoH.avg_speed(), 1.0e-6f * stats.alignments_DtoH.max_speed);
    log_stats(stderr, "  reads HtoD   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.read_HtoD.time, 1.0e-6f * stats.read_HtoD.avg_speed(), 1.0e-6f * stats.read_HtoD.max_speed);
    log_stats(stderr, "  reads I/O    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.read_io.time, 1.0e-6f * stats.read_io.avg_speed(), 1.0e-6f * stats.read_io.max_speed);
    log_stats(stderr, "    exposed    : %.2f sec (avg: %.3fK reads/s).\n", polling_time, 1.0e-3f * float(n_reads)/polling_time);
    log_stats(stderr, "  output I/O   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.io.time, 1.0e-6f * stats.io.avg_speed(), 1.0e-6f * stats.io.max_speed);

    std::vector<uint32>& mapped         = stats.mapped;
    uint32&              n_mapped       = stats.n_mapped;
    uint32&              n_unique       = stats.n_unique;
    uint32&              n_ambiguous    = stats.n_ambiguous;
    uint32&              n_nonambiguous = stats.n_nonambiguous;
    uint32&              n_multiple     = stats.n_multiple;
    {
        log_stats(stderr, "  mapped reads : %.2f %% - of these:\n", 100.0f * float(n_mapped)/float(n_reads) );
        log_stats(stderr, "    aligned uniquely      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_unique)/float(n_mapped), 100.0f * float(n_mapped - n_multiple)/float(n_reads) );
        log_stats(stderr, "    aligned unambiguously : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_nonambiguous)/float(n_mapped), 100.0f * float(n_nonambiguous)/float(n_reads) );
        log_stats(stderr, "    aligned ambiguously   : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_ambiguous)/float(n_mapped), 100.0f * float(n_ambiguous)/float(n_reads) );
        log_stats(stderr, "    aligned multiply      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_multiple)/float(n_mapped), 100.0f * float(n_multiple)/float(n_reads) );
        for (uint32 i = 0; i < mapped.size(); ++i)
        {
            if (float(mapped[i])/float(n_reads) > 1.0e-3f)
                log_stats(stderr, "    ed %4u : %.1f %%\n", i,
                100.0f * float(mapped[i])/float(n_reads) );
        }
    }

    log_visible(stderr, "Bowtie2 cuda driver... done\n");
    return 0;
}

//
// paired-end driver
//
int driver(
    const char*                              output_name, 
    const io::SequenceData&                  reference_data_host,
    const io::FMIndexData&                   driver_data_host,
    const io::PairedEndPolicy                pe_policy,
          io::SequenceDataStream&            read_data_stream1,
          io::SequenceDataStream&            read_data_stream2,
    const std::map<std::string,std::string>& options,
    const std::string&                       cmdline,
    const std::string&                       rg_id,
    const std::string&                       rg_string)
{
    log_visible(stderr, "Bowtie2 cuda driver... started\n");

    // WARNING: we don't do any error checking on passed parameters!
    Params params;
    params.pe_policy = pe_policy;
    {
        bool init = true;
        std::string config = string_option(options, "config", "" );
        if (config != "") { parse_options( params, load_options( config.c_str() ), init ); init = false; }
                            parse_options( params, options,                        init );
    }
    if (params.alignment_type == LocalAlignment &&
        params.scoring_mode == EditDistanceMode)
    {
        log_warning(stderr, "edit-distance scoring is incompatible with local alignment, switching to Smith-Waterman\n");
        params.scoring_mode = SmithWatermanMode;
    }

    // clear the persistance files
    if (params.persist_file != "")
        persist_clear( params.persist_file );

    // build an empty report
    FILE* html_output = (params.report != std::string("")) ? fopen( params.report.c_str(), "w" ) : NULL;
    if (html_output)
    {
        // encapsulate the document
        {
            html::html_object html( html_output );
            {
                const char* meta_list = "<meta http-equiv=\"refresh\" content=\"1\" />";

                { html::header_object hd( html_output, "Bowtie2 Report", html::style(), meta_list ); }
                { html::body_object body( html_output ); }
            }
        }
        fclose( html_output );
    }

    // compute band length
    const uint32 band_len = Aligner::band_length( params.max_dist );
    const SimpleFunc& score_min = EditDistanceMode ? params.scoring_scheme.ed.m_score_min : params.scoring_scheme.sw.m_score_min;

    // print command line options
    log_visible(stderr, "  mode           = %s\n", mapping_mode( params.mode ));
    log_visible(stderr, "  scoring        = %s\n", scoring_mode( params.scoring_mode ));
    log_visible(stderr, "  score-min      = %s:%.2f:%.2f\n", score_min.type_string(), score_min.k, score_min.m);
    log_visible(stderr, "  alignment type = %s\n", params.alignment_type == LocalAlignment ? "local" : "end-to-end");
    log_visible(stderr, "  pe-policy      = %s\n",
                                                   pe_policy == io::PE_POLICY_FF ? "ff" :
                                                   pe_policy == io::PE_POLICY_FR ? "fr" :
                                                   pe_policy == io::PE_POLICY_RF ? "rf" :
                                                                                   "rr" );
    log_visible(stderr, "  seed length    = %u\n", params.seed_len);
    log_visible(stderr, "  seed interval  = (%s, %.3f, %.3f)\n", params.seed_freq.type_symbol(), params.seed_freq.k, params.seed_freq.m);
    log_visible(stderr, "  seed rounds    = %u\n", params.max_reseed);
    log_visible(stderr, "  max hits       = %u\n", params.max_hits);
    log_visible(stderr, "  max edit dist  = %u (band len %u)\n", params.max_dist, band_len);
    log_visible(stderr, "  max effort     = %u\n", params.max_effort);
    log_visible(stderr, "  substitutions  = %u\n", params.allow_sub);
    log_visible(stderr, "  mapQ filter    = %u\n", params.mapq_filter);
    log_visible(stderr, "  randomized     = %s\n", params.randomized ? "yes" : "no");
    if (params.allow_sub)
        log_visible(stderr, "  subseed length = %u\n", params.subseed_len);

    const bool need_reverse =
        (params.allow_sub == 0 && USE_REVERSE_INDEX) ||
        (params.allow_sub == 1 && params.subseed_len == 0 && params.mode == BestMappingApprox);

    io::SequenceDataDevice reference_data( reference_data_host );

    io::FMIndexDataDevice driver_data( driver_data_host,
                        io::FMIndexDataDevice::FORWARD |
        (need_reverse ? io::FMIndexDataDevice::REVERSE : 0u) |
                        io::FMIndexDataDevice::SA );

    log_stats(stderr, "  allocated device driver data (%.2f GB)\n", float(driver_data.allocated()) / 1.0e9f );

    typedef FMIndexDef::type fm_index_type;

    fm_index_type fmi  = driver_data.index();
    fm_index_type rfmi = driver_data.rindex();

    Aligner aligner;

    size_t free, total;
    hipMemGetInfo(&free, &total);
    log_stats(stderr, "  device has %ld of %ld MB free\n", free/1024/1024, total/1024/1024);

    uint32 BATCH_SIZE;

    for (BATCH_SIZE = params.max_batch_size*1024; BATCH_SIZE >= 16*1024; BATCH_SIZE /= 2)
    {
        // leave some guard band of free memory
        const uint32 guard_band = 600*1024*1024;

        // gauge how much memory we'd need
        const std::pair<uint64,uint64> mem_stats = aligner.init_alloc( BATCH_SIZE, params, kPairedEnds, false );
        if (mem_stats.second < free - guard_band)
            break;
    }
    log_stats(stderr, "  processing reads in batches of %uK\n", BATCH_SIZE/1024);

    if (aligner.init( BATCH_SIZE, params, kPairedEnds ) == false)
        return 1;

    nvbio::cuda::check_error("cuda initializations");

    hipMemGetInfo(&free, &total);
    log_stats(stderr, "  ready to start processing: device has %ld MB free\n", free/1024/1024);

    size_t stack_size_limit;
    hipDeviceGetLimit( &stack_size_limit, hipLimitStackSize );
    log_debug(stderr, "    max cuda stack size: %u\n", stack_size_limit);

    float polling_time = 0.0f;
    Timer timer;
    Timer global_timer;
    global_timer.start();

    UberScoringScheme& scoring_scheme = params.scoring_scheme;

    Stats stats( params );

    aligner.output_file = io::OutputFile::open(output_name,
                                               io::PAIRED_END,
                                               io::BNT(reference_data_host));

    aligner.output_file->set_rg( rg_id.c_str(), rg_string.c_str() );
    aligner.output_file->set_program(
        "nvBowtie",
        "nvBowtie",
        NVBIO_VERSION_STRING,
        cmdline.c_str() );

    aligner.output_file->configure_mapq_evaluator(params.mapq_filter);

    aligner.output_file->header();

    // setup the input thread
    InputThreadPaired input_thread( &read_data_stream1, &read_data_stream2, stats, BATCH_SIZE );
    input_thread.create();

    uint32 input_set  = 0;
    uint32 n_reads    = 0;

    io::SequenceDataHost local_read_data_host1;
    io::SequenceDataHost local_read_data_host2;

    // loop through the batches of reads
    for (uint32 read_begin = 0; true; read_begin += BATCH_SIZE)
    {
        Timer polling_timer;
        polling_timer.start();

        // poll until the current input set is loaded...
        while (input_thread.read_data1[ input_set ] == NULL ||
               input_thread.read_data2[ input_set ] == NULL) { yield(); }

        // make sure the other writes are seen
        host_acquire_fence();

        polling_timer.stop();
        polling_time += polling_timer.seconds();

        io::SequenceDataHost* read_data_host1 = input_thread.read_data1[ input_set ];
        io::SequenceDataHost* read_data_host2 = input_thread.read_data2[ input_set ];
        if (read_data_host1 == (io::SequenceDataHost*)InputThread::INVALID ||
            read_data_host2 == (io::SequenceDataHost*)InputThread::INVALID)
        {
            log_verbose(stderr, "end of input reached\n");
            break;
        }

        if ((read_data_host1->max_sequence_len() > Aligner::MAX_READ_LEN) ||
            (read_data_host2->max_sequence_len() > Aligner::MAX_READ_LEN))
        {
            log_error(stderr, "unsupported read length %u (maximum is %u)\n",
                nvbio::max(read_data_host1->max_sequence_len(), read_data_host2->max_sequence_len()),
                Aligner::MAX_READ_LEN );
            break;
        }

        // make a local copy of the host batch
        local_read_data_host1 = *read_data_host1;
        local_read_data_host2 = *read_data_host2;

        // mark this set as ready to be reused
        input_thread.read_data1[ input_set ] = NULL;
        input_thread.read_data2[ input_set ] = NULL;

        // make sure the other threads see this change
        host_release_fence();

        // advance input set pointer
        input_set = (input_set + 1) % InputThread::BUFFERS;

        Timer timer;
        timer.start();

        aligner.output_file->start_batch( &local_read_data_host1, &local_read_data_host2 );

        io::SequenceDataDevice read_data1( local_read_data_host1/*, io::ReadDataDevice::READS | io::ReadDataDevice::QUALS*/ );
        io::SequenceDataDevice read_data2( local_read_data_host2/*, io::ReadDataDevice::READS | io::ReadDataDevice::QUALS*/ );

        timer.stop();
        stats.read_HtoD.add( read_data1.size(), timer.seconds() );

        const uint32 count = read_data1.size();
        log_info(stderr, "aligning reads [%u, %u]\n", read_begin, read_begin + count - 1u);
        log_verbose(stderr, "  %u reads\n", count);
        log_verbose(stderr, "  %.3f M bps (%.1f MB)\n",
            float(read_data1.bps() + read_data2.bps())/1.0e6f,
            float(read_data1.words()*sizeof(uint32)+read_data1.bps()*sizeof(char))/float(1024*1024)+
            float(read_data2.words()*sizeof(uint32)+read_data2.bps()*sizeof(char))/float(1024*1024));
        log_verbose(stderr, "  %.1f bps/read (min: %u, max: %u)\n",
            float(read_data1.bps()+read_data2.bps())/float(read_data1.size()+read_data2.size()),
            nvbio::min( read_data1.min_sequence_len(), read_data2.min_sequence_len() ),
            nvbio::max( read_data1.max_sequence_len(), read_data2.max_sequence_len() ));

        if (params.mode == AllMapping)
        {
            log_error(stderr, "paired-end all-mapping is not yet supported!\n");
            exit(1);
        }
        else
        {
            if (params.scoring_mode == EditDistanceMode)
            {
                best_approx_ed(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data1,
                    read_data2,
                    stats );
            }
            else
            {
                best_approx_sw(
                    aligner,
                    params,
                    fmi,
                    rfmi,
                    scoring_scheme,
                    reference_data,
                    driver_data,
                    read_data1,
                    read_data2,
                    stats );
            }
        }

        global_timer.stop();
        stats.global_time += global_timer.seconds();
        global_timer.start();

        aligner.output_file->end_batch();

        // increase the total reads counter
        n_reads += count;

        log_verbose(stderr, "  %.1f K reads/s\n", 1.0e-3f * float(n_reads) / stats.global_time);
    }

    input_thread.join();

    io::IOStats iostats;

    aligner.output_file->close();

    // transfer I/O statistics
    iostats = aligner.output_file->get_aggregate_statistics();

    stats.alignments_DtoH.add(iostats.alignments_DtoH_count, iostats.alignments_DtoH_time);
    stats.io                = iostats.output_process_timings;
    stats.n_reads           = n_reads;
    stats.n_mapped          = stats.paired.n_mapped;
    stats.n_ambiguous       = stats.paired.n_ambiguous;
    stats.n_nonambiguous    = stats.paired.n_unambiguous;
    stats.n_unique          = stats.paired.n_unique;
    stats.n_multiple        = stats.paired.n_multiple;
    stats.mapped            = stats.paired.mapped_ed_histogram;
    stats.f_mapped          = stats.paired.mapped_ed_histogram_fwd;
    stats.r_mapped          = stats.paired.mapped_ed_histogram_rev;
    memcpy(stats.mapq_bins, stats.paired.mapq_bins,             sizeof(stats.paired.mapq_bins));
    memcpy(stats.mapped2,   stats.paired.mapped_ed_correlation, sizeof(stats.paired.mapped_ed_correlation));

    delete aligner.output_file;

    global_timer.stop();
    stats.global_time += global_timer.seconds();

    nvbio::bowtie2::cuda::generate_report(stats, params.report.c_str());

    log_stats(stderr, "  total          : %.2f sec (avg: %.1fK reads/s).\n", stats.global_time, 1.0e-3f * float(n_reads)/stats.global_time);
    log_stats(stderr, "  mapping        : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.map.time, 1.0e-6f * stats.map.avg_speed(), 1.0e-6f * stats.map.max_speed, stats.map.device_time);
    log_stats(stderr, "  scoring        : %.2f sec (avg: %.1fM reads/s, max: %.3fM reads/s, %.2f device sec).).\n", stats.scoring_pipe.time, 1.0e-6f * stats.scoring_pipe.avg_speed(), 1.0e-6f * stats.scoring_pipe.max_speed, stats.scoring_pipe.device_time);
    log_stats(stderr, "    selecting    : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.select.time, 1.0e-6f * stats.select.avg_speed(), 1.0e-6f * stats.select.max_speed, stats.select.device_time);
    log_stats(stderr, "    sorting      : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.sort.time, 1.0e-6f * stats.sort.avg_speed(), 1.0e-6f * stats.sort.max_speed, stats.sort.device_time);
    log_stats(stderr, "    scoring(a)   : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.score.time, 1.0e-6f * stats.score.avg_speed(), 1.0e-6f * stats.score.max_speed, stats.score.device_time);
    log_stats(stderr, "    scoring(o)   : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.opposite_score.time, 1.0e-6f * stats.opposite_score.avg_speed(), 1.0e-6f * stats.opposite_score.max_speed, stats.opposite_score.device_time);
    log_stats(stderr, "    locating     : %.2f sec (avg: %.3fM seeds/s, max: %.3fM seeds/s, %.2f device sec).\n", stats.locate.time, 1.0e-6f * stats.locate.avg_speed(), 1.0e-6f * stats.locate.max_speed, stats.locate.device_time);
    log_stats(stderr, "  backtracing(a) : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.backtrack.time, 1.0e-6f * stats.backtrack.avg_speed(), 1.0e-6f * stats.backtrack.max_speed, stats.backtrack.device_time);
    log_stats(stderr, "  backtracing(o) : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.backtrack_opposite.time, 1.0e-6f * stats.backtrack_opposite.avg_speed(), 1.0e-6f * stats.backtrack_opposite.max_speed, stats.backtrack_opposite.device_time);
    log_stats(stderr, "  finalizing     : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s, %.2f device sec).\n", stats.finalize.time, 1.0e-6f * stats.finalize.avg_speed(), 1.0e-6f * stats.finalize.max_speed, stats.finalize.device_time);
    log_stats(stderr, "  results DtoH   : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.alignments_DtoH.time, 1.0e-6f * stats.alignments_DtoH.avg_speed(), 1.0e-6f * stats.alignments_DtoH.max_speed);
    log_stats(stderr, "  reads HtoD     : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.read_HtoD.time, 1.0e-6f * stats.read_HtoD.avg_speed(), 1.0e-6f * stats.read_HtoD.max_speed);
    log_stats(stderr, "  reads I/O      : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.read_io.time, 1.0e-6f * stats.read_io.avg_speed(), 1.0e-6f * stats.read_io.max_speed);
    log_stats(stderr, "    exposed      : %.2f sec (avg: %.3fK reads/s).\n", polling_time, 1.0e-3f * float(n_reads)/polling_time);
    log_stats(stderr, "  output I/O     : %.2f sec (avg: %.3fM reads/s, max: %.3fM reads/s).\n", stats.io.time, 1.0e-6f * stats.io.avg_speed(), 1.0e-6f * stats.io.max_speed);

    std::vector<uint32>& mapped         = stats.mapped;
    uint32&              n_mapped       = stats.n_mapped;
    uint32&              n_unique       = stats.n_unique;
    uint32&              n_ambiguous    = stats.n_ambiguous;
    uint32&              n_nonambiguous = stats.n_nonambiguous;
    uint32&              n_multiple     = stats.n_multiple;
    {
        log_stats(stderr, "  concordant reads : %.2f %% - of these:\n", 100.0f * float(n_mapped)/float(n_reads) );
        log_stats(stderr, "    aligned uniquely      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_unique)/float(n_mapped), 100.0f * float(n_mapped - n_multiple)/float(n_reads) );
        log_stats(stderr, "    aligned unambiguously : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_nonambiguous)/float(n_mapped), 100.0f * float(n_nonambiguous)/float(n_reads) );
        log_stats(stderr, "    aligned ambiguously   : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_ambiguous)/float(n_mapped), 100.0f * float(n_ambiguous)/float(n_reads) );
        log_stats(stderr, "    aligned multiply      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(n_multiple)/float(n_mapped), 100.0f * float(n_multiple)/float(n_reads) );
        for (uint32 i = 0; i < mapped.size(); ++i)
        {
            if (float(mapped[i])/float(n_reads) > 1.0e-3f)
                log_stats(stderr, "    ed %4u : %.1f %%\n", i,
                100.0f * float(mapped[i])/float(n_reads) );
        }

        log_stats(stderr, "  mate1 : %.2f %% - of these:\n", 100.0f * float(stats.mate1.n_mapped)/float(n_reads) );
        if (stats.mate1.n_mapped)
        {
            log_stats(stderr, "    aligned uniquely      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate1.n_unique)/float(stats.mate1.n_mapped), 100.0f * float(stats.mate1.n_mapped - stats.mate1.n_multiple)/float(n_reads) );
            log_stats(stderr, "    aligned unambiguously : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate1.n_unambiguous)/float(stats.mate1.n_mapped), 100.0f * float(stats.mate1.n_unambiguous)/float(n_reads) );
            log_stats(stderr, "    aligned ambiguously   : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate1.n_ambiguous)/float(stats.mate1.n_mapped), 100.0f * float(stats.mate1.n_ambiguous)/float(n_reads) );
            log_stats(stderr, "    aligned multiply      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate1.n_multiple)/float(stats.mate1.n_mapped), 100.0f * float(stats.mate1.n_multiple)/float(n_reads) );
        }

        log_stats(stderr, "  mate2 : %.2f %% - of these:\n", 100.0f * float(stats.mate2.n_mapped)/float(n_reads) );
        if (stats.mate2.n_mapped)
        {
            log_stats(stderr, "    aligned uniquely      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate2.n_unique)/float(stats.mate2.n_mapped), 100.0f * float(stats.mate2.n_mapped - stats.mate2.n_multiple)/float(n_reads) );
            log_stats(stderr, "    aligned unambiguously : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate2.n_unambiguous)/float(stats.mate2.n_mapped), 100.0f * float(stats.mate2.n_unambiguous)/float(n_reads) );
            log_stats(stderr, "    aligned ambiguously   : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate2.n_ambiguous)/float(stats.mate2.n_mapped), 100.0f * float(stats.mate2.n_ambiguous)/float(n_reads) );
            log_stats(stderr, "    aligned multiply      : %4.1f%% (%4.1f%% of total)\n", 100.0f * float(stats.mate2.n_multiple)/float(stats.mate2.n_mapped), 100.0f * float(stats.mate2.n_multiple)/float(n_reads) );
        }
    }

    log_visible(stderr, "Bowtie2 cuda driver... done\n");
    return 0;
}

} // namespace cuda
} // namespace bowtie2
} // namespace nvbio
